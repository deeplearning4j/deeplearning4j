/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//
#include <array/ExtraArguments.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <array>

#include "testlayers.h"

using namespace sd;

class LambdaTests : public NDArrayTests {
 public:
  LambdaTests() {
  }
};

template <typename Lambda>
SD_KERNEL void runLambda(double *input, double *output, LongType length, Lambda lambda) {
  auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (LongType e = tid; e < length; e += gridDim.x * blockDim.x) {
    output[e] = lambda(input[e]);
  }
}

void launcher(hipStream_t stream, double *input, double *output, LongType length) {
  auto f = LAMBDA_D(x) { return x + 1.; };

  runLambda<<<128, 128, 128, stream>>>(input, output, length, f);
}
