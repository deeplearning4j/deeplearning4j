#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//
#include <array/NDArray.h>
#include <array/NDArrayFactory.h>
#include <hip/hip_runtime.h>
#include <execution/LaunchContext.h>
#include <graph/Context.h>
#include <graph/Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>

#include <ops/declarable/CustomOperations.h>
#include <ops/specials_cuda.h>

#include "testlayers.h"

using namespace sd;
using namespace sd::graph;

class NDArrayCudaBasicsTests : public NDArrayTests {
 public:
};

//////////////////////////////////////////////////////////////////////////
static hipError_t allocateDeviceMem(LaunchContext& lc, std::vector<void*>& devicePtrs,
                                     const std::vector<std::pair<void*, size_t>>& hostData) {
  if (devicePtrs.size() != hostData.size())
    THROW_EXCEPTION("prepareDataForCuda: two input sts::vectors should same sizes !");

  hipError_t cudaResult;

  void* reductionPointer;
  cudaResult = hipMalloc(reinterpret_cast<void**>(&reductionPointer), 1024 * 1024);
  if (cudaResult != 0) return cudaResult;
  int* allocationPointer;
  cudaResult = hipMalloc(reinterpret_cast<void**>(&allocationPointer), 1024 * 1024);
  if (cudaResult != 0) return cudaResult;

  lc.setReductionPointer(reductionPointer);
  lc.setAllocationPointer(allocationPointer);
  hipStream_t stream = *lc.getCudaStream();

  for (int i = 0; i < devicePtrs.size(); ++i) {
    cudaResult = hipMalloc(reinterpret_cast<void**>(&devicePtrs[i]), hostData[i].second);
    if (cudaResult != 0) return cudaResult;
    hipMemcpyAsync(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice, stream);
  }
  return cudaResult;
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_1) {
  auto x = NDArrayFactory::create<int>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<int>('c', {5}, {5, 4, 3, 2, 1});

  ASSERT_TRUE(x.isActualOnDeviceSide());
  ASSERT_FALSE(x.isActualOnHostSide());
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_2) {
  auto x = NDArrayFactory::create<int>('c', {5});
  auto y = NDArrayFactory::create<int>('c', {5});

  ASSERT_TRUE(x.isActualOnDeviceSide());
  ASSERT_FALSE(x.isActualOnHostSide());
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_3) {
  auto x = NDArrayFactory::create<int>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<int>('c', {5}, {5, 4, 3, 2, 1});

  ASSERT_TRUE(x.isActualOnDeviceSide());
  ASSERT_FALSE(x.isActualOnHostSide());

  NDArray::registerSpecialUse({&x}, {&y});

  ASSERT_TRUE(x.isActualOnDeviceSide());
  ASSERT_FALSE(x.isActualOnHostSide());

  ASSERT_TRUE(y.isActualOnDeviceSide());
  ASSERT_FALSE(y.isActualOnHostSide());
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_01) {
  auto x = NDArrayFactory::create_<int>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create_<int>('c', {5}, {5, 4, 3, 2, 1});

  ASSERT_TRUE(x->isActualOnDeviceSide());
  ASSERT_FALSE(x->isActualOnHostSide());
  delete x;
  delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_02) {
  auto x = NDArrayFactory::create_<int>('c', {5});
  auto y = NDArrayFactory::create_<int>('c', {5});

  ASSERT_TRUE(x->isActualOnDeviceSide());
  ASSERT_FALSE(x->isActualOnHostSide());
  delete x;
  delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_03) {
  auto x = NDArrayFactory::create_<int>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create_<int>('c', {5}, {5, 4, 3, 2, 1});

  ASSERT_TRUE(x->isActualOnDeviceSide());
  ASSERT_FALSE(x->isActualOnHostSide());

  NDArray::registerSpecialUse({y}, {x});
  x->applyTransform(transform::Neg, *y);

  delete x;
  delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_Cosine_1) {
  auto x = NDArrayFactory::create_<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create_<double>('c', {5}, {5, 4, 3, 2, 1});

  ASSERT_TRUE(x->isActualOnDeviceSide());
  ASSERT_FALSE(x->isActualOnHostSide());

  NDArray::registerSpecialUse({y}, {x});
  x->applyTransform(transform::Cosine, *y);
  delete x;
  delete y;
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_1) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto z = NDArrayFactory::create<double>('c', {5}, {10, 10, 10, 10, 10});

  auto exp = NDArrayFactory::create<double>('c', {5}, {2, 4, 6, 8, 10});

  // making raw buffers

  Pointer nativeStream = (Pointer)malloc(sizeof(hipStream_t));
  CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
  hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t>(&nativeStream));
  auto stream = reinterpret_cast<hipStream_t>(&nativeStream);


  LaunchContext lc(stream, nullptr, nullptr);
  NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, x.buffer(), x.shapeInfo(), x.specialBuffer(),
                                             x.specialShapeInfo(), y.buffer(), y.shapeInfo(), y.specialBuffer(),
                                             y.specialShapeInfo(), z.buffer(), z.shapeInfo(), z.specialBuffer(),
                                             z.specialShapeInfo(), nullptr);
  z.tickWriteDevice();
  auto res = hipStreamSynchronize(stream);
  ASSERT_EQ(0, res);

  for (int e = 0; e < z.lengthOf(); e++) ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_2) {
  // allocating host-side arrays
  NDArray x('c', {5}, {1, 2, 3, 4, 5});
  NDArray y('c', {5}, {1, 2, 3, 4, 5});
  NDArray z('c', {5}, DOUBLE);

  NDArray exp('c', {5}, {2, 4, 6, 8, 10});

  Pointer nativeStream = (Pointer)malloc(sizeof(hipStream_t));
  CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
  hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t>(&nativeStream));
  auto stream = reinterpret_cast<hipStream_t>(&nativeStream);

  LaunchContext lc(stream, stream, nullptr, nullptr);
  NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, nullptr, x.shapeInfo(), x.specialBuffer(),
                                             x.specialShapeInfo(), nullptr, y.shapeInfo(), y.specialBuffer(),
                                             y.specialShapeInfo(), nullptr, z.shapeInfo(), z.specialBuffer(),
                                             z.specialShapeInfo(), nullptr);
  auto res = hipStreamSynchronize(stream);
  ASSERT_EQ(0, res);

  for (int e = 0; e < z.lengthOf(); e++) ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_3) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto z = NDArrayFactory::create<double>('c', {5}, {10, 10, 10, 10, 10});

  auto exp = NDArrayFactory::create<double>('c', {5}, {2, 4, 6, 8, 10});

  Pointer nativeStream = (Pointer)malloc(sizeof(hipStream_t));
  CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
  hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t>(&nativeStream));
  auto stream = reinterpret_cast<hipStream_t>(&nativeStream);

  LaunchContext lc(stream, stream, nullptr, nullptr);
  NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, x.buffer(), x.shapeInfo(), x.specialBuffer(),
                                             x.specialShapeInfo(), y.buffer(), y.shapeInfo(), y.specialBuffer(),
                                             y.specialShapeInfo(), z.buffer(), z.shapeInfo(), z.specialBuffer(),
                                             z.specialShapeInfo(), nullptr);
  z.tickWriteDevice();
  auto res = hipStreamSynchronize(stream);
  ASSERT_EQ(0, res);
  z.syncToHost();
  hipMemcpy(z.buffer(), z.specialBuffer(), z.lengthOf() * z.sizeOfT(), hipMemcpyDeviceToHost);
  res = hipStreamSynchronize(stream);
  z.tickWriteHost();
  ASSERT_EQ(0, res);
  for (int e = 0; e < z.lengthOf(); e++) {
    ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
  }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_4) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto z = NDArrayFactory::create<double>('c', {5});

  auto exp = NDArrayFactory::create<double>('c', {5}, {2, 4, 6, 8, 10});
  x.applyPairwiseTransform(pairwise::Add, y, z);
  for (int e = 0; e < z.lengthOf(); e++) {
    ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
  }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_5) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});

  auto exp = NDArrayFactory::create<double>('c', {5}, {2, 4, 6, 8, 10});
  x += y;
  // x.applyPairwiseTransform(pairwise::Add, &y, &z, nullptr);
  x.syncToHost();
  for (int e = 0; e < x.lengthOf(); e++) {
    ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
  }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_6) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>(2);  //.'c', { 5 }, { 1, 2, 3, 4, 5});
  // auto z = NDArrayFactory::create<double>('c', { 5 });

  auto exp = NDArrayFactory::create<double>('c', {5}, {3, 4, 5, 6, 7});
  x += y;
  x.syncToHost();
  for (int e = 0; e < x.lengthOf(); e++) {
    ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
  }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_7) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto exp = NDArrayFactory::create<double>('c', {5}, {3, 4, 5, 6, 7});
  x += 2.;
  x.syncToHost();
  for (int e = 0; e < x.lengthOf(); e++) {
    ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
  }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_1) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto z = NDArrayFactory::create<double>('c', {5});

  auto exp = NDArrayFactory::create<double>('c', {5}, {1, 4, 9, 16, 25});

  x.applyPairwiseTransform(pairwise::Multiply, y, z);

  for (int e = 0; e < z.lengthOf(); e++) {
    ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
  }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_2) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  NDArray z('c', {5}, DOUBLE);

  auto exp = NDArrayFactory::create<double>('c', {5}, {1, 4, 9, 16, 25});
  x.applyPairwiseTransform(pairwise::Multiply, y, z);
  for (int e = 0; e < z.lengthOf(); e++) {
    ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
  }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_3) {
  // allocating host-side arrays
  NDArray x('c', {5}, {1, 2, 3, 4, 5}, DOUBLE);
  NDArray y('c', {5}, {1., 2., 3., 4., 5.}, DOUBLE);
  auto z = NDArrayFactory::create<double>('c', {5});

  auto exp = NDArrayFactory::create<double>('c', {5}, {1, 4, 9, 16, 25});
  x.applyPairwiseTransform(pairwise::Multiply, y, z);

  for (int e = 0; e < z.lengthOf(); e++) {
    ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
  }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_4) {
  // allocating host-side arrays
  NDArray x('c', {5}, {1, 2, 3, 4, 5}, DOUBLE);
  NDArray y('c', {5}, {1., 2., 3., 4., 5.}, DOUBLE);

  auto exp = NDArrayFactory::create<double>('c', {5}, {1, 4, 9, 16, 25});


  x *= y;
  for (int e = 0; e < x.lengthOf(); e++) {
    ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
  }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestPrimitiveNeg_01) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<int>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<int>('c', {5}, {1, 2, 3, 4, 5});
  auto exp = NDArrayFactory::create<int>('c', {5}, {-1, -2, -3, -4, -5});

  auto stream = x.getContext()->getCudaStream();

  NativeOpExecutioner::execTransformSame(x.getContext(), transform::Neg, x.buffer(), x.shapeInfo(), x.specialBuffer(),
                                         x.specialShapeInfo(), y.buffer(), y.shapeInfo(), y.specialBuffer(),
                                         y.specialShapeInfo(), nullptr, nullptr, nullptr);
  auto res = hipStreamSynchronize(stream);
  ASSERT_EQ(0, res);
  y.tickWriteDevice();

  for (int e = 0; e < y.lengthOf(); e++) {
    ASSERT_NEAR(exp.e<int>(e), y.e<int>(e), 1e-5);
  }
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveNeg_2) {
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5});

  ASSERT_TRUE(x.isActualOnDeviceSide());
  ASSERT_FALSE(x.isActualOnHostSide());

  x.applyTransform(transform::Neg, y);
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveSqrt_1) {  // strict
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5});
  auto exp = NDArrayFactory::create<double>({1.000000, 1.414214, 1.732051, 2.000000, 2.236068});
  ASSERT_TRUE(x.isActualOnDeviceSide());
  ASSERT_FALSE(x.isActualOnHostSide());

  x.applyTransform(transform::Sqrt, y);
  ASSERT_TRUE(y.equalsTo(exp));
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveAssign_1) {  // strict
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5});

  x.applyTransform(transform::Assign, y);
  ASSERT_TRUE(y.equalsTo(x));
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveCosine_1) {  // strict
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5});
  auto exp = NDArrayFactory::create<double>('c', {5}, {0.540302, -0.416147, -0.989992, -0.653644, 0.283662});

  ASSERT_TRUE(x.isActualOnDeviceSide());
  ASSERT_FALSE(x.isActualOnHostSide());

  x.applyTransform(transform::Cosine, y);

  ASSERT_TRUE(exp.isSameShape(y));
  ASSERT_TRUE(exp.dataType() == y.dataType());

}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveCosine_2) {
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5});
  auto exp = NDArrayFactory::create<double>('c', {5}, {0.540302, -0.416147, -0.989992, -0.653644, 0.283662});

  ASSERT_TRUE(x.isActualOnDeviceSide());
  ASSERT_FALSE(x.isActualOnHostSide());
  x.applyTransform(transform::Cosine, y);
  ASSERT_TRUE(exp.isSameShape(y));
  ASSERT_TRUE(exp.dataType() == y.dataType());
  ASSERT_TRUE(exp.equalsTo(y));
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveCosine_3) {
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>('c', {5});
  auto exp = NDArrayFactory::create<double>({0.540302, -0.416147, -0.989992, -0.653644, 0.283662});

  ASSERT_TRUE(x.isActualOnDeviceSide());
  ASSERT_FALSE(x.isActualOnHostSide());
  x.applyTransform(transform::Cosine, y);
  ASSERT_TRUE(exp.isSameShape(y));
  ASSERT_TRUE(exp.equalsTo(y));

}

TEST_F(NDArrayCudaBasicsTests, TestRawBroadcast_2) {
  NDArray x = NDArrayFactory::create<double>('c', {2, 3, 4});
  NDArray y('c', {2, 4}, {10, 20, 30, 40, 50, 60, 70, 80}, DOUBLE);
  NDArray z('c', {2, 3, 4}, {100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100,
                             100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100},
            DOUBLE);

  NDArray exp('c', {2, 3, 4}, {10.,  40.,  90.,   160.,  50.,  120.,  210.,  320.,  90.,   200.,  330.,  480.,
                               650., 840., 1050., 1280., 850., 1080., 1330., 1600., 1050., 1320., 1610., 1920.},
              DOUBLE);
  x.linspace(1);
  x.syncToDevice();

  std::vector<LongType> dimensions = {0, 2};

  // evaluate xTad data
  shape::TAD xTad;
  xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
  xTad.createTadOnlyShapeInfo();
  xTad.createOffsets();

  // prepare input arrays for prepareDataForCuda function
  std::vector<std::pair<void*, size_t>> hostData;
  hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(LongType));  // 0 -- dimensions
  hostData.emplace_back(xTad.tadOnlyShapeInfo,
                        shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));     // 1 -- xTadShapeInfo
  hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(LongType));  // 2 -- xTadOffsets
  std::vector<void*> devicePtrs(hostData.size(), nullptr);

  // create cuda stream and LaunchContext
  hipError_t cudaResult;
  hipStream_t stream;
  cudaResult = hipStreamCreate(&stream);
  ASSERT_EQ(0, cudaResult);
  LaunchContext lc(&stream);

  // allocate required amount of global device memory and copy host data to it
  cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);
  ASSERT_EQ(0, cudaResult);

  // call cuda kernel which calculates result
  NativeOpExecutioner::execBroadcast(&lc, broadcast::Multiply, nullptr, x.shapeInfo(), x.specialBuffer(),
                                     x.specialShapeInfo(), nullptr, y.shapeInfo(), y.specialBuffer(),
                                     y.specialShapeInfo(), nullptr, z.shapeInfo(), z.specialBuffer(),
                                     z.specialShapeInfo(), (LongType*)devicePtrs[0], dimensions.size(),
                                     (LongType*)devicePtrs[1], (LongType*)devicePtrs[2], nullptr, nullptr);

  cudaResult = hipStreamSynchronize(stream);
  ASSERT_EQ(0, cudaResult);
  z.tickWriteDevice();

  // verify results
  for (int e = 0; e < z.lengthOf(); e++) ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

  // free allocated global device memory
  for (int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

  // delete cuda stream
  cudaResult = hipStreamDestroy(stream);
  ASSERT_EQ(0, cudaResult);
}

TEST_F(NDArrayCudaBasicsTests, TestRawBroadcast_3) {

  NDArray x('c', {2, 3, 4}, DOUBLE);
  NDArray y('c', {2, 4}, {10, 20, 30, 40, 50, 60, 70, 80}, DOUBLE);
  NDArray z('c', {2, 3, 4}, {100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100,
                             100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100},
            DOUBLE);

  NDArray exp('c', {2, 3, 4}, {10.,  40.,  90.,   160.,  50.,  120.,  210.,  320.,  90.,   200.,  330.,  480.,
                               650., 840., 1050., 1280., 850., 1080., 1330., 1600., 1050., 1320., 1610., 1920.},
              DOUBLE);
  x.linspace(1);
  x.syncToDevice();

  std::vector<LongType> dimensions = {0, 2};

  // evaluate xTad data
  shape::TAD xTad;
  xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
  xTad.createTadOnlyShapeInfo();
  xTad.createOffsets();

  // prepare input arrays for prepareDataForCuda function
  std::vector<std::pair<void*, size_t>> hostData;
  hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(LongType));  // 0 -- dimensions
  hostData.emplace_back(xTad.tadOnlyShapeInfo,
                        shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));     // 1 -- xTadShapeInfo
  hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(LongType));  // 2 -- xTadOffsets
  std::vector<void*> devicePtrs(hostData.size(), nullptr);

  // create cuda stream and LaunchContext
  hipError_t cudaResult;

  LaunchContext* pLc = x.getContext();  //(&stream);
  hipStream_t stream = pLc->getCudaStream();
  // allocate required amount of global device memory and copy host data to it

  for (int i = 0; i < devicePtrs.size(); ++i) {
    cudaResult = hipMalloc(reinterpret_cast<void**>(&devicePtrs[i]), hostData[i].second);
    ASSERT_EQ(0, cudaResult);
    hipMemcpyAsync(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice, stream);
  }

  NDArray::registerSpecialUse({&z}, {&x, &y});
  // call cuda kernel which calculates result
  NativeOpExecutioner::execBroadcast(pLc, broadcast::Multiply, nullptr, x.shapeInfo(), x.specialBuffer(),
                                     x.specialShapeInfo(), nullptr, y.shapeInfo(), y.specialBuffer(),
                                     y.specialShapeInfo(), nullptr, z.shapeInfo(), z.specialBuffer(),
                                     z.specialShapeInfo(), (LongType*)devicePtrs[0], dimensions.size(),
                                     (LongType*)devicePtrs[1], (LongType*)devicePtrs[2], nullptr, nullptr);


  // verify results
  for (int e = 0; e < z.lengthOf(); e++) ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

  // free allocated global device memory
  for (int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);
  ASSERT_TRUE(exp.equalsTo(z));

}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_1) {
  // allocating host-side arrays
  NDArray x('c', {2, 3}, {1, 2, 3, 4, 5, 6}, DOUBLE);
  NDArray y = NDArrayFactory::create<double>(3.);

  auto exp = NDArrayFactory::create<double>('c', {2, 3}, {3, 6, 9, 12, 15, 18});

  // making raw buffers
  x *= y;

  ASSERT_TRUE(exp.equalsTo(x));

}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_01) {
  // allocating host-side arrays
  NDArray x('c', {2, 3}, {1, 2, 3, 4, 5, 6}, DOUBLE);
  NDArray y = NDArrayFactory::create<double>(3.);  //'c', { 3 }, { 2., 3., 4.}, sd::DataType::DOUBLE);
  auto z = NDArrayFactory::create<double>('c', {2, 3});

  auto exp = NDArrayFactory::create<double>('c', {2, 3}, {3, 6, 9, 12, 15, 18});

  x.applyTrueBroadcast(BroadcastOpsTuple::Multiply(), y, z);  // *= y;
  ASSERT_TRUE(exp.equalsTo(z));

}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_02) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<double>('c', {2, 3}, {1, 2, 3, 4, 5, 6});  //, sd::DataType::DOUBLE);
  auto y = NDArrayFactory::create<double>('c', {2, 3},
                                          {3, 3, 3, 3, 3, 3});  //'c', { 3 }, { 2., 3., 4.}, sd::DataType::DOUBLE);
  auto z = NDArrayFactory::create<double>('c', {2, 3});

  auto exp = NDArrayFactory::create<double>('c', {2, 3}, {3, 6, 9, 12, 15, 18});
  x.applyTrueBroadcast(BroadcastOpsTuple::Multiply(), y, z);  // *= y;

  ASSERT_TRUE(exp.equalsTo(z));

}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_002) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<double>('c', {2, 3}, {1, 2, 3, 4, 5, 6});  //, sd::DataType::DOUBLE);
  auto y = NDArrayFactory::create<double>(
      'c', {2, 3}, {2., 3., 3., 3., 3., 3.});  //'c', { 3 }, { 2., 3., 4.}, sd::DataType::DOUBLE);
  auto z = NDArrayFactory::create<double>('c', {2, 3});

  auto exp = NDArrayFactory::create<double>('c', {2, 3}, {2, 6, 9, 12, 15, 18});
  x.applyPairwiseTransform(pairwise::Multiply, y, z);  // *= y;
  ASSERT_TRUE(exp.equalsTo(z));
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestBroadcastRaw_1) {
  // if (!Environment::getInstance().isExperimentalBuild())
  //    return;

  NDArray x('c', {2, 3, 4}, {100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100,
                             100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100},
            INT32);
  NDArray y('c', {3}, {10, 20, 30}, INT64);
  NDArray z('c', {2, 3, 4}, {100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100,
                             100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100, 100},
            INT32);
  NDArray exp('c', {2, 3, 4},
              {10, 11, 12, 13, 24, 25, 26, 27, 38, 39, 40, 41, 22, 23, 24, 25, 36, 37, 38, 39, 50, 51, 52, 53}, INT32);
  // real output [10, 11, 12, 13, 4, 5, 6, 7, 28, 29, 30, 31, 22, 23, 24, 25, 16, 17, 18, 19, 40, 41, 42, 43]
  x.linspace(0);
  x.syncToDevice();

  std::vector<LongType> dimensions = {1};

  // evaluate xTad data
  shape::TAD xTad;
  xTad.init(x.shapeInfo(), dimensions.data(), dimensions.size());
  xTad.createTadOnlyShapeInfo();
  xTad.createOffsets();

  // prepare input arrays for prepareDataForCuda function
  std::vector<std::pair<void*, size_t>> hostData;
  hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(LongType));  // 0 -- dimensions
  hostData.emplace_back(xTad.tadOnlyShapeInfo,
                        shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));     // 1 -- xTadShapeInfo
  hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(LongType));  // 2 -- xTadOffsets
  std::vector<void*> devicePtrs(hostData.size(), nullptr);

  // create cuda stream and LaunchContext
  hipError_t cudaResult;
  hipStream_t stream = x.getContext()->getCudaStream();
  LaunchContext* pLc = x.getContext();

  // allocate required amount of global device memory and copy host data to it
  for (size_t i = 0; i < devicePtrs.size(); ++i) {
    cudaResult = hipMalloc(&devicePtrs[i], hostData[i].second);  // if(cudaResult != 0) return cudaResult;
    ASSERT_EQ(cudaResult, 0);
    hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
  }

  // call cuda kernel which calculates result
  NativeOpExecutioner::execBroadcast(pLc, broadcast::Add, nullptr, x.shapeInfo(), x.specialBuffer(),
                                     x.specialShapeInfo(), nullptr, y.shapeInfo(), y.specialBuffer(),
                                     y.specialShapeInfo(), nullptr, z.shapeInfo(), z.specialBuffer(),
                                     z.specialShapeInfo(), (LongType*)devicePtrs[0], dimensions.size(),
                                     (LongType*)devicePtrs[1], (LongType*)devicePtrs[2], nullptr, nullptr);

  cudaResult = hipStreamSynchronize(stream);
  ASSERT_EQ(0, cudaResult);
  // free allocated global device memory
  for (int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply) {
  // allocating host-side arrays
  NDArray x('c', {2, 3}, {1, 2, 3, 4, 5, 6}, DOUBLE);
  NDArray y('c', {3}, {2., 3., 4.}, DOUBLE);
  // auto z = NDArrayFactory::create<double>('c', { 5 });

  auto exp = NDArrayFactory::create<double>('c', {2, 3}, {2, 6, 12, 8, 15, 24});
  x *= y;
}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_2) {
  // allocating host-side arrays
  NDArray x('c', {2, 3}, {1, 2, 3, 4, 5, 6}, DOUBLE);
  NDArray y('c', {3}, {2., 3., 4.}, DOUBLE);

  auto exp = NDArrayFactory::create<double>('c', {2, 3}, {11, 12, 13, 14, 15, 16});
  auto expZ = NDArrayFactory::create<double>('c', {2, 3}, {2, 6, 12, 8, 15, 24});

  x.applyTrueBroadcast(BroadcastOpsTuple::Multiply(), y, exp);
  ASSERT_TRUE(exp.equalsTo(expZ));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestReduceSum_1) {
  // allocating host-side arrays
  auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
  auto y = NDArrayFactory::create<double>(15);
  auto exp = NDArrayFactory::create<double>(15);

  auto stream = x.getContext()->getCudaStream();  // reinterpret_cast<hipStream_t *>(&nativeStream);

  NativeOpExecutioner::execReduceSameScalar(x.getContext(), reduce::Sum, x.buffer(), x.shapeInfo(), x.specialBuffer(),
                                            x.specialShapeInfo(), nullptr, y.buffer(), y.shapeInfo(), y.specialBuffer(),
                                            y.specialShapeInfo());
  auto res = hipStreamSynchronize(stream);
  ASSERT_EQ(0, res);
  y.syncToHost();

  ASSERT_NEAR(y.e<double>(0), 15, 1e-5);
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestDup1) {
  NDArray array('c', {2, 3}, {1, 2, 3, 4, 5, 6});
  auto arrC = array.dup('c');
  auto arrF = array.dup('f');
  ASSERT_TRUE(array.equalsTo(arrF));
  ASSERT_TRUE(array.equalsTo(arrC));

  ASSERT_TRUE(arrF.equalsTo(arrC));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, equalsTo_1) {
  NDArray x('c', {2, 5}, {1, 2, 3, 4, 5, 6, 7, 8, 9, 10}, DOUBLE);
  NDArray y('c', {2, 5}, {1, 2, 3, 4, 5, 6, 7, 8, 9, 10}, DOUBLE);

  ASSERT_TRUE(x.equalsTo(y));

  x.permutei({1, 0});
  y.permutei({1, 0});

  ASSERT_TRUE(x.equalsTo(y));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, equalsTo_2) {
  NDArray x('c', {2, 5}, {1, 2, 3, 4, 5, 6, 7, 8, 10, 10}, DOUBLE);
  NDArray y('c', {2, 5}, {1, 2, 5, 4, 5, 6, 7, 8, 9, 10}, DOUBLE);

  ASSERT_FALSE(x.equalsTo(y));

  x.permutei({1, 0});
  y.permutei({1, 0});

  ASSERT_FALSE(x.equalsTo(y));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, equalsTo_3) {
  NDArray x('c', {2, 5}, {1, 2, 3, 4, 5, 6, 7, 8, 9, 10}, DOUBLE);
  NDArray y('c', {2, 5}, {1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.f, 8.f, 9.f, 10.f}, FLOAT32);

  ASSERT_FALSE(x.equalsTo(y));

  x.permutei({1, 0});
  y.permutei({1, 0});

  ASSERT_FALSE(x.equalsTo(y));
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyReduce3_1) {
  NDArray x('c', {2, 3, 4}, {-10, -9, -8, -7, -6, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13},
            INT32);
  NDArray x2('c', {2, 3, 4}, {-10, -9, -8, -7, -6, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13},
             INT32);
  NDArray y('c', {2, 3, 4}, {-2, 3, -4, 5, -2, 3, -4, 5, -2, 3, -4, 5, -2, 3, -4, 5, -2, 3, -4, 5, -2, 3, -4, 5},
            INT32);
  NDArray k('c', {2, 3}, {-2, 3, -4, 5, -2, 3}, INT32);
  NDArray k2('c', {3, 2}, {-2, 3, -4, 5, -2, 3}, INT32);

  NDArray exp1('c', {3}, {4.f, 20.f, 36.f}, FLOAT32);
  NDArray exp2('c', {2, 3}, {-10.f, -2.f, 6.f, 14.f, 22.f, 30.f}, FLOAT32);
  NDArray exp3('c', {4}, {38.f, 41.f, 44.f, 47.f}, FLOAT32);
  NDArray exp4('c', {4}, {114.f, 117.f, 120.f, 123.f}, FLOAT32);

  NDArray z = x.applyReduce3(reduce3::Dot, y, {0, 2});
  ASSERT_TRUE(z.equalsTo(&exp1));

  z = x.applyReduce3(reduce3::Dot, k, {0, 1});
  ASSERT_TRUE(z.equalsTo(&exp3));

  x.permutei({0, 2, 1});
  y.permutei({0, 2, 1});

  z = y.applyReduce3(reduce3::Dot, x, {1});
  ASSERT_TRUE(z.equalsTo(&exp2));

  x2.permutei({1, 0, 2});

  z = x2.applyReduce3(reduce3::Dot, k2, {0, 1});
  ASSERT_TRUE(z.equalsTo(&exp4));
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyReduce3_2) {
  NDArray x('c', {2, 3, 4}, {-10, -9, -8.5, -7, -6, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13},
            DOUBLE);
  NDArray x2('c', {2, 3, 4}, {-10, -9, -8, -7, -6, -5, -4, -3, -2, -1, 0.5, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13},
             DOUBLE);
  NDArray y('c', {2, 3, 4}, {-2, 3, -4, 5, -2, 3, -4, 5, -2, 3, -4, 5, -2.5, 3, -4, 5, -2, 3, -4, 5, -2, 3, -4, 5},
            DOUBLE);
  NDArray k('c', {2, 3}, {-2, 3, -4, 5.5, -2, 3}, DOUBLE);
  NDArray k2('c', {3, 2}, {-2, 3, -4, 5, -2, 3.5}, DOUBLE);

  NDArray exp1('c', {3}, {5., 20., 36.}, DOUBLE);
  NDArray exp2('c', {2, 3}, {-8., -2., 6., 13., 22., 30.}, DOUBLE);
  NDArray exp3('c', {4}, {39., 42.5, 47., 49.5}, DOUBLE);
  NDArray exp4('c', {4}, {119., 122.5, 125., 129.5}, DOUBLE);

  NDArray z = x.applyReduce3(reduce3::Dot, y, {0, 2});
  ASSERT_TRUE(z.equalsTo(&exp1));

  z = x.applyReduce3(reduce3::Dot, k, {0, 1});
  ASSERT_TRUE(z.equalsTo(&exp3));

  x.permutei({0, 2, 1});
  y.permutei({0, 2, 1});

  z = y.applyReduce3(reduce3::Dot, x, {1});
  ASSERT_TRUE(z.equalsTo(&exp2));

  x2.permutei({1, 0, 2});

  z = x2.applyReduce3(reduce3::Dot, k2, {0, 1});
  ASSERT_TRUE(z.equalsTo(&exp4));
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyReduce3_3) {
  NDArray x1('c', {2, 2, 2}, {1, 2, 3, 4, 5, 6, 7, 8}, INT32);
  NDArray x2('c', {2, 2, 2}, {-1, -2, -3, -4, -5, -6, -7, -8}, INT32);
  NDArray x3('c', {3, 2}, {1.5, 1.5, 1.5, 1.5, 1.5, 1.5}, DOUBLE);
  NDArray x4('c', {3, 2}, {1, 2, 3, 4, 5, 6}, DOUBLE);

  NDArray exp1('c', {}, std::vector<double>{-204}, FLOAT32);
  NDArray exp2('c', {}, std::vector<double>{31.5}, DOUBLE);

  auto z = x1.applyReduce3(reduce3::Dot, x2);
  ASSERT_EQ(z,exp1);

  z = x3.applyReduce3(reduce3::Dot, x4);
  ASSERT_EQ(z,exp2);

  x1.permutei({2, 1, 0});
  x2.permutei({2, 1, 0});
  x3.permutei({1, 0});
  x4.permutei({1, 0});

  z = x1.applyReduce3(reduce3::Dot, x2);
  ASSERT_EQ(z,exp1);

  z = x3.applyReduce3(reduce3::Dot, x4);
  ASSERT_EQ(z,exp2);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyAllReduce3_1) {
  GTEST_SKIP() << "Hangs on cuda";

  NDArray x1('c', {2, 3, 2},
             {
                 1,
                 2,
                 3,
                 4,
                 5,
                 6,
                 7,
                 8,
                 -1,
                 -2,
                 -3,
                 -4,
             },
             INT32);
  NDArray x2('c', {2, 2, 2}, {-1, -2, -3, -4, -5, -6, -7, -8}, INT32);
  NDArray x3('c', {3, 2}, {1.5, 1.5, 1.5, 1.5, 1.5, 1.5}, DOUBLE);
  NDArray x4('c', {3, 2}, {1, 2, 3, 4, 5, 6}, DOUBLE);

  NDArray exp1('c', {3, 2}, {-88.f, -124.f, 6.f, -2.f, 22.f, 14.f}, FLOAT32);
  NDArray exp2('c', {6, 4}, {-36.f, -44.f, -52.f, -60.f, -42.f, -52.f, -62.f, -72.f, 2.f,  0.f,  -2.f, -4.f,
                             6.f,   4.f,   2.f,   0.f,   10.f,  8.f,   6.f,   4.f,   14.f, 12.f, 10.f, 8.f},
               FLOAT32);
  NDArray exp3('c', {1, 1}, std::vector<double>{31.5}, DOUBLE);
  NDArray exp4('c', {3, 3}, {4.5, 10.5, 16.5, 4.5, 10.5, 16.5, 4.5, 10.5, 16.5}, DOUBLE);

  std::vector<LongType> dims = {0, 1, 2};
  std::vector<LongType> dims0 = {0};
  std::vector<LongType> dims1 = {1};
  std::vector<LongType> dims01 = {0,1};
  std::vector<LongType> dims02 = {0,2};


  auto z = x1.applyAllReduce3(reduce3::Dot, x2, &dims02);
  ASSERT_TRUE(z.equalsTo(&exp1));

  z = x1.applyAllReduce3(reduce3::Dot, x2, &dims0);
  ASSERT_TRUE(z.equalsTo(&exp2));

  z = x3.applyAllReduce3(reduce3::Dot, x4, &dims01);
  ASSERT_TRUE(z.equalsTo(&exp3));

  z = x3.applyAllReduce3(reduce3::Dot, x4, &dims1);
  ASSERT_TRUE(z.equalsTo(&exp4));

  x1.permutei({2, 1, 0});
  x2.permutei({2, 1, 0});
  x3.permutei({1, 0});
  x4.permutei({1, 0});

  z = x1.applyAllReduce3(reduce3::Dot, x2,&dims02);
  ASSERT_TRUE(z.equalsTo(&exp1));

  z = x3.applyAllReduce3(reduce3::Dot, x4, {0});
  ASSERT_TRUE(z.equalsTo(&exp4));
}

//////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyIndexReduce_test1) {
  NDArray x('c', {2, 3}, {0, 10, 1, 2, 2.5, -4}, DOUBLE);

  NDArray scalar('c', {}, std::vector<double>{100}, INT64);
  NDArray vec1('c', {2}, {100, 100}, INT64);
  NDArray vec2('c', {3}, {100, 100, 100}, INT64);

  NDArray exp1('c', {}, std::vector<double>{1}, INT64);
  NDArray exp2('c', {2}, {1, 1}, INT64);
  NDArray exp3('c', {3}, {1, 0, 0}, INT64);

  NDArray exp4('c', {}, std::vector<double>{2}, INT64);
  NDArray exp5('c', {2}, {1, 1}, INT64);
  NDArray exp6('c', {3}, {1, 0, 0}, INT64);

  std::vector<LongType> dims = {0, 1, 2};
  std::vector<LongType> dims0 = {0};
  std::vector<LongType> dims1 = {1};
  std::vector<LongType> dims01 = {0,1};

  x.applyIndexReduce(indexreduce::IndexMax, scalar, &dims01);
  ASSERT_TRUE(scalar.equalsTo(&exp1));

  x.applyIndexReduce(indexreduce::IndexMax, vec1, &dims1);
  ASSERT_TRUE(vec1.equalsTo(&exp2));

  x.applyIndexReduce(indexreduce::IndexMax, vec2, &dims0);
  ASSERT_TRUE(vec2.equalsTo(&exp3));

  x.permutei({1, 0});

  x.applyIndexReduce(indexreduce::IndexMax, scalar, &dims01);
  ASSERT_TRUE(scalar.equalsTo(&exp4));

  x.applyIndexReduce(indexreduce::IndexMax, vec1, &dims0);
  ASSERT_TRUE(vec1.equalsTo(&exp5));

  x.applyIndexReduce(indexreduce::IndexMax, vec2, &dims1);
  ASSERT_TRUE(vec2.equalsTo(&exp6));
}

//////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyIndexReduce_test2) {
  NDArray x('c', {2, 3}, {0, 10, 1, 2, 2.5, -4}, DOUBLE);

  NDArray exp1('c', {}, std::vector<double>{1}, INT64);
  NDArray exp2('c', {2}, {1, 1}, INT64);
  NDArray exp3('c', {3}, {1, 0, 0}, INT64);

  NDArray exp4('c', {}, std::vector<double>{2}, INT64);
  NDArray exp5('c', {2}, {1, 1}, INT64);
  NDArray exp6('c', {3}, {1, 0, 0}, INT64);

  std::vector<LongType> dims = {0, 1};
  std::vector<LongType> dims1 = {1};
  std::vector<LongType> dims0 = {0};
  auto z = x.applyIndexReduce(indexreduce::IndexMax, &dims);
  ASSERT_TRUE(z.equalsTo(&exp1));

  z = x.applyIndexReduce(indexreduce::IndexMax,&dims1);
  ASSERT_TRUE(z.equalsTo(&exp2));

  z = x.applyIndexReduce(indexreduce::IndexMax, &dims0);
  ASSERT_TRUE(z.equalsTo(&exp3));

  x.permutei({1, 0});

  z = x.applyIndexReduce(indexreduce::IndexMax, &dims);
  ASSERT_TRUE(z.equalsTo(&exp4));

  z = x.applyIndexReduce(indexreduce::IndexMax, &dims0);
  ASSERT_TRUE(z.equalsTo(&exp5));

  z = x.applyIndexReduce(indexreduce::IndexMax, &dims1);
  ASSERT_TRUE(z.equalsTo(&exp6));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_float_test1) {
  NDArray x('c', {2, 3, 2},
            {
                1,
                2,
                3,
                4,
                5,
                6,
                7,
                8,
                -1,
                -2,
                -3,
                -4,
            },
            INT32);

  NDArray z1('c', {}, std::vector<double>{100}, DOUBLE);
  NDArray z2('c', {2, 2}, {100, 100, 100, 100}, FLOAT32);
  NDArray z3('c', {3}, {100, 100, 100}, DOUBLE);
  NDArray z4('c', {3, 2}, {100, 100, 100, 100, 100, 100}, FLOAT32);
  NDArray z5('c', {2}, {100, 100}, FLOAT32);

  NDArray exp1('c', {}, std::vector<double>{2.166667}, DOUBLE);
  NDArray exp2('c', {2, 2}, {3.f, 4.f, 1.f, 0.666667f}, FLOAT32);
  NDArray exp3('c', {3}, {4.5, 1, 1}, DOUBLE);
  NDArray exp4('c', {3, 2}, {4, 5, 1, 1, 1, 1}, FLOAT32);
  NDArray exp5('c', {2}, {3.5f, 0.833333f}, FLOAT32);


  std::vector<LongType> dims = {0, 1, 2};
  std::vector<LongType> dims1 = {1};
  std::vector<LongType> dims02 = {0,2};
  x.reduceAlongDimension(reduce::Mean, z1, &dims);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  x.reduceAlongDimension(reduce::Mean, z2, &dims1);
  ASSERT_TRUE(z2.equalsTo(&exp2));

  x.reduceAlongDimension(reduce::Mean, z3, &dims02);
  ASSERT_TRUE(z3.equalsTo(&exp3));

  x.permutei({1, 0, 2});  // 3x2x2

  x.reduceAlongDimension(reduce::Mean, z1, &dims);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  x.reduceAlongDimension(reduce::Mean, z4, &dims1);
  ASSERT_TRUE(z4.equalsTo(&exp4));

  x.reduceAlongDimension(reduce::Mean, z5, &dims02);
  ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_float_test2) {
  NDArray x('c', {2, 3, 2},
            {
                1,
                2,
                3,
                4,
                5,
                6,
                7,
                8,
                -1,
                -2,
                -3,
                -4,
            },
            DOUBLE);

  NDArray exp1('c', {}, std::vector<double>{2.166667}, DOUBLE);
  NDArray exp2('c', {2, 2}, {3, 4, 1, 0.666667}, DOUBLE);
  NDArray exp3('c', {3}, {4.5, 1, 1}, DOUBLE);
  NDArray exp4('c', {3, 2}, {4, 5, 1, 1, 1, 1}, DOUBLE);
  NDArray exp5('c', {2}, {3.5, 0.833333}, DOUBLE);

  std::vector<LongType> dims = {0, 1, 2};
  std::vector<LongType> dims1 = {1};
  std::vector<LongType> dims02 = {0,2};

  NDArray z1 = x.reduceAlongDimension(reduce::Mean, &dims);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  NDArray z2 = x.reduceAlongDimension(reduce::Mean, &dims1);
  ASSERT_TRUE(z2.equalsTo(&exp2));

  NDArray z3 = x.reduceAlongDimension(reduce::Mean,&dims02);
  ASSERT_TRUE(z3.equalsTo(&exp3));

  x.permutei({1, 0, 2});  // 3x2x2

  NDArray z4 = x.reduceAlongDimension(reduce::Mean, &dims);
  ASSERT_TRUE(z4.equalsTo(&exp1));

  NDArray z5 = x.reduceAlongDimension(reduce::Mean,&dims1);
  ASSERT_TRUE(z5.equalsTo(&exp4));

  NDArray z6 = x.reduceAlongDimension(reduce::Mean, &dims02);
  ASSERT_TRUE(z6.equalsTo(&exp5));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, EqualityTest1) {
  auto arrayA = NDArrayFactory::create_<float>('f', {3, 5});
  auto arrayB = NDArrayFactory::create_<float>('f', {3, 5});
  auto arrayC = NDArrayFactory::create_<float>('f', {3, 5});

  auto arrayD = NDArrayFactory::create_<float>('f', {2, 4});
  auto arrayE = NDArrayFactory::create_<float>('f', {1, 15});

  for (int i = 0; i < arrayA->rows(); i++) {
    for (int k = 0; k < arrayA->columns(); k++) {
      arrayA->p(i, k, (float)i);
    }
  }

  for (int i = 0; i < arrayB->rows(); i++) {
    for (int k = 0; k < arrayB->columns(); k++) {
      arrayB->p(i, k, (float)i);
    }
  }

  for (int i = 0; i < arrayC->rows(); i++) {
    for (int k = 0; k < arrayC->columns(); k++) {
      arrayC->p(i, k, (float)i + 1);
    }
  }

  ASSERT_TRUE(arrayA->equalsTo(arrayB, 1e-5));

  ASSERT_FALSE(arrayC->equalsTo(arrayB, 1e-5));

  ASSERT_FALSE(arrayD->equalsTo(arrayB, 1e-5));

  ASSERT_FALSE(arrayE->equalsTo(arrayB, 1e-5));

  delete arrayA;
  delete arrayB;
  delete arrayC;
  delete arrayD;
  delete arrayE;
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_same_test1) {
  NDArray x('c', {2, 3, 2}, {1.5f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.5f, 8.f, -1.f, -2.f, -3.5f, -4.f}, FLOAT32);

  NDArray z1('c', {}, std::vector<double>{100}, FLOAT32);
  NDArray z2('c', {2, 2}, {100, 100, 100, 100}, FLOAT32);
  NDArray z3('c', {3}, {100, 100, 100}, FLOAT32);
  NDArray z4('c', {3, 2}, {100, 100, 100, 100, 100, 100}, FLOAT32);
  NDArray z5('c', {2}, {100, 100}, FLOAT32);

  NDArray exp1('c', {}, std::vector<double>{26.5f}, FLOAT32);
  NDArray exp2('c', {2, 2}, {9.5f, 12.f, 3.f, 2.f}, FLOAT32);
  NDArray exp3('c', {3}, {19.f, 4.f, 3.5f}, FLOAT32);
  NDArray exp4('c', {3, 2}, {9.f, 10.f, 2.f, 2.f, 1.5f, 2.f}, FLOAT32);
  NDArray exp5('c', {2}, {21.5f, 5.f}, FLOAT32);

  std::vector<LongType> dims = {0, 1, 2};
  std::vector<LongType> dims1 = {1};
  std::vector<LongType> dims02 = {0,2};


  x.reduceAlongDimension(reduce::Sum, z1, &dims);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  x.reduceAlongDimension(reduce::Sum, z2, &dims1);
  ASSERT_TRUE(z2.equalsTo(&exp2));

  x.reduceAlongDimension(reduce::Sum, z3, &dims02);
  ASSERT_TRUE(z3.equalsTo(&exp3));

  x.permutei({1, 0, 2});  // 3x2x2

  x.reduceAlongDimension(reduce::Sum, z1, &dims);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  x.reduceAlongDimension(reduce::Sum, z4, &dims1);
  ASSERT_TRUE(z4.equalsTo(&exp4));

  x.reduceAlongDimension(reduce::Sum, z5, &dims02);
  ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_same_test2) {
  NDArray x('c', {2, 3, 2},
            {
                1.5,
                2,
                3,
                4,
                5,
                6,
                7.5,
                8,
                -1,
                -2,
                -3.5,
                -4,
            },
            INT64);

  NDArray exp1('c', {}, std::vector<double>{26}, INT64);
  NDArray exp2('c', {2, 2}, {9, 12, 3, 2}, INT64);
  NDArray exp3('c', {3}, {18, 4, 4}, INT64);
  NDArray exp4('c', {3, 2}, {8, 10, 2, 2, 2, 2}, INT64);
  NDArray exp5('c', {2}, {21, 5}, INT64);

  std::vector<LongType> dims = {0, 1, 2};
  std::vector<LongType> dims1 = {1};
  std::vector<LongType> dims02 = {0,2};

  NDArray z1 = x.reduceAlongDimension(reduce::Sum, &dims);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  NDArray z2 = x.reduceAlongDimension(reduce::Sum, &dims1);
  ASSERT_TRUE(z2.equalsTo(&exp2));

  NDArray z3 = x.reduceAlongDimension(reduce::Sum, &dims02);
  ASSERT_TRUE(z3.equalsTo(&exp3));

  x.permutei({1, 0, 2});  // 3x2x2

  NDArray z4 = x.reduceAlongDimension(reduce::Sum, &dims);
  ASSERT_TRUE(z4.equalsTo(&exp1));

  NDArray z5 = x.reduceAlongDimension(reduce::Sum, &dims1);
  ASSERT_TRUE(z5.equalsTo(&exp4));

  NDArray z6 = x.reduceAlongDimension(reduce::Sum,&dims02);
  ASSERT_TRUE(z6.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_bool_test1) {
  NDArray x('c', {2, 3, 2}, {0.5, 2, 3, -4, 5, 6, -7.5, 8, -1, -0.5, -3.5, 4}, DOUBLE);

  NDArray z1('c', {}, std::vector<double>{true}, BOOL);
  NDArray z2('c', {2, 2}, {true, true, true, true}, BOOL);
  NDArray z3('c', {3}, {true, true, true}, BOOL);
  NDArray z4('c', {3, 2}, {true, true, true, true, true, true}, BOOL);
  NDArray z5('c', {2}, {true, true}, BOOL);

  NDArray exp1('c', {}, std::vector<double>{true}, BOOL);
  NDArray exp2('c', {2, 2}, {true, true, false, true}, BOOL);
  NDArray exp3('c', {3}, {true, true, true}, BOOL);
  NDArray exp4('c', {3, 2}, {true, true, true, false, true, true}, BOOL);
  NDArray exp5('c', {2}, {true, true}, BOOL);

  std::vector<LongType> dims = {0, 1, 2};
  std::vector<LongType> dims1 = {1};
  std::vector<LongType> dims02 = {0,2};

  x.reduceAlongDimension(reduce::IsPositive, z1, &dims);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  x.reduceAlongDimension(reduce::IsPositive, z2, &dims1);
  ASSERT_TRUE(z2.equalsTo(&exp2));

  x.reduceAlongDimension(reduce::IsPositive, z3, &dims02);
  ASSERT_TRUE(z3.equalsTo(&exp3));

  x.permutei({1, 0, 2});  // 3x2x2

  x.reduceAlongDimension(reduce::IsPositive, z1, &dims);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  x.reduceAlongDimension(reduce::IsPositive, z4, &dims1);
  ASSERT_TRUE(z4.equalsTo(&exp4));

  x.reduceAlongDimension(reduce::IsPositive, z5,&dims02);
  ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_bool_test2) {
  NDArray x('c', {2, 3, 2}, {0.5, 2, 3, -4, 5, 6, -7.5, 8, -1, -0.5, -3.5, 4}, INT32);

  NDArray exp1('c', {}, std::vector<double>{1}, BOOL);
  NDArray exp2('c', {2, 2}, {1, 1, 0, 1}, BOOL);
  NDArray exp3('c', {3}, {1, 1, 1}, BOOL);
  NDArray exp4('c', {3, 2}, {0, 1, 1, 0, 1, 1}, BOOL);
  NDArray exp5('c', {2}, {1, 1}, BOOL);

  std::vector<LongType> dims = {0, 1, 2};
  std::vector<LongType> dims1 = {1};
  std::vector<LongType> dims02 = {0,2};

  NDArray z1 = x.reduceAlongDimension(reduce::IsPositive, &dims);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  NDArray z2 = x.reduceAlongDimension(reduce::IsPositive, &dims1);
  ASSERT_TRUE(z2.equalsTo(&exp2));

  NDArray z3 = x.reduceAlongDimension(reduce::IsPositive, &dims02);
  ASSERT_TRUE(z3.equalsTo(&exp3));

  x.permutei({1, 0, 2});  // 3x2x2

  NDArray z4 = x.reduceAlongDimension(reduce::IsPositive,&dims);
  ASSERT_TRUE(z4.equalsTo(&exp1));

  NDArray z5 = x.reduceAlongDimension(reduce::IsPositive, &dims1);
  ASSERT_TRUE(z5.equalsTo(&exp4));

  NDArray z6 = x.reduceAlongDimension(reduce::IsPositive, &dims02);
  ASSERT_TRUE(z6.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_long_test1) {
  NDArray x('c', {2, 3, 2}, {0.5f, 2.f, 3.f, -0.f, 5.f, 6.f, -7.5f, 0.f, -1.f, -0.5f, -3.5f, 4.f}, FLOAT32);

  NDArray z1('c', {}, std::vector<double>{100}, INT64);
  NDArray z2('c', {2, 2}, {100, 100, 100, 100}, INT64);
  NDArray z3('c', {3}, {100, 100, 100}, INT64);
  NDArray z4('c', {3, 2}, {100, 100, 100, 100, 100, 100}, INT64);
  NDArray z5('c', {2}, {100, 100}, INT64);

  NDArray exp1('c', {}, std::vector<double>{2}, INT64);
  NDArray exp2('c', {2, 2}, {0, 1, 0, 1}, INT64);
  NDArray exp3('c', {3}, {1, 1, 0}, INT64);
  NDArray exp4('c', {3, 2}, {0, 1, 0, 1, 0, 0}, INT64);
  NDArray exp5('c', {2}, {1, 1}, INT64);

  std::vector<LongType> dims = {0, 1, 2};
  std::vector<LongType> dims1 = {1};
  std::vector<LongType> dims02 = {0,2};

  x.reduceAlongDimension(reduce::CountZero, z1,&dims);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  x.reduceAlongDimension(reduce::CountZero, z2,&dims1);
  ASSERT_TRUE(z2.equalsTo(&exp2));

  x.reduceAlongDimension(reduce::CountZero, z3, &dims02);
  ASSERT_TRUE(z3.equalsTo(&exp3));

  x.permutei({1, 0, 2});  // 3x2x2

  x.reduceAlongDimension(reduce::CountZero, z1,&dims);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  x.reduceAlongDimension(reduce::CountZero, z4, &dims1);
  ASSERT_TRUE(z4.equalsTo(&exp4));

  x.reduceAlongDimension(reduce::CountZero, z5,&dims02);
  ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_long_test2) {
  NDArray x('c', {2, 3, 2}, {0.5, 2, 3, -0, 5, 6, -7.5, 0, -1, -0.5, -3.5, 4}, INT32);

  NDArray exp1('c', {}, std::vector<double>{4}, INT64);
  NDArray exp2('c', {2, 2}, {1, 1, 0, 2}, INT64);
  NDArray exp3('c', {3}, {2, 2, 0}, INT64);
  NDArray exp4('c', {3, 2}, {1, 1, 0, 2, 0, 0}, INT64);
  NDArray exp5('c', {2}, {2, 2}, INT64);

  std::vector<LongType> dims = {0, 1, 2};
  std::vector<LongType> dims1 = {1};
  std::vector<LongType> dims02 = {0,2};

  NDArray z1 = x.reduceAlongDimension(reduce::CountZero, &dims);
  ASSERT_EQ(z1,exp1);
  ASSERT_TRUE(z1.equalsTo(&exp1));

  NDArray z2 = x.reduceAlongDimension(reduce::CountZero, &dims1);
  ASSERT_EQ(z2,exp2);

  NDArray z3 = x.reduceAlongDimension(reduce::CountZero, &dims02);
  ASSERT_EQ(exp3,z3);

  x.permutei({1, 0, 2});  // 3x2x2

  NDArray z4 = x.reduceAlongDimension(reduce::CountZero, &dims);
  ASSERT_EQ(z4,exp1);

  NDArray z5 = x.reduceAlongDimension(reduce::CountZero, &dims1);
  ASSERT_EQ(exp4,z5);

  NDArray z6 = x.reduceAlongDimension(reduce::CountZero, &dims02);
  ASSERT_EQ(exp5,z6);
}

TEST_F(NDArrayCudaBasicsTests, BroadcastOpsTest1) {
  auto x = NDArrayFactory::create<float>('c', {5, 5});
  auto z = NDArrayFactory::create<float>('c', {5, 5});
  auto row = NDArrayFactory::linspace(1.0f, 5.0f, 5);
  NDArray expRow('c',
                 {

                     5,
                 },
                 {1, 2, 3, 4, 5}, FLOAT32);
  NDArray exp('c', {5, 5}, {1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5}, FLOAT32);

  ASSERT_EQ(expRow,*row);
  \
  std::vector<LongType> dims = {0, 1, 2};
  std::vector<LongType> dims1 = {1};

  x.applyBroadcast(broadcast::Add, &dims1, *row, z);
  x += *row;

  ASSERT_EQ(x,z);

  delete row;
}

TEST_F(NDArrayCudaBasicsTests, BroadcastOpsTest2) {
  auto x = NDArrayFactory::create<float>('c', {5, 5});
  auto row = NDArrayFactory::linspace(1.0f, 5.0f, 5);
  NDArray expRow('c',
                 {

                     5,
                 },
                 {1, 2, 3, 4, 5}, FLOAT32);
  NDArray exp('c', {5, 5}, {1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5}, FLOAT32);

  std::vector<LongType> dims1 = {1};

  ASSERT_EQ(expRow,*row);
  x.applyBroadcast(broadcast::Add, &dims1, *row, x);
  ASSERT_EQ(exp,x);
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestBroadcast_1) {
  NDArray exp('c', {2, 3, 2, 2},
              {1., 1., 1., 1., 2., 2., 2., 2., 3., 3., 3., 3., 1., 1., 1., 1., 2., 2., 2., 2., 3., 3., 3., 3.}, DOUBLE);

  auto input = NDArrayFactory::create<double>('c', {2, 3, 2, 2});
  auto bias = NDArrayFactory::create<double>('c', {1, 3});
  std::vector<LongType> dims1 = {1};
  bias.linspace(1);
  input.applyBroadcast(broadcast::Add,&dims1, bias, input);
  ASSERT_EQ(exp,input);
}

TEST_F(NDArrayCudaBasicsTests, TestFloat16_1) {
  auto x = NDArrayFactory::create<float>({1, 2, 3, 4, 5, 7, 8, 9});
  auto y = NDArrayFactory::create<float>({1, 2, 3, 4, 5, 7, 8, 9});
  ASSERT_EQ(x,y);
}

TEST_F(NDArrayCudaBasicsTests, TestFloat16_2) {
  auto x = NDArrayFactory::create<float16>('c', {9}, {1, 2, 3, 4, 5, 6, 7, 8, 9});
  auto y = NDArrayFactory::create<float16>('c', {9}, {1, 2, 3, 4, 5, 6, 7, 8, 9});
  ASSERT_EQ(x,y);
}

TEST_F(NDArrayCudaBasicsTests, TestFloat16_3) {
  auto x = NDArrayFactory::create<bfloat16>({1, 2, 3, 4, 5, 7, 8, 9});
  auto y = NDArrayFactory::create<bfloat16>({1, 2, 3, 4, 5, 7, 8, 9});
  ASSERT_EQ(x,y);
}

TEST_F(NDArrayCudaBasicsTests, TestFloat_4) {
  auto x = NDArrayFactory::create<float>({1, 2, 3, 4, 5, 7, 8, 9});
  auto y = NDArrayFactory::create<float>({2, 4, 5, 5, 6, 7, 8, 9});
  ASSERT_FALSE(x.equalsTo(&y));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat_5) {
  auto x = NDArrayFactory::create<float>('c', {3, 3}, {1, 2, 3, 4, 5, 6, 7, 8, 9});
  auto y = NDArrayFactory::create<float>('c', {3, 3}, {2, 4, 5, 5, 6, 7, 8, 9, 10});
  ASSERT_FALSE(x.equalsTo(&y));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat_6) {
  auto x = NDArrayFactory::create<float>('f', {3, 3}, {1, 2, 3, 4, 5, 6, 7, 8, 9});
  auto y = NDArrayFactory::create<float>('f', {3, 3}, {2, 4, 5, 5, 6, 7, 8, 9, 10});
  ASSERT_FALSE(x.equalsTo(&y));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_05) {
  auto x = NDArrayFactory::create<float>('c', {8, 8, 8});
  auto y = NDArrayFactory::create<float>('c', {1, 8, 8});
  auto expected = NDArrayFactory::create<float>('c', {8, 8, 8});
  NDArray res2 = NDArrayFactory::create<float>(expected.ordering(), expected.getShapeAsVector());
  x = 1.;
  y = 2.;
  expected = 3.;
  res2 = 0.f;

  x.applyTrueBroadcast(BroadcastOpsTuple::Add(), y, res2);  // *= y;

  ASSERT_EQ(expected,res2);

}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_5) {
  auto x = NDArrayFactory::create<float>('c', {8, 8, 8});
  auto y = NDArrayFactory::create<float>('c', {8, 1, 8});
  auto expected = NDArrayFactory::create<float>('c', {8, 8, 8});
  NDArray res2(expected);
  x = 1.;
  y = 2.;
  expected = 3.;

  auto result = x + y;

  ASSERT_EQ(expected,result);
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_51) {
  auto x = NDArrayFactory::create<float>('c', {8, 8, 8});
  auto y = NDArrayFactory::create<float>('c', {8, 8});
  auto expected = NDArrayFactory::create<float>('c', {8, 8, 8});
  NDArray res2(expected);
  x = 1.;
  y = 2.;
  expected = 3.;
  auto result = x + y;

  ASSERT_EQ(expected,result);
}

TEST_F(NDArrayCudaBasicsTests, Tile_Test_2_1) {
  auto x = NDArrayFactory::create<float>('c', {2, 1, 2});
  x = 10.;
  auto y = x.tile({1, 2, 1});
  auto exp = NDArrayFactory::create<float>('c', {2, 2, 2});
  exp = 10.;

  ASSERT_EQ(exp,y);
}

TEST_F(NDArrayCudaBasicsTests, Tile_Test_2_2) {
  auto x = NDArrayFactory::create<float>('f', {2, 1, 2});
  x = 10.;
  auto y = x.tile({1, 2, 1});
  auto exp = NDArrayFactory::create<float>('f', {2, 2, 2});
  exp = 10.;
  ASSERT_EQ(exp,y);
}

TEST_F(NDArrayCudaBasicsTests, Tile_Test_2_3) {
  auto x = NDArrayFactory::create<float>('f', {2, 1, 2});
  x = 10.;
  x.p(1, 0, 1, 20);
  x.syncToDevice();
  auto y = x.tile({1, 2, 1});
  auto exp = NDArrayFactory::create<float>('f', {2, 2, 2});
  exp = 10.;
  exp.p(1, 0, 1, 20.);
  exp.p(1, 1, 1, 20.);
  exp.syncToDevice();
  ASSERT_EQ(exp,y);
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_2) {
  double expBuff[] = {2., 3, 3., 4., 4., 5, 5., 6., 6., 7, 7., 8.};
  NDArray a('c', {4, 4}, {1, 2, 3, 4, 5, 6, 7, 8, 9, 2, 3, 2, 1, 0, 4, 7}, FLOAT32);
  auto x = NDArrayFactory::create<double>('c', {3, 2, 1});
  auto y = NDArrayFactory::create<double>('c', {1, 2});
  auto expected = NDArrayFactory::create<double>(expBuff, 'c', {3, 2, 2});

  x.linspace(1);
  y.linspace(1);
  auto result = x + y;
  ASSERT_EQ(expected,result);

}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, assign_2) {
  NDArray x('c', {4}, {1.5f, 2.5f, 3.5f, 4.5f}, FLOAT32);
  NDArray y('c', {4}, INT32);
  NDArray expected('c', {4}, {1, 2, 3, 4}, INT32);

  y.assign(x);

  ASSERT_EQ(expected,y);
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, subarray_1) {
  NDArray x('c', {2, 3, 4}, {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24},
            FLOAT32);
  NDArray y('f', {2, 3, 4}, {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24},
            FLOAT32);

  LongType shapeExpX0[] = {1, 2, 12, 8192, 1, 99};
  float buffExpX0[] = {1.f, 13.f};
  LongType shapeExpX1[] = {1, 2, 12, 8192, 1, 99};
  float buffExpX1[] = {2.f, 14.f};
  LongType shapeExpX2[] = {3, 2, 1, 1, 12, 4, 1, 8192, 1, 99};
  float buffExpX2[] = {1.f, 13.f};
  LongType shapeExpX3[] = {2, 2, 4, 12, 1, 8192, 1, 99};
  float buffExpX3[] = {9.f, 10.f, 11.f, 12.f, 21.f, 22.f, 23.f, 24.f};
  LongType shapeExpX4[] = {3, 2, 1, 4, 12, 4, 1, 8192, 1, 99};
  float buffExpX4[] = {9.f, 10.f, 11.f, 12.f, 21.f, 22.f, 23.f, 24.f};
  LongType shapeExpX5[] = {2, 2, 3, 12, 4, 8192, 1, 99};
  float buffExpX5[] = {4.f, 8.f, 12.f, 16.f, 20.f, 24.f};

  LongType shapeExpY0[] = {1, 2, 1, 8192, 1, 99};
  float buffExpY0[] = {1.f, 2.f};
  LongType shapeExpY1[] = {1, 2, 1, 8192, 1, 99};
  float buffExpY1[] = {7.f, 8.f};
  LongType shapeExpY2[] = {3, 2, 1, 1, 1, 2, 6, 8192, 1, 102};
  float buffExpY2[] = {1.f, 2.f};
  LongType shapeExpY3[] = {2, 2, 4, 1, 6, 8192, 1, 99};
  float buffExpY3[] = {5.f, 11.f, 17.f, 23.f, 6.f, 12.f, 18.f, 24.f};
  LongType shapeExpY4[] = {3, 2, 1, 4, 1, 2, 6, 8192, 1, 102};
  float buffExpY4[] = {5.f, 11.f, 17.f, 23.f, 6.f, 12.f, 18.f, 24.f};
  LongType shapeExpY5[] = {2, 2, 3, 1, 2, 8192, 1, 99};
  float buffExpY5[] = {19.f, 21.f, 23.f, 20.f, 22.f, 24.f};

  NDArray x0 = x(0, {1, 2});
  NDArray xExp(buffExpX0, shapeExpX0);

  ASSERT_EQ(xExp,x0);

  NDArray x1 = x(1, {1, 2});
  NDArray x1Exp(buffExpX1, shapeExpX1);
  ASSERT_EQ(x1Exp,x1);



  NDArray x2 = x(0, {1, 2}, true);
  NDArray x2Exp(buffExpX2, shapeExpX2);
  ASSERT_EQ(x2Exp,x2);


  NDArray x3 = x(2, {1});
  NDArray x3Exp(buffExpX3, shapeExpX3);
  ASSERT_EQ(x3Exp,x3);

  NDArray x4 = x(2, {1}, true);
  NDArray x4Exp(buffExpX4, shapeExpX4);
  ASSERT_EQ(x4Exp,x4);

  NDArray x5 = x(3, {2});
  NDArray x5Exp(buffExpX5, shapeExpX5);
  ASSERT_EQ(x5Exp,x5);

  // ******************* //
  NDArray y0 = y(0, {1, 2});
  NDArray y0Exp(buffExpY0, shapeExpY0);

  ASSERT_EQ(y0Exp,y0);


  NDArray y1 = y(1, {1, 2});
  NDArray y1Exp(buffExpY1, shapeExpY1);

  ASSERT_EQ(y1Exp,y1);

  NDArray y2 = y(0, {1, 2}, true);
  NDArray y2Exp(buffExpY2, shapeExpY2);
  ASSERT_EQ(y2Exp,y2);

  NDArray y3 = y(2, {1});
  NDArray y3Exp(buffExpY3, shapeExpY3);

  ASSERT_EQ(y3Exp,y3);

  NDArray y4 = y(2, {1}, true);
  NDArray y4Exp = NDArrayFactory::create<float>('f', {2, 1, 4}, {5, 6, 11, 12, 17, 18, 23, 24});

  ASSERT_EQ(y4Exp,y4);


  NDArray y5 = y(3, {2});
  NDArray y5Exp(buffExpY5, shapeExpY5);
  ASSERT_EQ(y5Exp,y5);


}
//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Test_diagonal_1) {
  auto x = NDArrayFactory::create<float>('c', {2, 3}, {1, 2, 3, 4, 5, 6});
  auto exp = NDArrayFactory::create<float>('c', {2, 1}, {1, 5});

  auto diag = x.diagonal('c');
  for (LongType e = 0; e < exp.lengthOf(); ++e) {
    printf("VAL[%ld] = %f\n", e, diag.e<float>(e));
  }

  for (LongType e = 0; e < exp.lengthOf(); ++e) {
    ASSERT_NEAR(diag.e<float>(e), exp.e<float>(e), 1.e-5);
  }
  double eps(1.e-5);
  NDArray tmp(FLOAT32, x.getContext());  // scalar = 0

  ExtraArguments extras({eps,eps,eps});
  NativeOpExecutioner::execReduce3Scalar(diag.getContext(), reduce3::EqualsWithEps, diag.buffer(), diag.shapeInfo(),
                                         diag.specialBuffer(), diag.specialShapeInfo(),
                                         extras.argumentsAsT(FLOAT32), exp.buffer(), exp.shapeInfo(),
                                         exp.specialBuffer(), exp.specialShapeInfo(), tmp.buffer(), tmp.shapeInfo(),
                                         tmp.specialBuffer(), tmp.specialShapeInfo());
  hipStream_t stream = x.getContext()->getCudaStream();
  auto res = hipStreamSynchronize(stream);
  ASSERT_TRUE(exp.isSameShape(diag));
  ASSERT_TRUE(exp.equalsTo(diag));
}

TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_02) {
  auto x = NDArrayFactory::linspace<float>(1.f, 60.f, 60);
  auto exp = NDArrayFactory::create<float>(
      'c', {3, 4, 5},
      {1.0f,  2.0f,  3.0f,  4.0f,  5.0f,  6.0f,  7.0f,  8.0f,  9.0f,  10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f,
       16.0f, 17.0f, 18.0f, 19.0f, 20.0f, 21.0f, 22.0f, 23.0f, 24.0f, 25.0f, 26.0f, 27.0f, 28.0f, 29.0f, 30.0f,
       31.0f, 32.0f, 33.0f, 34.0f, 35.0f, 36.0f, 37.0f, 38.0f, 39.0f, 40.0f, 41.0f, 42.0f, 43.0f, 44.0f, 45.0f,
       46.0f, 47.0f, 48.0f, 49.0f, 50.0f, 51.0f, 52.0f, 53.0f, 54.0f, 55.0f, 56.0f, 57.0f, 58.0f, 59.0f, 60.0});
  x->reshapei('c', {3, 4, 5});

  x->permutei({0, 1, 2});
  x->streamline();

  ASSERT_TRUE(exp.isSameShape(x));
  ASSERT_TRUE(exp.equalsTo(x));
  delete x;
}

TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_0) {
  auto x = NDArrayFactory::create<float>('c', {1, 60});
  x.linspace(1);
  auto exp = NDArrayFactory::create<float>(
      'c', {3, 4, 5},
      {1.0f,  2.0f,  3.0f,  4.0f,  5.0f,  6.0f,  7.0f,  8.0f,  9.0f,  10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f,
       16.0f, 17.0f, 18.0f, 19.0f, 20.0f, 21.0f, 22.0f, 23.0f, 24.0f, 25.0f, 26.0f, 27.0f, 28.0f, 29.0f, 30.0f,
       31.0f, 32.0f, 33.0f, 34.0f, 35.0f, 36.0f, 37.0f, 38.0f, 39.0f, 40.0f, 41.0f, 42.0f, 43.0f, 44.0f, 45.0f,
       46.0f, 47.0f, 48.0f, 49.0f, 50.0f, 51.0f, 52.0f, 53.0f, 54.0f, 55.0f, 56.0f, 57.0f, 58.0f, 59.0f, 60.0});
  x.reshapei('c', {3, 4, 5});

  x.permutei({0, 1, 2});
  x.streamline();
  ASSERT_TRUE(exp.isSameShape(&x));
  ASSERT_TRUE(exp.equalsTo(&x));
}
TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_1) {
  auto x = NDArrayFactory::create<float>('c', {1, 60});
  x.linspace(1);
  auto exp = NDArrayFactory::create<float>(
      'c', {3, 4, 5},
      {1.0f,  2.0f,  3.0f,  4.0f,  5.0f,  6.0f,  7.0f,  8.0f,  9.0f,  10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f,
       16.0f, 17.0f, 18.0f, 19.0f, 20.0f, 21.0f, 22.0f, 23.0f, 24.0f, 25.0f, 26.0f, 27.0f, 28.0f, 29.0f, 30.0f,
       31.0f, 32.0f, 33.0f, 34.0f, 35.0f, 36.0f, 37.0f, 38.0f, 39.0f, 40.0f, 41.0f, 42.0f, 43.0f, 44.0f, 45.0f,
       46.0f, 47.0f, 48.0f, 49.0f, 50.0f, 51.0f, 52.0f, 53.0f, 54.0f, 55.0f, 56.0f, 57.0f, 58.0f, 59.0f, 60.0});
  x.reshapei('c', {3, 4, 5});

  x.permutei({0, 1, 2});
  x.streamline();

  ASSERT_TRUE(exp.isSameShape(&x));
  ASSERT_TRUE(exp.equalsTo(&x));
}
TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_2) {
  auto xx = NDArrayFactory::linspace<float>(1.f, 60.f, 60);
  delete xx;
}
TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_3) {
  auto x = NDArrayFactory::create<float>('c', {1, 60});
  for (int l = 0; l < x.lengthOf(); l++) x.p(l, float(l + 1.f));
  auto exp = NDArrayFactory::create<float>(
      'c', {3, 4, 5},
      {1.0f,  2.0f,  3.0f,  4.0f,  5.0f,  6.0f,  7.0f,  8.0f,  9.0f,  10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f,
       16.0f, 17.0f, 18.0f, 19.0f, 20.0f, 21.0f, 22.0f, 23.0f, 24.0f, 25.0f, 26.0f, 27.0f, 28.0f, 29.0f, 30.0f,
       31.0f, 32.0f, 33.0f, 34.0f, 35.0f, 36.0f, 37.0f, 38.0f, 39.0f, 40.0f, 41.0f, 42.0f, 43.0f, 44.0f, 45.0f,
       46.0f, 47.0f, 48.0f, 49.0f, 50.0f, 51.0f, 52.0f, 53.0f, 54.0f, 55.0f, 56.0f, 57.0f, 58.0f, 59.0f, 60.0});
  x.reshapei('c', {3, 4, 5});

  x.permutei({0, 1, 2});
  x.streamline();
  ASSERT_TRUE(exp.isSameShape(&x));
  ASSERT_TRUE(exp.equalsTo(&x));
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_1) {
  auto x = NDArrayFactory::empty<float>();
  ASSERT_TRUE(x.isActualOnHostSide());
  ASSERT_TRUE(x.isEmpty());
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_2) {
  auto x = NDArrayFactory::empty_<float>();

  ASSERT_TRUE(x->isEmpty());
  delete x;
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_3) {
  auto x = NDArrayFactory::empty(FLOAT32);

  ASSERT_TRUE(x.isEmpty());
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_4) {
  auto x = NDArrayFactory::empty_(FLOAT32);

  ASSERT_TRUE(x->isEmpty());
  delete x;
}
