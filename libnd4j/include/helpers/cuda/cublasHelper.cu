/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <hipblas.h>
#include <hipsolver.h>
#include <exceptions/cuda_exception.h>
#include <execution/AffinityManager.h>
#include <helpers/logger.h>

#include "../cublasHelper.h"
#include "config.h"

#ifdef HAVE_CUDNN
#include <hipDNN.h>

#endif

namespace sd {
std::mutex CublasHelper::_mutex;

static void* handle_() {
  auto _handle = new hipblasHandle_t();
  auto status = hipblasCreate(_handle);  // initialize CUBLAS context
  if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("cuBLAS handle creation failed !", status);

  return reinterpret_cast<void*>(_handle);
}

static void* solver_() {
  auto cusolverH = new hipsolverHandle_t();
  auto status = hipsolverDnCreate(cusolverH);
  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("cuSolver handle creation failed !", status);

  return cusolverH;
}

static void* cudnn_() {
#ifdef HAVE_CUDNN
  auto cudnnH = new hipdnnHandle_t();
  auto status = hipdnnCreate(cudnnH);
  if (status != HIPDNN_STATUS_SUCCESS) throw cuda_exception::build("cuDNN handle creation failed !", status);

  return cudnnH;
#endif
  return nullptr;
}

static void destroyHandle_(void* handle) {
  auto ch = reinterpret_cast<hipblasHandle_t*>(handle);
  auto status = hipblasDestroy(*ch);
  if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("cuBLAS handle destruction failed !", status);

  delete ch;
}

CublasHelper::CublasHelper() {
  auto numDevices = AffinityManager::numberOfDevices();
  auto currentDevice = AffinityManager::currentDeviceId();
  _cache.resize(numDevices);
  _solvers.resize(numDevices);
  _cudnn.resize(numDevices);
  for (int e = 0; e < numDevices; e++) {
    AffinityManager::setCurrentNativeDevice(e);

    _cache[e] = handle_();
    _solvers[e] = solver_();
    _cudnn[e] = cudnn_();
  }

  // don't forget to restore back original device
  AffinityManager::setCurrentNativeDevice(currentDevice);
}

CublasHelper::~CublasHelper() {
  auto numDevices = AffinityManager::numberOfDevices();

  for (int e = 0; e < numDevices; e++) destroyHandle_(_cache[e]);
}

CublasHelper& CublasHelper::getInstance() {
  static CublasHelper instance;
  return instance;
}

void* CublasHelper::cudnn() {
  auto deviceId = AffinityManager::currentDeviceId();
  if (deviceId < 0 || deviceId > _cudnn.size())
    throw cuda_exception::build("requested deviceId doesn't look valid", deviceId);

  return _cudnn[deviceId];
}

void* CublasHelper::handle() {
  auto deviceId = AffinityManager::currentDeviceId();
  return handle(deviceId);
}

void* CublasHelper::solver() {
  auto deviceId = AffinityManager::currentDeviceId();
  if (deviceId < 0 || deviceId > _solvers.size())
    throw cuda_exception::build("requested deviceId doesn't look valid", deviceId);

  return _solvers[deviceId];
}

void* CublasHelper::handle(int deviceId) {
  if (deviceId < 0 || deviceId > _cache.size())
    throw cuda_exception::build("requested deviceId doesn't look valid", deviceId);

  return _cache[deviceId];
}
}  // namespace sd
