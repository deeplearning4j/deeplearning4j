/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//
#include <array/DataTypeUtils.h>
#include <array/PrimaryPointerDeallocator.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <exceptions/cuda_exception.h>
#include <execution/AffinityManager.h>
#include <execution/LaunchContext.h>
#include <helpers/ConstantHelper.h>
#include <helpers/logger.h>
#include <helpers/shape.h>
#include <ops/specials.h>

#define CONSTANT_LIMIT 49152

__constant__ char deviceConstantMemory[CONSTANT_LIMIT];

namespace sd {
static void *getConstantSpace() {
  Pointer dConstAddr;
  auto dZ = hipGetSymbolAddress(reinterpret_cast<void **>(&dConstAddr), deviceConstantMemory);

  if (dZ != 0) throw cuda_exception::build("hipGetSymbolAddress(...) failed", dZ);

  return dConstAddr;
}

int ConstantHelper::getCurrentDevice() { return AffinityManager::currentDeviceId(); }

int ConstantHelper::getNumberOfDevices() { return AffinityManager::numberOfDevices(); }

ConstantHelper::ConstantHelper() {
  auto initialDevice = getCurrentDevice();

  auto numDevices = getNumberOfDevices();
  _devicePointers.resize(numDevices);
  _deviceOffsets.resize(numDevices);
  _cache.resize(numDevices);
  _counters.resize(numDevices);

  // filling all pointers
  for (int e = 0; e < numDevices; e++) {
    auto res = hipSetDevice(e);
    if (res != 0) throw cuda_exception::build("hipSetDevice failed", res);
    auto constant = getConstantSpace();

    SD_MAP_IMPL<ConstantDescriptor, ConstantHolder *> devCache;

    _devicePointers[e] = constant;
    _deviceOffsets[e] = 0;
    _cache[e] = devCache;
    _counters[e] = 0L;
  }

  //
  auto res = hipSetDevice(initialDevice);
  if (res != 0) throw cuda_exception::build("Final hipSetDevice failed", res);
}

ConstantHelper::~ConstantHelper() {
  for (const auto &v : _cache) {
    for (const auto &c : v) {
      delete c.second;
    }
  }
}

ConstantHelper &ConstantHelper::getInstance() {
  static ConstantHelper instance;
  return instance;
}

void *ConstantHelper::replicatePointer(void *src, size_t numBytes, memory::Workspace *workspace) {
  std::lock_guard<std::mutex> lock(_mutex);

  auto deviceId = getCurrentDevice();
  Pointer constantPtr = nullptr;
  LongType constantOffset = 0L;
  if (_devicePointers[deviceId] == 0) {
    auto constant = getConstantSpace();

    // filling default ptr, which will be 0 probably
    _devicePointers[deviceId] = constant;
    _deviceOffsets[deviceId] = 0;
    constantPtr = constant;
  } else {
    constantPtr = _devicePointers[deviceId];
    constantOffset = _deviceOffsets[deviceId];
  }

  int8_t *ptr = nullptr;
  ALLOCATE_SPECIAL(ptr, workspace, numBytes, int8_t);
  auto res = hipMemcpy(ptr, src, numBytes, hipMemcpyHostToDevice);
  if (res != 0) {
    std::string errorMessage = "hipMemcpy failed with error code " + std::to_string(res);
    auto lastError = hipGetLastError(); // get last error
    if (lastError != hipSuccess) {
      errorMessage += "; last error: " + std::string(hipGetErrorString(lastError));
    }

    THROW_EXCEPTION(errorMessage.c_str());

  }

  constantPtr = ptr;
  return reinterpret_cast<int8_t *>(constantPtr) + constantOffset;
}

ConstantDataBuffer *ConstantHelper::constantBuffer(const ConstantDescriptor &descriptor, DataType dataType) {
  const auto deviceId = getCurrentDevice();

  // all cache modifications are synchronous
  _mutexHolder.lock();

  if (_cache[deviceId].count(descriptor) == 0) {
    _cache[deviceId][descriptor] = new ConstantHolder();
  }
  auto holder = _cache[deviceId][descriptor];

  // release cache lock
  _mutexHolder.unlock();

  ConstantDataBuffer *result;

  // access to this holder instance is synchronous
  std::lock_guard<std::mutex> lock(*holder->mutex());

  if (holder->hasBuffer(dataType)) {
    result = holder->getConstantDataBuffer(dataType);
  } else {
    auto numBytes = descriptor.length() * DataTypeUtils::sizeOf(dataType);
    auto cbuff = std::make_shared<PointerWrapper>(new int8_t[numBytes], std::make_shared<PointerDeallocator>());
    _counters[deviceId] += numBytes;

    // create buffer with this dtype
    if (descriptor.isFloat()) {
      BUILD_DOUBLE_SELECTOR(
          sd::DataType::DOUBLE, dataType, sd::SpecialTypeConverter::convertGeneric,
          (nullptr, const_cast<double *>(descriptor.floatValues().data()), descriptor.length(), cbuff->pointer()),
          (sd::DataType::DOUBLE, double), SD_COMMON_TYPES);
    } else if (descriptor.isInteger()) {
      BUILD_DOUBLE_SELECTOR(sd::DataType::INT64, dataType, sd::SpecialTypeConverter::convertGeneric,
                            (nullptr, const_cast<sd::LongType *>(descriptor.integerValues().data()),
                                descriptor.length(), cbuff->pointer()),
                            (sd::DataType::INT64, sd::LongType), SD_COMMON_TYPES);
    }

    // we don't have deallocator here.
    // TODO: we probably want to make use deallocator here, if we're not using constant memory
    auto dbuff = std::make_shared<PointerWrapper>(
        replicatePointer(cbuff->pointer(), descriptor.length() * DataTypeUtils::sizeOf(dataType)));

    ConstantDataBuffer *dataBuffer = new ConstantDataBuffer(cbuff, dbuff, descriptor.length(), dataType);

    holder->addBuffer(*dataBuffer, dataType);
    result = holder->getConstantDataBuffer(dataType);
  }

  return result;
}

LongType ConstantHelper::getCachedAmount(int deviceId) {
  int numDevices = getNumberOfDevices();
  if (deviceId > numDevices || deviceId < 0)
    return 0L;
  else
    return _counters[deviceId];
}
}  // namespace sd
