/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 06.02.2019
// @author raver119@gmail.com
//
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/StringUtils.h>
#include <helpers/logger.h>
#include <memory/Workspace.h>

#include "helpers/DebugHelper.h"

namespace sd {

//////////////////////////////////////////////////////////////////////////
PointersManager::PointersManager(const LaunchContext* context, const std::string& funcName) {
  _context = const_cast<LaunchContext*>(context);
  _funcName = funcName;
}
//////////////////////////////////////////////////////////////////////////
void* PointersManager::allocateDevMem(const size_t sizeInBytes) {
  void* dst = nullptr;
  if (_context->getWorkspace() == nullptr) {
    hipError_t cudaResult = hipMalloc(reinterpret_cast<void**>(&dst), sizeInBytes);
    if (cudaResult != 0)
      throw cuda_exception::build(_funcName + ": cannot allocate global memory on device!", cudaResult);
  } else {
    dst = _context->getWorkspace()->allocateBytes(memory::MemoryType::DEVICE, sizeInBytes);
  }
  return dst;
}

//////////////////////////////////////////////////////////////////////////
void* PointersManager::replicatePointer(const void* src, const size_t numberOfBytes) {
  void* dst = allocateDevMem(numberOfBytes);
  if (src) {
    if (_context != nullptr)
      hipMemcpyAsync(dst, src, numberOfBytes, hipMemcpyHostToDevice, *_context->getCudaStream());
    else
      hipMemcpy(dst, src, numberOfBytes, hipMemcpyHostToDevice);
  }
  _pOnGlobMem.emplace_back(dst);

  return dst;
}

//////////////////////////////////////////////////////////////////////////
void PointersManager::synchronize() const {
  if (_context != nullptr) {
    hipError_t cudaResult = hipStreamSynchronize(*_context->getCudaStream());
    if (cudaResult != 0) throw cuda_exception::build(_funcName + ": cuda stream synchronization failed !", cudaResult);
  } else {
    sd_printf("<%s> syncStream isn't possible: no stream set!", _funcName.c_str());
  }
}

//////////////////////////////////////////////////////////////////////////
PointersManager::~PointersManager() {
 // for (auto& p : _pOnGlobMem) hipFree(p);
}

////////////////////////////////////////////////////////////////////////
template <typename T>
static SD_KERNEL void printDevContentOnDev_(const void* pDev, const LongType len, const int tid) {
  PointersManager::printDevContentOnDev<T>(pDev, len, tid);
}

////////////////////////////////////////////////////////////////////////
template <typename T>
void PointersManager::printDevContentOnDevFromHost(const void* pDev, const LongType len, const int tid) {
  printDevContentOnDev_<T><<<512, 512, 1024, *LaunchContext ::defaultContext()->getCudaStream()>>>(pDev, len, tid);
  auto res = hipStreamSynchronize(*LaunchContext ::defaultContext()->getCudaStream());
  DebugHelper::checkGlobalErrorCode("concat general case failed(...) failed");

}
template void PointersManager::printDevContentOnDevFromHost<LongType>(const void* pDev, const LongType len,
                                                                          const int tid);
template void PointersManager::printDevContentOnDevFromHost<int>(const void* pDev, const LongType len,
                                                                 const int tid);
template void PointersManager::printDevContentOnDevFromHost<float>(const void* pDev, const LongType len,
                                                                   const int tid);
template void PointersManager::printDevContentOnDevFromHost<double>(const void* pDev, const LongType len,
                                                                    const int tid);


////////////////////////////////////////////////////////////////////////
template <typename T>
void PointersManager::printDevContentOnHost(const void* pDev, const LongType len) const {
  printf("host print out\n");
  void* pHost = operator new(sizeof(T) * len);

  hipMemcpyAsync(pHost, pDev, sizeof(T) * len, hipMemcpyDeviceToHost, *_context->getCudaStream());
  hipError_t cudaResult = hipStreamSynchronize(*_context->getCudaStream());
  if (cudaResult != 0) THROW_EXCEPTION("PointersManager::printCudaHost: hipStreamSynchronize failed!");

  for (LongType i = 0; i < len; ++i) printf("%f, ", (double)reinterpret_cast<T*>(pHost)[i]);
  printf("\n");

  operator delete(pHost);
}

template void PointersManager::printDevContentOnHost<LongType>(const void* pDev, const LongType len) const;
template void PointersManager::printDevContentOnHost<int>(const void* pDev, const LongType len) const;
template void PointersManager::printDevContentOnHost<float>(const void* pDev, const LongType len) const;
template void PointersManager::printDevContentOnHost<double>(const void* pDev, const LongType len) const;

}  // namespace sd
