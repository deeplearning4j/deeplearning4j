/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
//  @author raver119@gmail.com
//
#include <array/CudaPointerDeallocator.h>

namespace sd {

void CudaPointerDeallocator::release(void *ptr) {
  if (ptr == nullptr) return;

  // Check if this is a valid device pointer before freeing
  hipPointerAttribute_t attributes;
  hipError_t result = hipPointerGetAttributes(&attributes, ptr);

  if (result == hipSuccess) {
    // Only free if it's a regular device pointer
    // hipMemoryTypeDevice is for regular allocations we can free
    if (attributes.type == hipMemoryTypeDevice) {
      hipFree(ptr);
    }
    // Don't free other types (like constant memory)
  } else {
    // Clear the error and don't try to free this pointer
    hipGetLastError(); // Clear the error state
  }
}
}  // namespace sd
