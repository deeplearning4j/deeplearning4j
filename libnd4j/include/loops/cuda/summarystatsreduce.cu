#include "hip/hip_runtime.h"
/* ******************************************************************************
*
*
* This program and the accompanying materials are made available under the
* terms of the Apache License, Version 2.0 which is available at
* https://www.apache.org/licenses/LICENSE-2.0.
*
*  See the NOTICE file distributed with this work for additional
*  information regarding copyright ownership.
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
* WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
* License for the specific language governing permissions and limitations
* under the License.
*
* SPDX-License-Identifier: Apache-2.0
******************************************************************************/

//
// @author raver119@gmail.com
//

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <helpers/DebugHelper.h>

#include <helpers/shape.h>
#include <loops/summarystatsreduce.h>
#include <ops/specials_cuda.h>
#include <system/Environment.h>
#include <system/op_boilerplate.h>
#include <types/float16.h>
#include <types/types.h>

using namespace simdOps;

namespace functions {
namespace summarystats {

template <typename X, typename Z>
SD_KERNEL void summaryStatsReduceKernel(
    int op, void * dx, sd::LongType * xShapeInfo, sd::LongType xRank,
    void* extraParams, void* z, sd::LongType * zShapeInfo, sd::LongType zRank,
    sd::LongType* dimension, sd::LongType dimensionLength, int postProcessOrNot,
    bool biasCorrected, sd::LongType* allocationBuffer, void* reductionBuffer,
    sd::LongType * tadOnlyShapeInfo, sd::LongType * tadOffsets) {

  SummaryStatsReduce<X, Z>::transform(
      op, dx, xShapeInfo, extraParams, z, zShapeInfo, dimension, dimensionLength,
      postProcessOrNot, allocationBuffer, reductionBuffer, tadOnlyShapeInfo, tadOffsets);
}

/**
*
* @param sPartialsRef
* @param tid
* @param extraParams
 */
template <typename X, typename Z>
template <typename OpType>
SD_DEVICE void SummaryStatsReduce<X, Z>::aggregatePartials(SummaryStatsData<X>* sPartials, sd::LongType tid,
                                                           sd::LongType numElements, void* vextraParams) {
  // start the shared memory loop on the next power of 2 less
  // than the block size.  If block size is not a power of 2,
  // accumulate the intermediate sums in the remainder range.

  auto extraParams = static_cast<Z*>(vextraParams);
  sd::LongType floorPow2 = numElements;

  if (floorPow2 & (floorPow2 - 1)) {
    while (floorPow2 & (floorPow2 - 1)) {
      floorPow2 &= floorPow2 - 1;
    }

    if (tid >= floorPow2) {
      SummaryStatsData<X> prev = sPartials[tid - floorPow2];
      SummaryStatsData<X> curr = sPartials[tid];
      sPartials[tid - floorPow2] = update(prev, curr, extraParams);
    }
    __syncthreads();
  }

  for (sd::LongType activeThreads = floorPow2 >> 1; activeThreads; activeThreads >>= 1) {
    if (tid < activeThreads && tid + activeThreads < numElements) {
      SummaryStatsData<X> curr = sPartials[tid];
      SummaryStatsData<X> next = sPartials[tid + activeThreads];
      sPartials[tid] = update(curr, next, extraParams);
    }
    __syncthreads();
  }
}

/**
* @param n n is the number of
*        elements to loop through
* @param dx the data to operate on
* @param xVectorInfo the meta data for the vector:
*                              0 is the offset
*                              1 is the increment/stride
*                              2 is the real length of the buffer (n and dx.length won't always be the same)
*                              3 is the element wise stride for the buffer
*                              4 is the number of elements it takes to get to the next row/column/tensor
* @param gpuInformation
*                              0 is the block size
*                              1 is the grid size
*                              2 is the shared memory size
* @param problemDefinition
*                          0 is the number of elements per vector
*                          1 is the number of vectors
 */
template <typename X, typename Z>
template <typename OpType>
SD_DEVICE void SummaryStatsReduce<X, Z>::transform(void * vx, sd::LongType * xShapeInfo, void* vextraParams,
                                                   void* vz, sd::LongType * zShapeInfo, sd::LongType* dimension,
                                                   sd::LongType dimensionLength, int postProcessOrNot,
                                                   sd::LongType* allocationBuffer,
                                                   void* vreductionBuffer, sd::LongType * tadOnlyShapeInfo,
                                                   sd::LongType * tadOffsets) {
  auto dx = static_cast<X *>(vx);
  auto z = static_cast<Z*>(vz);
  auto extraParams = static_cast<Z*>(vextraParams);
  auto reductionBuffer = static_cast<Z*>(vreductionBuffer);

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ volatile bool resultScalar;

  int numElements = blockDim.x;
  // shared memory space for storing intermediate results
  __shared__ SummaryStatsData<X> sPartials[SD_CUDA_BLOCK_SIZE];

  // Cache shape information for x buffer
  __shared__ sd::LongType xRank;
  __shared__  sd::LongType* xShapePtr;
  __shared__  sd::LongType* xStridePtr;

  // Cache shape information for TAD
  __shared__ sd::LongType tadRank;
  __shared__  sd::LongType* tadShapePtr;
  __shared__  sd::LongType* tadStridePtr;

  Z startingVal = startingValue(dx);

  SummaryStatsData<X> val;
  val.initWithValue(startingVal);
  val.n = 0;
  sPartials[threadIdx.x] = val;

  // length for the tad
  __shared__ volatile int xLength;
  __shared__ volatile int resultLength;

  SummaryStatsData<X> reduction;
  reduction.initWithValue(0.0);
  reduction.n = 0;

  if (threadIdx.x == 0) {
    if (zShapeInfo != nullptr)
      resultLength = shape::length(zShapeInfo);
    else
      resultLength = 1;



    if (resultLength <= 1)
      resultScalar = 1;

    xLength = shape::length(xShapeInfo);

    // Cache x shape information
    xRank = shape::rank(xShapeInfo);
    xShapePtr = shape::shapeOf(xShapeInfo);
    xStridePtr = shape::stride(xShapeInfo);

    // Cache TAD shape information
    if (tadOnlyShapeInfo != nullptr && !resultScalar) {
      tadRank = shape::rank(tadOnlyShapeInfo);
      tadShapePtr = shape::shapeOf(tadOnlyShapeInfo);
      tadStridePtr = shape::stride(tadOnlyShapeInfo);
    }
  }
  __syncthreads();

  if (!resultScalar) {
    __shared__ int tadLength;
    __shared__ int numTads;

    if (threadIdx.x == 0) {
      tadLength = shape::length(tadOnlyShapeInfo);
      numTads = shape::length(xShapeInfo) / tadLength;
    }
    __syncthreads();

    for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
      auto tadOffsetForBlock = tadOffsets[r];

      val.initWithValue(startingVal);
      val.n = 0;
      sPartials[threadIdx.x] = val;

      for (int i = threadIdx.x; i < tadLength; i += blockDim.x) {
        sd::LongType xCoords[SD_MAX_RANK];
        sd::LongType xOffset;
        INDEX2COORDS(i, tadRank, tadShapePtr, xCoords);
        COORDS2INDEX(tadRank, tadStridePtr, xCoords, xOffset);
        auto xOffsetFinal = tadOffsetForBlock + xOffset;
        SummaryStatsData<X> indexVal2;
        indexVal2.initWithValue(dx[xOffsetFinal]);

        sPartials[threadIdx.x] = update(sPartials[threadIdx.x], OpType::op(indexVal2, extraParams), extraParams);
      }
      __syncthreads();
      aggregatePartials<OpType>(sPartials, threadIdx.x, sd::math::sd_min<int>(blockDim.x, tadLength), extraParams);

      __syncthreads();
      if (threadIdx.x == 0) {
        z[r] = OpType::getValue(postProcessOrNot, sPartials[threadIdx.x]);
      }
      __syncthreads();
    }
  } else if (resultScalar) {
    __shared__ int n;
    if (threadIdx.x == 0) {
      n = shape::length(xShapeInfo);
    }
    __syncthreads();

    for (sd::LongType i = tid; i < n; i += blockDim.x * gridDim.x) {
      sd::LongType xCoords[SD_MAX_RANK];
      sd::LongType xOffset;
      INDEX2COORDS(i, xRank, xShapePtr, xCoords);
      COORDS2INDEX(xRank, xStridePtr, xCoords, xOffset);
      SummaryStatsData<X> indexVal2;
      indexVal2.initWithValue(dx[xOffset]);
      reduction = update(reduction, indexVal2, extraParams);
    }
    sPartials[threadIdx.x] = reduction;

    __syncthreads();
    aggregatePartials<OpType>(sPartials, threadIdx.x, blockDim.x, extraParams);
    __syncthreads();

    if (gridDim.x > 1) {
      __shared__ bool amLast;
      unsigned int* tc = (unsigned int*)reductionBuffer;
      tid = threadIdx.x;
      if (threadIdx.x == 0) {
        SummaryStatsData<X>* pBuffer = (SummaryStatsData<X>*)reductionBuffer;
        pBuffer[blockIdx.x] = sPartials[0];
      }
      __threadfence();
      __syncthreads();

      if (tid == 0) {
        unsigned int ticket = atomicInc(&tc[16384], gridDim.x);
        amLast = (ticket == gridDim.x - 1);
      }

      __syncthreads();

      if (amLast) {
        tc[16384] = 0;
        SummaryStatsData<X>* pBuffer = (SummaryStatsData<X>*)reductionBuffer;

        Z startingVal = startingValue(dx);

        SummaryStatsData<X> val;
        val.initWithValue(startingVal);
        val.n = 0;
        sPartials[threadIdx.x] = val;

        for (int i = threadIdx.x; i < gridDim.x; i += blockDim.x) {
          sPartials[threadIdx.x] = update(sPartials[threadIdx.x], pBuffer[i], extraParams);
        }

        __syncthreads();
        aggregatePartials<OpType>(sPartials, threadIdx.x, gridDim.x, extraParams);
        __syncthreads();

        if (tid == 0) {
          z[0] = OpType::getValue(postProcessOrNot, sPartials[0]);
        }
      }
    } else {
      if (tid == 0) {
        unsigned int* tc = (unsigned*)reductionBuffer;
        tc[16384] = 0;
        z[0] = OpType::getValue(postProcessOrNot, sPartials[0]);
      }
    }
  }
}

template <typename X, typename Y>
SD_DEVICE void SummaryStatsReduce<X, Y>::transform( int opNum, void * dx, sd::LongType * xShapeInfo,
                                                   void* extraParams, void* z, sd::LongType * zShapeInfo,
                                                   sd::LongType* dimension, sd::LongType dimensionLength, int postProcessOrNot, sd::LongType* allocationBuffer, void* reductionBuffer,
                                                   sd::LongType * tadOnlyShapeInfo,
                                                   sd::LongType * tadOffsets) {
  DISPATCH_BY_OPNUM_TT(transform,
                       PARAMS(dx, xShapeInfo, extraParams, z, zShapeInfo, dimension, dimensionLength, postProcessOrNot,
                              allocationBuffer, reductionBuffer, tadOnlyShapeInfo, tadOffsets),
                       SUMMARY_STATS_OPS);
}

template <typename X, typename Z>
SD_HOST void SummaryStatsReduce<X, Z>::execSummaryStatsReduceScalar(
    dim3& launchDims, hipStream_t* stream, int opNum, void * vx, sd::LongType * xShapeInfo,
    sd::LongType * hxShapeInfo, void* vextraParams, void* vz, sd::LongType * zShapeInfo,
    sd::LongType * hzShapeInfo, sd::LongType * tadShapeInfo, sd::LongType * tadOffsets,
    bool biasCorrected, void* reductionBuffer) {
  auto x = static_cast<X *>(vx);
  auto extraParams = static_cast<Z*>(vextraParams);
  auto z = reinterpret_cast<Z*>(vz);
  auto reductionPointerA = reinterpret_cast<Z*>(reductionBuffer);

  if (sd::Environment::getInstance().isDebugAndVerbose()) printf("D16 opNum:[%i]\n", opNum);

  summaryStatsReduceKernel<X, Z><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(
      opNum,
      x,
      xShapeInfo,
      shape::rank(hxShapeInfo),
      extraParams,
      z,
      zShapeInfo,
      shape::rank(hzShapeInfo),
      nullptr,
      0,
      1,
      biasCorrected,
      nullptr,
      reductionPointerA,
      tadShapeInfo,
      tadOffsets);

  // this is blocking method since method should return scalar
  sd::DebugHelper::checkErrorCode(stream, "execSSReduceScalar(...) failed");
}

template <typename X, typename Z>
SD_HOST void SummaryStatsReduce<X, Z>::execSummaryStatsReduce(
    dim3& launchDims, hipStream_t* stream, int opNum, void * vx, sd::LongType * xShapeInfo,
    sd::LongType * hxShapeInfo, void* vextraParams, void* vz, sd::LongType * zShapeInfo,
    sd::LongType * hzShapeInfo, sd::LongType * tadShapeInfo, sd::LongType * tadOffsets,
    bool biasCorrected, void* reductionBuffer) {
  auto x = static_cast<X *>(vx);
  auto z = static_cast<Z*>(vz);
  auto extraParams = static_cast<Z*>(vextraParams);

  if (sd::Environment::getInstance().isDebugAndVerbose()) printf("F17 opNum:[%i]\n", opNum);

  auto reductionPointerA = reinterpret_cast<Z*>(reductionBuffer);

  summaryStatsReduceKernel<X, Z><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(
      opNum, x, xShapeInfo, shape::rank(hxShapeInfo), extraParams, z, zShapeInfo, shape::rank(hzShapeInfo), nullptr, 1,
      1, biasCorrected, nullptr, reductionPointerA, tadShapeInfo, tadOffsets);

  DEBUG_KERNEL(stream, opNum);
}

BUILD_DOUBLE_TEMPLATE(template class SummaryStatsReduce, , SD_COMMON_TYPES, SD_FLOAT_TYPES);

}
}
