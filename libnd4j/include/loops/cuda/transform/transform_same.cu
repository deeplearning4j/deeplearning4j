#include "hip/hip_runtime.h"
/* ******************************************************************************
*
*
* This program and the accompanying materials are made available under the
* terms of the Apache License, Version 2.0 which is available at
* https://www.apache.org/licenses/LICENSE-2.0.
*
*  See the NOTICE file distributed with this work for additional
*  information regarding copyright ownership.
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
* WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
* License for the specific language governing permissions and limitations
* under the License.
*
* SPDX-License-Identifier: Apache-2.0
******************************************************************************/

//
// @author raver119@gmail.com
//
#include <helpers/DebugHelper.h>
#include <loops/legacy_ops.h>
#include <loops/transform_same.h>
#include <system/Environment.h>
#include <system/op_boilerplate.h>
#include <types/types.h>


using namespace simdOps;

template <typename X, typename OpType>
SD_KERNEL void transformSameSimple(const void *x, const sd::LongType *xShapeInfo, long long int xRank, void *params, void *z,
                                  const sd::LongType *zShapeInfo, long long int zRank,
                                  sd::LongType *allocationPointer,
                                  void *reductionPointer, const sd::LongType *tadShapeInfo,
                                  const sd::LongType *tadOffsets) {
 functions::transform::TransformSame<X>::template transformCuda<OpType>(
     x, xShapeInfo, params, z, zShapeInfo, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets);
}

namespace functions {
namespace transform {

template <typename X>
SD_HOST void TransformSame<X>::executeTransformShaped(dim3 launchDims, hipStream_t *stream, const int opNum,
                                                     const void *x, const sd::LongType *xShape, sd::LongType xRank,
                                                     void *extraParams, void *z, const sd::LongType *zShape,
                                                     sd::LongType zRank, sd::LongType *allocationPointer, void *reductionPointer,
                                                     const sd::LongType *tadShapeInfo,
                                                     const sd::LongType *tadOffsets) {
 DISPATCH_BY_OPNUM_T(intermediateShaped,
                     PARAMS(launchDims, stream, x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer,
                            reductionPointer, tadShapeInfo, tadOffsets),
                     TRANSFORM_SAME_OPS);

 sd::DebugHelper::checkErrorCode(stream, "transformAny(...) failed");
}

template <typename X>
template <typename OpType>
SD_DEVICE void TransformSame<X>::transformCuda(const void *vx, const sd::LongType *xShapeInfo, void *vparams, void *vz,
                                               const sd::LongType *zShapeInfo, sd::LongType *allocationPointer,
                                               void *vreductionPointer, const sd::LongType *tadShapeInfo,
                                               const sd::LongType *tadOffsets) {
 auto x = static_cast<const X *>(vx);
 auto z = static_cast<X *>(vz);
 auto params = static_cast<X *>(vparams);
 auto reductionPointer = static_cast<X *>(vreductionPointer);

 if (OpType::requiresSpecial) {
   OpType::execSpecialCuda(x, xShapeInfo, z, zShapeInfo, params, allocationPointer, reductionPointer, tadShapeInfo,
                           tadOffsets);
   return;
 } else {
   __shared__ sd::LongType length;

   // Cache shape information for x buffer
   __shared__ sd::LongType xRank;
   __shared__ const sd::LongType* xShapePtr;
   __shared__ const sd::LongType* xStridePtr;

   // Cache shape information for z buffer
   __shared__ sd::LongType zRank;
   __shared__ const sd::LongType* zShapePtr;
   __shared__ const sd::LongType* zStridePtr;

   if (threadIdx.x == 0) {
     length = shape::length(xShapeInfo);

     // Cache x shape information
     xRank = shape::rank(xShapeInfo);
     xShapePtr = shape::shapeOf(xShapeInfo);
     xStridePtr = shape::stride(xShapeInfo);

     // Cache z shape information
     zRank = shape::rank(zShapeInfo);
     zShapePtr = shape::shapeOf(zShapeInfo);
     zStridePtr = shape::stride(zShapeInfo);
   }
   __syncthreads();

   auto tid = blockIdx.x * blockDim.x + threadIdx.x;
   int totalThreads = gridDim.x * blockDim.x;

   for (sd::LongType i = tid; i < length; i += totalThreads) {
     sd::LongType xCoords[SD_MAX_RANK];
     sd::LongType zCoords[SD_MAX_RANK];
     sd::LongType xOffset;
     sd::LongType zOffset;

     INDEX2COORDS(i, xRank, xShapePtr, xCoords);
     COORDS2INDEX(xRank, xStridePtr, xCoords, xOffset);
     INDEX2COORDS(i, zRank, zShapePtr, zCoords);
     COORDS2INDEX(zRank, zStridePtr, zCoords, zOffset);

     z[zOffset] = OpType::op(x[xOffset], params);
   }
 }
};
template <typename X>
template <typename OpType>
SD_HOST void TransformSame<X>::intermediateShaped(dim3 launchDims, hipStream_t *stream, const void *x,
                                                 const sd::LongType *xShape, sd::LongType xRank, void *extraParams, void *z,
                                                 const sd::LongType *zShape, sd::LongType zRank,
                                                 sd::LongType *allocationPointer,
                                                 void *reductionPointer, const sd::LongType *tadShapeInfo,
                                                 const sd::LongType *tadOffsets) {
 transformSameSimple<X, OpType><<<launchDims.x, launchDims.x, launchDims.z, *stream>>>(
     x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets);
 sd::DebugHelper::checkErrorCode(stream, "transformSame(...) failed");
}

BUILD_SINGLE_TEMPLATE(template class TransformSame, , SD_COMMON_TYPES);
}  // namespace transform
}  // namespace functions
