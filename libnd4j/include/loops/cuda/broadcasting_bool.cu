#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing,
 * software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See
 * the License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <helpers/StringUtils.h>
#include <loops/broadcasting_bool.h>
#include <loops/legacy_ops.h>
#include <system/Environment.h>
#include <system/op_boilerplate.h>
#include <types/types.h>

    using namespace simdOps;

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z, typename OpClass>
static SD_KERNEL void broadcastBoolSimple(
    void const* x,
    sd::LongType const* xShapeInfo,
    void const* y,
    sd::LongType const* yShapeInfo,
    void* z,
    sd::LongType const* zShapeInfo,
    void* extraParams,
    sd::LongType* dimension,
    sd::LongType dimensionLength,
    sd::LongType const* tadOnlyShapeInfo,
    sd::LongType const* tadOffsets,
    sd::LongType const* tadOnlyShapeInfoZ,
    sd::LongType const* tadOffsetsZ) {

  functions::broadcast::BroadcastBool<X, Z>::template transformCuda<OpClass>(
      x,
      xShapeInfo,
      y,
      yShapeInfo,
      z,
      zShapeInfo,
      extraParams,
      dimension,
      dimensionLength,
      tadOnlyShapeInfo,
      tadOffsets,
      tadOnlyShapeInfoZ,
      tadOffsetsZ);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z, typename OpClass>
static SD_KERNEL void broadcastBoolSimple(
    const void* x,
    const sd::LongType* xShapeInfo,
    const void* y,
    const sd::LongType* yShapeInfo,
    void* z,
    const sd::LongType* zShapeInfo,
    void* extraParams) {

  functions::broadcast::BroadcastBool<X, Z>::template transformCuda<OpClass>(
      x,
      xShapeInfo,
      y,
      yShapeInfo,
      z,
      zShapeInfo,
      extraParams);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z, typename OpClass>
static SD_KERNEL void broadcastBoolInverseSimple(
    void const* x,
    sd::LongType const* xShapeInfo,
    void const* y,
    sd::LongType const* yShapeInfo,
    void* z,
    sd::LongType const* zShapeInfo,
    void* extraParams,
    sd::LongType* dimension,
    sd::LongType dimensionLength,
    sd::LongType const* tadOnlyShapeInfo,
    sd::LongType const* tadOffsets,
    sd::LongType const* tadOnlyShapeInfoZ,
    sd::LongType const* tadOffsetsZ) {

  functions::broadcast::BroadcastBool<X, Z>::template transformInverseCuda<OpClass>(
      x,
      xShapeInfo,
      y,
      yShapeInfo,
      z,
      zShapeInfo,
      extraParams,
      dimension,
      dimensionLength,
      tadOnlyShapeInfo,
      tadOffsets,
      tadOnlyShapeInfoZ,
      tadOffsetsZ);
}

namespace functions {
namespace broadcast {

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
template <typename OpClass>
SD_HOST void BroadcastBool<X, Z>::intermediateBroadcast(
    dim3 launchDims,
    hipStream_t* stream,
    void const* x,
    sd::LongType const* xShapeInfo,
    void const* y,
    sd::LongType const* yShapeInfo,
    void* z,
    sd::LongType const* zShapeInfo,
    void* extraParams,
    sd::LongType* dimension,
    sd::LongType dimensionLength,
    sd::LongType const* tadOnlyShapeInfo,
    sd::LongType const* tadOffsets,
    sd::LongType const* tadOnlyShapeInfoZ,
    sd::LongType const* tadOffsetsZ) {

  broadcastBoolSimple<X, Z, OpClass>
      <<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(
          x,
          xShapeInfo,
          y,
          yShapeInfo,
          z,
          zShapeInfo,
          extraParams,
          dimension,
          dimensionLength,
          tadOnlyShapeInfo,
          tadOffsets,
          tadOnlyShapeInfoZ,
          tadOffsetsZ);

  sd::DebugHelper::checkErrorCode(stream, "intermediateBroadcastBool(...) failed");
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
template <typename OpClass>
SD_HOST void BroadcastBool<X, Z>::intermediateBroadcast(
    dim3 launchDims,
    hipStream_t* stream,
    const void* x,
    const sd::LongType* xShapeInfo,
    const void* y,
    const sd::LongType* yShapeInfo,
    void* z,
    const sd::LongType* zShapeInfo,
    void* extraParams) {

  broadcastBoolSimple<X, Z, OpClass>
      <<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(
          x,
          xShapeInfo,
          y,
          yShapeInfo,
          z,
          zShapeInfo,
          extraParams);

  sd::DebugHelper::checkErrorCode(stream, "intermediateBroadcastBool(...) failed");
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
SD_HOST void BroadcastBool<X, Y>::execBroadcast(
    dim3 launchDims,
    hipStream_t* stream,
    int opNum,
    void const* x,
    sd::LongType const* xShapeInfo,
    void const* y,
    sd::LongType const* yShapeInfo,
    void* z,
    sd::LongType const* zShapeInfo,
    void* extraParams,
    sd::LongType* dimension,
    sd::LongType dimensionLength,
    sd::LongType const* tadOnlyShapeInfo,
    sd::LongType const* tadOffsets,
    sd::LongType const* tadOnlyShapeInfoZ,
    sd::LongType const* tadOffsetsZ) {

  DISPATCH_BY_OPNUM_TT(
      intermediateBroadcast,
      PARAMS(launchDims, stream, x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, extraParams,
             dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets,
             tadOnlyShapeInfoZ, tadOffsetsZ),
      OPS_A(BROADCAST_BOOL_OPS));

  sd::DebugHelper::checkErrorCode(stream, "execBroadcast(...) failed");
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
SD_HOST void BroadcastBool<X, Y>::execBroadcast(
    dim3 launchDims,
    hipStream_t* stream,
    const int opNum,
    const void* x,
    const sd::LongType* xShapeInfo,
    const void* y,
    const sd::LongType* yShapeInfo,
    void* z,
    const sd::LongType* zShapeInfo,
    void* extraParams) {

  DISPATCH_BY_OPNUM_TT(
      intermediateBroadcast,
      PARAMS(launchDims, stream, x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, extraParams),
      OPS_A(BROADCAST_BOOL_OPS));

  DEBUG_KERNEL(stream, opNum);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
template <typename OpClass>
SD_HOST void BroadcastBool<X, Z>::intermediateInverseBroadcast(
    dim3 launchDims,
    hipStream_t* stream,
    void const* x,
    sd::LongType const* xShapeInfo,
    void const* y,
    sd::LongType const* yShapeInfo,
    void* z,
    sd::LongType const* zShapeInfo,
    void* extraParams,
    sd::LongType* dimension,
    sd::LongType dimensionLength,
    sd::LongType const* tadOnlyShapeInfo,
    sd::LongType const* tadOffsets,
    sd::LongType const* tadOnlyShapeInfoZ,
    sd::LongType const* tadOffsetsZ) {

  broadcastBoolInverseSimple<X, Z, OpClass>
      <<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(
          x,
          xShapeInfo,
          y,
          yShapeInfo,
          z,
          zShapeInfo,
          extraParams,
          dimension,
          dimensionLength,
          tadOnlyShapeInfo,
          tadOffsets,
          tadOnlyShapeInfoZ,
          tadOffsetsZ);

  sd::DebugHelper::checkErrorCode(stream, "intermediateBroadcastBool(...) failed");
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
SD_HOST void BroadcastBool<X, Y>::execInverseBroadcast(
    dim3 launchDims,
    hipStream_t* stream,
    int opNum,
    void const* x,
    sd::LongType const* xShapeInfo,
    void const* y,
    sd::LongType const* yShapeInfo,
    void* z,
    sd::LongType const* zShapeInfo,
    void* extraParams,
    sd::LongType* dimension,
    sd::LongType dimensionLength,
    sd::LongType const* tadOnlyShapeInfo,
    sd::LongType const* tadOffsets,
    sd::LongType const* tadOnlyShapeInfoZ,
    sd::LongType const* tadOffsetsZ) {

  DISPATCH_BY_OPNUM_TT(
      intermediateInverseBroadcast,
      PARAMS(launchDims, stream, x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, extraParams,
             dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets,
             tadOnlyShapeInfoZ, tadOffsetsZ),
      OPS_A(BROADCAST_BOOL_OPS));

  sd::DebugHelper::checkErrorCode(stream, "execInverseBroadcast(...) failed");
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
template <typename OpType>
SD_DEVICE void BroadcastBool<X, Z>::transformInverseCuda(
    void const* vx,
    sd::LongType const* xShapeInfo,
    void const* vy,
    sd::LongType const* yShapeInfo,
    void* vz,
    sd::LongType const* zShapeInfo,
    void* vextraParams,
    sd::LongType* dimension,
    sd::LongType dimensionLength,
    sd::LongType const* tadOnlyShapeInfo,
    sd::LongType const* tadOffsets,
    sd::LongType const* tadOnlyShapeInfoZ,
    sd::LongType const* tadOffsetsZ) {

  if (tadOnlyShapeInfoZ == nullptr) {
    tadOnlyShapeInfoZ = tadOnlyShapeInfo;
    tadOffsetsZ       = tadOffsets;
  }

  auto x          = reinterpret_cast<const X*>(vx);
  auto y          = reinterpret_cast<const X*>(vy);
  auto z          = reinterpret_cast<Z*>(vz);
  auto extraParams= reinterpret_cast<X*>(vextraParams);

  __shared__ sd::LongType tadLength;
  __shared__ int numTads;

  __shared__ sd::LongType xRank;
  __shared__ const sd::LongType* xShapePtr;
  __shared__ const sd::LongType* xStridePtr;

  __shared__ sd::LongType tadRank;
  __shared__ const sd::LongType* tadShapePtr;
  __shared__ const sd::LongType* tadStridePtr;

  __shared__ sd::LongType tadRankZ;
  __shared__ const sd::LongType* tadShapePtrZ;
  __shared__ const sd::LongType* tadStridePtrZ;

  if (threadIdx.x == 0) {
    tadLength = shape::length(tadOnlyShapeInfo);
    numTads   = shape::length(yShapeInfo) / tadLength;

    xRank     = shape::rank(xShapeInfo);
    xShapePtr = shape::shapeOf(xShapeInfo);
    xStridePtr= shape::stride(xShapeInfo);

    tadRank     = shape::rank(tadOnlyShapeInfo);
    tadShapePtr = shape::shapeOf(tadOnlyShapeInfo);
    tadStridePtr= shape::stride(tadOnlyShapeInfo);

    tadRankZ     = shape::rank(tadOnlyShapeInfoZ);
    tadShapePtrZ = shape::shapeOf(tadOnlyShapeInfoZ);
    tadStridePtrZ= shape::stride(tadOnlyShapeInfoZ);
  }
  __syncthreads();

  for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
    auto rZ = z + tadOffsetsZ[r];
    auto rY = y + tadOffsets[r];

    for (sd::LongType i = threadIdx.x; i < tadLength; i += blockDim.x) {
      sd::LongType coordsX[SD_MAX_RANK];
      sd::LongType coordsY[SD_MAX_RANK];
      sd::LongType coordsZ[SD_MAX_RANK];
      sd::LongType xOffset;
      sd::LongType yOffset;
      sd::LongType zOffset;

      // for x
      INDEX2COORDS(i, xRank, xShapePtr, coordsX);
      COORDS2INDEX(xRank, xStridePtr, coordsX, xOffset);

      // for y (tad)
      INDEX2COORDS(i, tadRank, tadShapePtr, coordsY);
      COORDS2INDEX(tadRank, tadStridePtr, coordsY, yOffset);

      // for z (tadZ)
      INDEX2COORDS(i, tadRankZ, tadShapePtrZ, coordsZ);
      COORDS2INDEX(tadRankZ, tadStridePtrZ, coordsZ, zOffset);

      rZ[zOffset] = OpType::op(x[xOffset], rY[yOffset], extraParams);
    }
  }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
template <typename OpType>
SD_DEVICE void BroadcastBool<X, Z>::transformCuda(
    void const* vx,
    sd::LongType const* xShapeInfo,
    void const* vy,
    sd::LongType const* yShapeInfo,
    void* vz,
    sd::LongType const* zShapeInfo,
    void* vextraParams,
    sd::LongType* dimension,
    sd::LongType dimensionLength,
    sd::LongType const* tadOnlyShapeInfo,
    sd::LongType const* tadOffsets,
    sd::LongType const* tadOnlyShapeInfoZ,
    sd::LongType const* tadOffsetsZ) {

  if (tadOnlyShapeInfoZ == nullptr) {
    tadOnlyShapeInfoZ = tadOnlyShapeInfo;
    tadOffsetsZ       = tadOffsets;
  }

  auto x          = reinterpret_cast<const X*>(vx);
  auto y          = reinterpret_cast<const X*>(vy);
  auto z          = reinterpret_cast<Z*>(vz);
  auto extraParams= reinterpret_cast<X*>(vextraParams);

  __shared__ sd::LongType tadLength;
  __shared__ sd::LongType numTads;

  __shared__ sd::LongType xRank;
  __shared__ const sd::LongType* xShapePtr;
  __shared__ const sd::LongType* xStridePtr;

  __shared__ sd::LongType yRank;
  __shared__ const sd::LongType* yShapePtr;
  __shared__ const sd::LongType* yStridePtr;

  __shared__ sd::LongType zRank;
  __shared__ const sd::LongType* zShapePtr;
  __shared__ const sd::LongType* zStridePtr;

  if (threadIdx.x == 0) {
    tadLength = shape::length(tadOnlyShapeInfo);
    numTads   = shape::length(xShapeInfo) / tadLength;

    xRank     = shape::rank(tadOnlyShapeInfo);
    xShapePtr = shape::shapeOf(tadOnlyShapeInfo);
    xStridePtr= shape::stride(tadOnlyShapeInfo);

    yRank     = shape::rank(yShapeInfo);
    yShapePtr = shape::shapeOf(yShapeInfo);
    yStridePtr= shape::stride(yShapeInfo);

    zRank     = shape::rank(tadOnlyShapeInfoZ);
    zShapePtr = shape::shapeOf(tadOnlyShapeInfoZ);
    zStridePtr= shape::stride(tadOnlyShapeInfoZ);
  }
  __syncthreads();

  for (sd::LongType r = blockIdx.x; r < numTads; r += gridDim.x) {
    auto rX = x + tadOffsets[r];
    auto rZ = z + tadOffsetsZ[r];

    for (sd::LongType i = threadIdx.x; i < tadLength; i += blockDim.x) {
      sd::LongType coordsX[SD_MAX_RANK];
      sd::LongType coordsY[SD_MAX_RANK];
      sd::LongType coordsZ[SD_MAX_RANK];
      sd::LongType xOffset;
      sd::LongType yOffset;
      sd::LongType zOffset;

      INDEX2COORDS(i, xRank, xShapePtr, coordsX);
      COORDS2INDEX(xRank, xStridePtr, coordsX, xOffset);

      INDEX2COORDS(i, yRank, yShapePtr, coordsY);
      COORDS2INDEX(yRank, yStridePtr, coordsY, yOffset);

      INDEX2COORDS(i, zRank, zShapePtr, coordsZ);
      COORDS2INDEX(zRank, zStridePtr, coordsZ, zOffset);

      rZ[zOffset] = OpType::op(rX[xOffset], y[yOffset], extraParams);
    }
  }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
template <typename OpType>
SD_DEVICE void BroadcastBool<X, Z>::transformCuda(
    const void* vx,
    const sd::LongType* xShapeInfo,
    const void* vy,
    const sd::LongType* yShapeInfo,
    void* vz,
    const sd::LongType* zShapeInfo,
    void* vextraParams) {

  const auto x = reinterpret_cast<const X*>(vx);
  const auto y = reinterpret_cast<const X*>(vy);
  auto z       = reinterpret_cast<Z*>(vz);
  auto extraParams = reinterpret_cast<X*>(vextraParams);

  __shared__ sd::LongType zLen;
  __shared__ int xRank, yRank, zRank;
  __shared__ bool xzSameOffsets, yzSameOffsets;

  // We'll store stride data for each shape in shared mem
  __shared__ const sd::LongType* xStridePtr;
  __shared__ const sd::LongType* yStridePtr;
  __shared__ const sd::LongType* zStridePtr;

  __shared__ const sd::LongType* xShapePtr;
  __shared__ const sd::LongType* yShapePtr;
  __shared__ const sd::LongType* zShapePtr;

  if (threadIdx.x == 0) {
    zLen           = shape::length(zShapeInfo);

    xRank          = shape::rank(xShapeInfo);
    yRank          = shape::rank(yShapeInfo);
    zRank          = shape::rank(zShapeInfo);

    xShapePtr      = shape::shapeOf(xShapeInfo);
    yShapePtr      = shape::shapeOf(yShapeInfo);
    zShapePtr      = shape::shapeOf(zShapeInfo);

    xStridePtr     = shape::stride(xShapeInfo);
    yStridePtr     = shape::stride(yShapeInfo);
    zStridePtr     = shape::stride(zShapeInfo);

    xzSameOffsets  = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
    yzSameOffsets  = shape::haveSameShapeAndStrides(yShapeInfo, zShapeInfo);
  }
  __syncthreads();

  const auto tid          = blockIdx.x * blockDim.x + threadIdx.x;
  const auto totalThreads = blockDim.x * gridDim.x;

  for (sd::LongType i2 = tid; i2 < zLen; i2 += totalThreads) {
    sd::LongType coords[SD_MAX_RANK];
    INDEX2COORDS(i2, zRank, zShapePtr, coords);

    sd::LongType zOffset;
    COORDS2INDEX(zRank, zStridePtr, coords, zOffset);

    sd::LongType xOffset;
    if (xzSameOffsets) {
      xOffset = zOffset;
    } else {
      COORDS2INDEX(xRank, xStridePtr, coords, xOffset);
    }

    sd::LongType yOffset;
    if (yzSameOffsets) {
      yOffset = zOffset;
    } else {
      COORDS2INDEX(yRank, yStridePtr, coords, yOffset);
    }

    z[zOffset] = OpType::op(x[xOffset], y[yOffset], extraParams);
  }
}

// build the class
BUILD_DOUBLE_TEMPLATE(template class BroadcastBool, , SD_COMMON_TYPES, SD_BOOL_TYPES);

}  // namespace broadcast
}  // namespace functions
