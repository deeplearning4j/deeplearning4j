#include "hip/hip_runtime.h"
/* ******************************************************************************
*
*
* This program and the accompanying materials are made available under the
* terms of the Apache License, Version 2.0 which is available at
* https://www.apache.org/licenses/LICENSE-2.0.
*
*  See the NOTICE file distributed with this work for additional
*  information regarding copyright ownership.
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
* WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
* License for the specific language governing permissions and limitations
* under the License.
*
* SPDX-License-Identifier: Apache-2.0
******************************************************************************/

//
// @author raver119@gmail.com
//
#include <ops/specials_cuda.h>

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
SD_KERNEL void execOesTadKernelKey(void *vx, sd::LongType const *xShapeInfo, void *vy, sd::LongType const *yShapeInfo,
                                   sd::LongType *dimension, long long int dimensionLength, sd::LongType const *tadShapeInfo,
                                   sd::LongType const *tadOffsets, bool descending) {
  auto x = static_cast<X *>(vx);
  auto y = static_cast<Y *>(vy);

  __shared__ int xLength;
  __shared__ int xTadLength;
  __shared__ int numTads;
  __shared__ int tadRank;
  __shared__ sd::LongType *tadShape;
  __shared__ sd::LongType *tadStride;

  if (threadIdx.x == 0) {
    xLength = shape::length(xShapeInfo);
    xTadLength = shape::length(tadShapeInfo);
    numTads = xLength / xTadLength;

    // Cache shape information
    tadRank = shape::rank(tadShapeInfo);
    tadShape = shape::shapeOf(tadShapeInfo);
    tadStride = shape::stride(tadShapeInfo);
  }
  __syncthreads();

  for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
    auto dx = x + tadOffsets[r];
    auto dy = y + tadOffsets[r];

    // this is general loop, we go uncached
    int iterations = xTadLength;

    for (int i = 0; i < iterations; i++) {
      if (i % 2 == 0) {
        for (int tid = threadIdx.x; tid < xTadLength; tid += blockDim.x) {
          auto top = 2 * tid + 1;
          if (top < xTadLength) {
            sd::LongType t0Coords[SD_MAX_RANK], t1Coords[SD_MAX_RANK];
            sd::LongType t0Offset, t1Offset;

            INDEX2COORDS(top - 1, tadRank, tadShape, t0Coords);
            COORDS2INDEX(tadRank, tadStride, t0Coords, t0Offset);
            INDEX2COORDS(top, tadRank, tadShape, t1Coords);
            COORDS2INDEX(tadRank, tadStride, t1Coords, t1Offset);

            if (!descending == (dx[t0Offset] > dx[t1Offset])) {
              X dt0 = dx[t0Offset];
              dx[t0Offset] = dx[t1Offset];
              dx[t1Offset] = dt0;

              Y dy0 = dy[t0Offset];
              dy[t0Offset] = dy[t1Offset];
              dy[t1Offset] = dy0;
            }
          }
        }
      } else {
        for (int tid = threadIdx.x; tid < xTadLength; tid += blockDim.x) {
          auto top = 2 * tid + 2;
          if (top < xTadLength) {
            sd::LongType t0Coords[SD_MAX_RANK], t1Coords[SD_MAX_RANK];
            sd::LongType t0Offset, t1Offset;

            INDEX2COORDS(top - 1, tadRank, tadShape, t0Coords);
            COORDS2INDEX(tadRank, tadStride, t0Coords, t0Offset);
            INDEX2COORDS(top, tadRank, tadShape, t1Coords);
            COORDS2INDEX(tadRank, tadStride, t1Coords, t1Offset);

            if (!descending == (dx[t0Offset] > dx[t1Offset])) {
              X dt0 = dx[t0Offset];
              dx[t0Offset] = dx[t1Offset];
              dx[t1Offset] = dt0;

              Y dy0 = dy[t0Offset];
              dy[t0Offset] = dy[t1Offset];
              dy[t1Offset] = dy0;
            }
          }
        }
      }
      __syncthreads();
    }
  }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void execOesTadKernel(void *vx, sd::LongType const *xShapeInfo, sd::LongType *dimension,
                                sd::LongType dimensionLength,
                                sd::LongType const *tadShapeInfo, sd::LongType const *tadOffsets, bool descending) {
  auto x = static_cast<T *>(vx);
  const int sharedSize = 32768;

  __shared__ int xLength;
  __shared__ int xTadLength;
  __shared__ int numTads;
  __shared__ T *shmem;
  __shared__ bool cached;
  __shared__ int tadRank;
  __shared__ sd::LongType *tadShape;
  __shared__ sd::LongType *tadStride;

  if (threadIdx.x == 0) {
    xLength = shape::length(xShapeInfo);
    xTadLength = shape::length(tadShapeInfo);
    numTads = xLength / xTadLength;

    extern __shared__ unsigned char shrd[];
    shmem = (T *)shrd;

    cached = xTadLength <= (sharedSize / sizeof(T));

    // Cache shape information
    tadRank = shape::rank(tadShapeInfo);
    tadShape = shape::shapeOf(tadShapeInfo);
    tadStride = shape::stride(tadShapeInfo);
  }
  __syncthreads();

  for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
    auto dx = x + tadOffsets[r];

    // this is general loop, we go uncached
    int iterations = xTadLength;
    if (cached) {
      for (int tid = threadIdx.x; tid < xTadLength; tid += blockDim.x) {
        sd::LongType xCoords[SD_MAX_RANK];
        sd::LongType xOffset;
        INDEX2COORDS(tid, tadRank, tadShape, xCoords);
        COORDS2INDEX(tadRank, tadStride, xCoords, xOffset);
        shmem[tid] = dx[xOffset];
      }

      __syncthreads();
      dx = shmem;
    }

    for (int i = 0; i < iterations; i++) {
      if (i % 2 == 0) {
        for (int tid = threadIdx.x; tid < xTadLength; tid += blockDim.x) {
          auto top = 2 * tid + 1;
          if (top < xTadLength) {
            sd::LongType t0Coords[SD_MAX_RANK], t1Coords[SD_MAX_RANK];
            sd::LongType t0Offset, t1Offset;

            INDEX2COORDS(top - 1, tadRank, tadShape, t0Coords);
            COORDS2INDEX(tadRank, tadStride, t0Coords, t0Offset);
            INDEX2COORDS(top, tadRank, tadShape, t1Coords);
            COORDS2INDEX(tadRank, tadStride, t1Coords, t1Offset);

            if (!descending == (dx[t0Offset] > dx[t1Offset])) {
              T dt0 = dx[t0Offset];
              dx[t0Offset] = dx[t1Offset];
              dx[t1Offset] = dt0;
            }
          }
        }
      } else {
        for (int tid = threadIdx.x; tid < xTadLength; tid += blockDim.x) {
          auto top = 2 * tid + 2;
          if (top < xTadLength) {
            sd::LongType t0Coords[SD_MAX_RANK], t1Coords[SD_MAX_RANK];
            sd::LongType t0Offset, t1Offset;

            INDEX2COORDS(top - 1, tadRank, tadShape, t0Coords);
            COORDS2INDEX(tadRank, tadStride, t0Coords, t0Offset);
            INDEX2COORDS(top, tadRank, tadShape, t1Coords);
            COORDS2INDEX(tadRank, tadStride, t1Coords, t1Offset);

            if (!descending == (dx[t0Offset] > dx[t1Offset])) {
              T dt0 = dx[t0Offset];
              dx[t0Offset] = dx[t1Offset];
              dx[t1Offset] = dt0;
            }
          }
        }
      }
      __syncthreads();
    }

    if (cached) {
      dx = x + tadOffsets[r];
      for (int tid = threadIdx.x; tid < xTadLength; tid += blockDim.x) {
        sd::LongType xCoords[SD_MAX_RANK];
        sd::LongType xOffset;
        INDEX2COORDS(tid, tadRank, tadShape, xCoords);
        COORDS2INDEX(tadRank, tadStride, xCoords, xOffset);
        dx[xOffset] = shmem[tid];
      }
    }
  }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
SD_HOST void oesTadGeneric(dim3 &launchDims, hipStream_t *stream, void *vx, sd::LongType const *xShapeInfo,
                           sd::LongType *dimension, sd::LongType dimensionLength, sd::LongType const *tadShapeInfo,
                           sd::LongType const *tadOffsets, bool descending) {
  execOesTadKernel<T><<<launchDims.y, launchDims.x, launchDims.z, *stream>>>(vx, xShapeInfo, dimension, dimensionLength,
                                                                             tadShapeInfo, tadOffsets, descending);

  sd::DebugHelper::checkErrorCode(stream, "execOesTadKernel failed");
}

template <typename X, typename Y>
SD_HOST void oesTadGenericKey(dim3 &launchDims, hipStream_t *stream, void *vx, sd::LongType const *xShapeInfo,
                              void *vy, sd::LongType const *yShapeInfo, sd::LongType *dimension,
                              sd::LongType dimensionLength,
                              sd::LongType const *tadShapeInfo, sd::LongType const *tadOffsets, bool descending) {
  execOesTadKernelKey<X, Y><<<launchDims.y, launchDims.x, launchDims.z, *stream>>>(
      vx, xShapeInfo, vy, yShapeInfo, dimension, dimensionLength, tadShapeInfo, tadOffsets, descending);
  sd::DebugHelper::checkErrorCode(stream, "execOesTadKernelKey failed");
}

BUILD_SINGLE_TEMPLATE(template void oesTadGeneric,
                      (dim3 & launchDims, hipStream_t *stream, void *vx, sd::LongType const *xShapeInfo,
                       sd::LongType *dimension, sd::LongType dimensionLength, sd::LongType const *tadShapeInfo,
                       sd::LongType const *tadOffsets, bool descending),
                      SD_COMMON_TYPES);

BUILD_DOUBLE_TEMPLATE(template void oesTadGenericKey,
                      (dim3 & launchDims, hipStream_t *stream, void *vx, sd::LongType const *xShapeInfo, void *vy,
                       sd::LongType const *yShapeInfo, sd::LongType *dimension, sd::LongType dimensionLength,
                       sd::LongType const *tadShapeInfo, sd::LongType const *tadOffsets, bool descending),
                      SD_COMMON_TYPES, SD_COMMON_TYPES);