#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//
#include <loops/special_kernels.h>


namespace sd {

///////////////////////////////////////////////////////////////////////
template <typename T>
SD_DEVICE void averagingKernel(void **vdx, void *vdz, int n, LongType length, bool propagate) {
  auto dx = reinterpret_cast<T **>(vdx);
  auto dz = reinterpret_cast<T *>(vdz);

  __shared__ T *shmem;

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char sharedmem[];
    shmem = (T *)sharedmem;
  }
  __syncthreads();

  // each block cycles over it's own part of arrays
  for (int r = blockDim.x * blockIdx.x; r < length; r += blockDim.x * gridDim.x) {
    shmem[threadIdx.x] = (T)0.0f;

    LongType baseIdx = r;

    // aggregation step, we roll over all arrays
    for (int ar = 0; ar < n; ar++) {
      T *cdata = (T *)dx[ar];
      cdata += baseIdx;

      if (baseIdx + threadIdx.x < length) shmem[threadIdx.x] += cdata[threadIdx.x];
    }

    // average data in shared memory
    if (baseIdx + threadIdx.x < length) shmem[threadIdx.x] /= n;

    // div step & write out step
    if (dz != nullptr) {
      T *wdata = dz + baseIdx;

      if (baseIdx + threadIdx.x < length) {
        wdata[threadIdx.x] = shmem[threadIdx.x];
      }
    }

    // propagate averaged data to all arrays
    if (propagate)
      for (int ar = 0; ar < n; ar++) {
        T *cdata = (T *)dx[ar];
        cdata += baseIdx;

        if (baseIdx + threadIdx.x < length) cdata[threadIdx.x] = shmem[threadIdx.x];
      }
  }
}

///////////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void execAveragingKernel(void **vdx, void *vdz, int n, LongType length, bool propagate) {
  averagingKernel<T>(vdx, vdz, n, length, propagate);
}

///////////////////////////////////////////////////////////////////////
template <typename T>
SD_HOST void averagingKernelGeneric(dim3 &launchDims, hipStream_t *stream, void **vdx, void *vdz, int n,
                                    LongType length, bool propagate) {
  execAveragingKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vdx, vdz, n, length, propagate);
  DebugHelper::checkErrorCode(stream, "averaging(...) failed");
}

BUILD_SINGLE_TEMPLATE( void averagingKernelGeneric,
                      (dim3 & launchDims, hipStream_t *stream, void **vdx, void *vdz, int n, sd::LongType length,
                       bool propagate),
                      SD_COMMON_TYPES);
}  // namespace sd
