#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//
#include <loops/special_kernels.h>


namespace sd {

///////////////////////////////////////////////////////////////////////
/**
 * This kernel accumulates X arrays, and stores z into Z
 *
 * @tparam T
 * @param x
 * @param z
 * @param n
 * @param length
 */
template <typename T>
SD_DEVICE void accumulateKernel(void **vx, void *vz, int n, const LongType length) {
  auto x = reinterpret_cast<T **>(vx);
  auto z = reinterpret_cast<T *>(vz);

  __shared__ T *shmem;

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char sharedmem[];
    shmem = (T *)sharedmem;
  }
  __syncthreads();

  for (int r = blockDim.x * blockIdx.x; r < length; r += blockDim.x * gridDim.x) {
    shmem[threadIdx.x] = 0.0f;

    LongType baseIdx = r;

    // aggregation step, we roll over all arrays
    for (int ar = 0; ar < n; ar++) {
      T *cdata = (T *)x[ar];
      cdata += baseIdx;

      if (baseIdx + threadIdx.x < length) shmem[threadIdx.x] += cdata[threadIdx.x];
    }

    T *wdata = z + baseIdx;

    // saving accumulated values
    if (baseIdx + threadIdx.x < length) wdata[threadIdx.x] = shmem[threadIdx.x];
  }
}

///////////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void execAccumulateKernel(void **vx, void *vz, int n, const LongType length) {
  accumulateKernel<T>(vx, vz, n, length);
}

///////////////////////////////////////////////////////////////////////
template <typename T>
SD_HOST void accumulateKernelGeneric(dim3 &launchDims, hipStream_t *stream, void **vx, void *vz, int n,
                                     const LongType length) {
  execAccumulateKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, vz, n, length);
  DebugHelper::checkErrorCode(stream, "accumulate(...) failed");
}

BUILD_SINGLE_TEMPLATE( void accumulateKernelGeneric,
                      (dim3 & launchDims, hipStream_t *stream, void **vx, void *vz, int n, const sd::LongType length),
                      SD_COMMON_TYPES);
}  // namespace sd
