#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//
#include <loops/special_kernels.h>


namespace sd {

///////////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void execConvertHalfs(half *dx, LongType n, void *dz) {
  auto z = reinterpret_cast<T *>(dz);
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  for (LongType i = tid; i < n; i += blockDim.x * gridDim.x) z[i] = static_cast<T>(__half2float(dx[i]));
}

///////////////////////////////////////////////////////////////////////
template <typename T>
SD_HOST void convertHalfsToGeneric(dim3 &launchDims, hipStream_t *stream, half *dx, LongType n, void *dz) {
  execConvertHalfs<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(dx, n, dz);
  DebugHelper::checkErrorCode(stream, "convertHalfsToGeneric(...) failed");
}

BUILD_SINGLE_TEMPLATE( void convertHalfsToGeneric,
                      (dim3 & launchDims, hipStream_t *stream, half *dx, sd::LongType n, void *dz), SD_COMMON_TYPES);
}  // namespace sd
