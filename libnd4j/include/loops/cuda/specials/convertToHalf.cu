#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//
#include <loops/special_kernels.h>


namespace sd {

////////////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void execConvertToHalf(void *dx, LongType n, half *dz) {
  auto x = reinterpret_cast<T *>(dx);
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  for (LongType i = tid; i < n; i += blockDim.x * gridDim.x) dz[i] = __float2half(static_cast<T>(x[i]));
}

////////////////////////////////////////////////////////////////////////
template <typename T>
SD_HOST void convertToHalfGeneric(dim3 &launchDims, hipStream_t *stream, void *dx, LongType n, half *dz) {
  execConvertToHalf<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(dx, n, dz);
  DebugHelper::checkErrorCode(stream, "convertToHalfs(...) failed");
}

BUILD_SINGLE_TEMPLATE( void convertToHalfGeneric,
                      (dim3 & launchDims, hipStream_t *stream, void *dx, sd::LongType n, half *dz), SD_COMMON_TYPES);

}  // namespace sd
