#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//
#include <array/NDArrayFactory.h>
#include <ops/declarable/helpers/legacy_helpers.h>
#include <ops/ops.h>
#include <system/op_boilerplate.h>


namespace sd {
namespace ops {
namespace helpers {

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T>
void cubeDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
  auto functor = LAMBDA_TT(x, y) { return y * (3 * x * x); });

  input->applyPairwiseLambda(epsilon, functor, output);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void cubeDerivative(LaunchContext* context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
  BUILD_SINGLE_SELECTOR(theFirst->dataType(), cubeDerivative_, (theFirst, theSecond, theOutput), SD_FLOAT_TYPES);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// return (x >= X(0.f) ? y: -y);
template <typename T>
void reduceNorm1_(NDArray* input, NDArray* epsilon, NDArray* output) {
  auto functor = LAMBDA_TT(x, y) { return x > T(0.f) ? y : -y; });

  input->applyPairwiseLambda(epsilon, functor, output);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void reduceNorm1(LaunchContext* context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
  BUILD_SINGLE_SELECTOR(theFirst->dataType(), reduceNorm1_, (theFirst, theSecond, theOutput), SD_FLOAT_TYPES);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////
template <typename T>
void sigmCrossEntropy_(NDArray* logits, NDArray* labels, NDArray* output) {
  auto functor = LAMBDA_TT(x, y) {
    return math::sd_max<T>(x, (T)0.f) - x * y + math::sd_log<T, T>((T)1.f + math::sd_exp<T, T>(-math::sd_abs<T,T>(x)));
  });

  logits->applyPairwiseLambda(labels, functor, output);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void sigmCrossEntropy(LaunchContext* context, NDArray* logits, NDArray* labels, NDArray* output) {
  BUILD_SINGLE_SELECTOR(logits->dataType(), sigmCrossEntropy_, (logits, labels, output), SD_FLOAT_TYPES);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////
template <typename T>
void sigmCrossEntropyGrad_(NDArray* logits, NDArray* labels, NDArray* output) {
  // 1 - labels - 1 / (1 + exp(logits))
  auto functor = LAMBDA_TT(x, y) {
    if (x <= 0) return static_cast<T>(1.) - y - static_cast<T>(1.) / (static_cast<T>(1.) + math::sd_exp<T, T>(x));
    auto e = math::sd_exp<T, T>(-x);
    return static_cast<T>(1.) - y - e / (static_cast<T>(1.) + e);
  });

  logits->applyPairwiseLambda(labels, functor, output);
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void sigmCrossEntropyGrad(LaunchContext* context, NDArray* logits, NDArray* labels, NDArray* output) {
  BUILD_SINGLE_SELECTOR(logits->dataType(), sigmCrossEntropyGrad_, (logits, labels, output), SD_FLOAT_TYPES);
}

template <typename T>
void softSignDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
  auto functor = LAMBDA_TT(x, y) {
    T ss = (T)1.f + math::sd_abs<T,T>(x);
    return y * ((T)1.0f / (ss * ss));
  });

  input->applyPairwiseLambda(epsilon, functor, output);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void softSignDerivative(LaunchContext* context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
  BUILD_SINGLE_SELECTOR(theFirst->dataType(), softSignDerivative_, (theFirst, theSecond, theOutput), SD_FLOAT_TYPES);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T>
void softPlusDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
  auto functor = LAMBDA_TT(x, y) {
    T p = math::sd_pow<T, T, T>(static_cast<T>(M_E), x);
    return y * (p / (p + 1.));
  });

  input->applyPairwiseLambda(epsilon, functor, output);
}

void softPlusDerivative(LaunchContext* context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
  BUILD_SINGLE_SELECTOR(theFirst->dataType(), softPlusDerivative_, (theFirst, theSecond, theOutput), SD_FLOAT_TYPES);
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///
/// \param input
/// \param epsilon
/// \param output
template <typename T>
void sigmoidDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
  auto functor = LAMBDA_TT(x, y) {
    T s = math::sd_sigmoid<T, T>(x);
    return y * (s * ((T)1.0f - s));
  });

  input->applyPairwiseLambda(epsilon, functor, output);
}

void sigmoidDerivative(LaunchContext* context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
  BUILD_SINGLE_SELECTOR(theFirst->dataType(), sigmoidDerivative_, (theFirst, theSecond, theOutput), SD_FLOAT_TYPES);
}

template <typename T>
void hardSigmoidDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
  auto functor = LAMBDA_TT(x, y) { return y * simdOps::HardSigmoidDerivative<T>::op(x, nullptr); });

  input->applyPairwiseLambda(epsilon, functor, output);
}

void hardSigmoidDerivative(LaunchContext* context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
  BUILD_SINGLE_SELECTOR(theFirst->dataType(), hardSigmoidDerivative_, (theFirst, theSecond, theOutput), SD_FLOAT_TYPES);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T>
void logSumExp_(NDArray* input, NDArray* axis, NDArray* output) {
  // reduce along axis with
  NDArray tempInput = input->dup();
  input->applyTransform(transform::Exp, &tempInput);
  std::vector<LongType> axisVector;
  if (axis != nullptr) {
    axisVector.resize(axis->lengthOf());
    for (size_t i = 0; i < axisVector.size(); ++i) axisVector[i] = axis->e<int>(i);
  }
  tempInput.reduceAlongDimension(reduce::Sum, output, &axisVector);
  output->applyTransform(transform::Log, output);
}

template <typename T>
void logSumExp_(NDArray* input, NDArray* subtrah, NDArray* axis, NDArray* output) {
  // reduce along axis with
  NDArray tempInput = input->dup();
  input->applyPairwiseTransform(pairwise::Subtract, subtrah, &tempInput);
  tempInput.applyTransform(transform::Exp, &tempInput);

  std::vector<LongType> axisVector;
  if (axis != nullptr) {
    axisVector.resize(axis->lengthOf());
    for (size_t i = 0; i < axisVector.size(); ++i) axisVector[i] = axis->e<int>(i);
  }
  tempInput.reduceAlongDimension(reduce::Sum, output, &axisVector);
  output->applyTransform(transform::Log, output);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void logSumExp(LaunchContext* context, NDArray* input, NDArray* axis, NDArray* output) {
  BUILD_SINGLE_SELECTOR(input->dataType(), logSumExp_, (input, axis, output), SD_FLOAT_TYPES);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void logSumExp(LaunchContext* context, NDArray* input, NDArray* subtrah, NDArray* axis, NDArray* output) {
  BUILD_SINGLE_SELECTOR(input->dataType(), logSumExp_, (input, subtrah, axis, output), SD_FLOAT_TYPES);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T>
void weightedCrossEntropyWithLogitsFunctor_(NDArray * targets, NDArray * input, NDArray * weights,
                                            NDArray* output) {
  T posWeight = weights->e<T>(0);

  auto mainRoutineT1 = LAMBDA_TT(_x, _z, posWeight) {
    T targetWeight = (1. + (posWeight - (T)1.f) * _z);
    return (1. - _z) * _x +
           targetWeight * (math::sd_log<T, T>((T)1.f + math::sd_exp<T, T>(-math::sd_abs<T,T>(_x))) +
                                            math::sd_max(-_x, T(0.f)));
  });

  auto mainRoutineT2 = LAMBDA_TTT(_x, _z, _w) {
    return (((T)1.0 - _z) * _x) + _w * (math::sd_log<T, T>(T(1.) + math::sd_exp<T, T>(-math::sd_abs<T,T>(_x))) + math::sd_max(-_x, T(0.f)));
  });

  if (weights->isScalar()) {
    input->applyPairwiseLambda(targets, mainRoutineT1, output);
  } else {
    std::unique_ptr<NDArray> targetVector(new NDArray(*weights));
    targetVector->applyScalar(scalar::Add, -1.f, targetVector.get());

    *targets = (*targetVector * *targets) + T(1.f);
    input->applyPairwiseLambda(targets, mainRoutineT1, output);

  }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void weightedCrossEntropyWithLogitsFunctor(LaunchContext* context, NDArray * targets, NDArray * input,
                                           NDArray * weights, NDArray* output) {
  NDArray::prepareSpecialUse({output}, {targets, input, weights});

  BUILD_SINGLE_SELECTOR(targets->dataType(), weightedCrossEntropyWithLogitsFunctor_, (targets, input, weights, output),
                        SD_FLOAT_TYPES);

  NDArray::registerSpecialUse({output}, {targets, input, weights});
}



}  // namespace helpers
}  // namespace ops
}  // namespace sd
