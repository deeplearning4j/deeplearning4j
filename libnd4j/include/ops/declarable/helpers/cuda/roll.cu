#include "hip/hip_runtime.h"
/* ******************************************************************************
*
*
* This program and the accompanying materials are made available under the
* terms of the Apache License, Version 2.0 which is available at
* https://www.apache.org/licenses/LICENSE-2.0.
*
*  See the NOTICE file distributed with this work for additional
*  information regarding copyright ownership.
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
* WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
* License for the specific language governing permissions and limitations
* under the License.
*
* SPDX-License-Identifier: Apache-2.0
******************************************************************************/

//
//  @author raver119@gmail.com
//
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <ops/declarable/helpers/roll.h>

#include "execution/cuda/LaunchDims.h"


namespace sd {
namespace ops {
namespace helpers {

template <typename T>
static void SD_DEVICE rollKernelLinearStage1Dev(const void *vx, const LongType *xShapeInfo, void *vz,
                                                const LongType *zShapeInfo, LongType fullLength,
                                                int actualShift) {
  auto x = reinterpret_cast<const T *>(vx);
  auto z = reinterpret_cast<T *>(vz);

  // Cache shape information for x buffer
  __shared__ sd::LongType xRank;
  __shared__ const sd::LongType* xShapePtr;
  __shared__ const sd::LongType* xStridePtr;

  // Cache shape information for z buffer
  __shared__ sd::LongType zRank;
  __shared__ const sd::LongType* zShapePtr;
  __shared__ const sd::LongType* zStridePtr;

  if (threadIdx.x == 0) {
    // Cache x shape information
    xRank = shape::rank(xShapeInfo);
    xShapePtr = shape::shapeOf(xShapeInfo);
    xStridePtr = shape::stride(xShapeInfo);

    // Cache z shape information
    zRank = shape::rank(zShapeInfo);
    zShapePtr = shape::shapeOf(zShapeInfo);
    zStridePtr = shape::stride(zShapeInfo);
  }
  __syncthreads();

  auto tid = threadIdx.x + blockIdx.x * blockDim.x;

  LongType xCoords[SD_MAX_RANK];
  LongType zCoords[SD_MAX_RANK];
  LongType xOffsetA;
  LongType xOffsetB;
  LongType zOffsetA;
  LongType zOffsetB;

  for (LongType i = tid; i < actualShift; i += blockDim.x * gridDim.x) {
    int sourceIndex = fullLength - actualShift + i;

    INDEX2COORDS(i, xRank, xShapePtr, xCoords);
    COORDS2INDEX(xRank, xStridePtr, xCoords, xOffsetA);
    INDEX2COORDS(sourceIndex, xRank, xShapePtr, xCoords);
    COORDS2INDEX(xRank, xStridePtr, xCoords, xOffsetB);

    INDEX2COORDS(i, zRank, zShapePtr, zCoords);
    COORDS2INDEX(zRank, zStridePtr, zCoords, zOffsetA);
    INDEX2COORDS(sourceIndex, zRank, zShapePtr, zCoords);
    COORDS2INDEX(zRank, zStridePtr, zCoords, zOffsetB);

    auto eA = x[xOffsetA];
    auto eB = x[xOffsetB];

    z[zOffsetA] = eB;
    z[zOffsetB] = eA;
  }
}
template <typename T>
static void SD_KERNEL rollKernelLinearStage1(const void *vx, const LongType *xShapeInfo, void *vz,
                                            const LongType *zShapeInfo, LongType fullLength, int actualShift) {
 rollKernelLinearStage1Dev<T>(vx, xShapeInfo, vz, zShapeInfo, fullLength, actualShift);
}

template <typename T>
static void SD_KERNEL rollKernelLinearStage2(const void *vx, const LongType *xShapeInfo, void *vz,
                                             const LongType *zShapeInfo, LongType fullLength, int actualShift,
                                             int shiftCount) {
 auto x = reinterpret_cast<const T *>(vx);
 auto z = reinterpret_cast<T *>(vz);

 // Cache shape information for x buffer
 __shared__ sd::LongType xRank;
 __shared__ const sd::LongType* xShapePtr;
 __shared__ const sd::LongType* xStridePtr;

 // Cache shape information for z buffer
 __shared__ sd::LongType zRank;
 __shared__ const sd::LongType* zShapePtr;
 __shared__ const sd::LongType* zStridePtr;

 if (threadIdx.x == 0) {
   // Cache x shape information
   xRank = shape::rank(xShapeInfo);
   xShapePtr = shape::shapeOf(xShapeInfo);
   xStridePtr = shape::stride(xShapeInfo);

   // Cache z shape information
   zRank = shape::rank(zShapeInfo);
   zShapePtr = shape::shapeOf(zShapeInfo);
   zStridePtr = shape::stride(zShapeInfo);
 }
 __syncthreads();

 auto tid = threadIdx.x + blockIdx.x * blockDim.x;

 LongType xCoords[SD_MAX_RANK];
 LongType zCoords[SD_MAX_RANK];
 LongType xOffsetA;
 LongType xOffsetB;
 LongType zOffsetA;
 LongType zOffsetB;

 for (int count = 1; count < shiftCount; ++count) {
   for (int i = tid; i < actualShift; i += blockDim.x * gridDim.x) {
     int destinationIndex = fullLength - (count + 1) * actualShift + i;
     int sourceIndex = fullLength - count * actualShift + i;

     INDEX2COORDS(destinationIndex, xRank, xShapePtr, xCoords);
     COORDS2INDEX(xRank, xStridePtr, xCoords, xOffsetA);
     INDEX2COORDS(sourceIndex, xRank, xShapePtr, xCoords);
     COORDS2INDEX(xRank, xStridePtr, xCoords, xOffsetB);

     INDEX2COORDS(destinationIndex, zRank, zShapePtr, zCoords);
     COORDS2INDEX(zRank, zStridePtr, zCoords, zOffsetA);
     INDEX2COORDS(sourceIndex, zRank, zShapePtr, zCoords);
     COORDS2INDEX(zRank, zStridePtr, zCoords, zOffsetB);

     auto eA = x[xOffsetB];
     auto eB = x[xOffsetA];

     z[zOffsetA] = eA;
     z[zOffsetB] = eB;
   }

   __syncthreads();
 }
}
template <typename T>
static void SD_KERNEL rollKernelLinearStage3(const void *vx, const LongType *xShapeInfo, void *vz,
                                             const LongType *zShapeInfo, LongType fullLength, int actualShift,
                                             int remainShift) {
 auto x = reinterpret_cast<const T *>(vx);
 auto z = reinterpret_cast<T *>(vz);

 // Cache shape information for x buffer
 __shared__ sd::LongType xRank;
 __shared__ const sd::LongType* xShapePtr;
 __shared__ const sd::LongType* xStridePtr;

 // Cache shape information for z buffer
 __shared__ sd::LongType zRank;
 __shared__ const sd::LongType* zShapePtr;
 __shared__ const sd::LongType* zStridePtr;

 if (threadIdx.x == 0) {
   // Cache x shape information
   xRank = shape::rank(xShapeInfo);
   xShapePtr = shape::shapeOf(xShapeInfo);
   xStridePtr = shape::stride(xShapeInfo);

   // Cache z shape information
   zRank = shape::rank(zShapeInfo);
   zShapePtr = shape::shapeOf(zShapeInfo);
   zStridePtr = shape::stride(zShapeInfo);
 }
 __syncthreads();

 auto tid = threadIdx.x + blockIdx.x * blockDim.x;

 for (int i = tid; i < actualShift; i += blockDim.x * gridDim.x) {
   int remainIdx = i + actualShift;
   int sourceIndex = remainIdx + remainShift;

   LongType xCoordsA[SD_MAX_RANK];
   LongType xCoordsB[SD_MAX_RANK];
   LongType zCoordsA[SD_MAX_RANK];
   LongType zCoordsB[SD_MAX_RANK];
   LongType xOffsetA;
   LongType xOffsetB;
   LongType zOffsetA;
   LongType zOffsetB;

   INDEX2COORDS(remainIdx, xRank, xShapePtr, xCoordsA);
   COORDS2INDEX(xRank, xStridePtr, xCoordsA, xOffsetA);
   INDEX2COORDS(sourceIndex, xRank, xShapePtr, xCoordsB);
   COORDS2INDEX(xRank, xStridePtr, xCoordsB, xOffsetB);

   INDEX2COORDS(remainIdx, zRank, zShapePtr, zCoordsA);
   COORDS2INDEX(zRank, zStridePtr, zCoordsA, zOffsetA);
   INDEX2COORDS(sourceIndex, zRank, zShapePtr, zCoordsB);
   COORDS2INDEX(zRank, zStridePtr, zCoordsB, zOffsetB);

   auto eA = x[xOffsetA];
   auto eB = x[xOffsetB];

   z[zOffsetA] = eB;
   z[zOffsetB] = eA;
 }
}
template <typename T>
static void SD_DEVICE swapTadsKernel(void *vx, void *vz, const LongType *zShapeInfo, LongType tadLength) {
 auto x = reinterpret_cast<T *>(vx);
 auto z = reinterpret_cast<T *>(vz);

 // Cache shape information for z buffer
 __shared__ sd::LongType zRank;
 __shared__ const sd::LongType* zShapePtr;
 __shared__ const sd::LongType* zStridePtr;

 if (threadIdx.x == 0) {
   // Cache z shape information
   zRank = shape::rank(zShapeInfo);
   zShapePtr = shape::shapeOf(zShapeInfo);
   zStridePtr = shape::stride(zShapeInfo);
 }
 __syncthreads();

 auto tid = threadIdx.x + blockIdx.x * blockDim.x;

 for (int e = threadIdx.x; e < tadLength; e += blockDim.x) {
   LongType zCoords[SD_MAX_RANK];
   LongType zOffset;

   INDEX2COORDS(e, zRank, zShapePtr, zCoords);
   COORDS2INDEX(zRank, zStridePtr, zCoords, zOffset);

   auto eA = x[zOffset];
   auto eB = z[zOffset];

   x[zOffset] = eB;
   z[zOffset] = eA;
 }
}
template <typename T>
static void SD_KERNEL rollKernelFullAnyDimensionStage1(const void *vx, const LongType *xTadShapeInfo,
                                                      const LongType *xTadOffsets, void *vz,
                                                      const LongType *zTadShapeInfo,
                                                      const LongType *zTadOffsets, int numTads, LongType tadLength, int dim, LongType sizeAt,
                                                      int theShift) {
 auto x = reinterpret_cast<const T *>(vx);
 auto z = reinterpret_cast<T *>(vz);

 for (int e = blockIdx.x + theShift; e < sizeAt - theShift; e += gridDim.x) {
   int sourceIndex = dim * sizeAt + e - theShift;
   int targetIndex = dim * sizeAt + e;

   swapTadsKernel<T>(z + xTadOffsets[sourceIndex], z + xTadOffsets[targetIndex], zTadShapeInfo, tadLength);
 }
}

template <typename T>
static void SD_KERNEL rollKernelFullAnyDimensionStage2(void *vx, const LongType *xTadShapeInfo,
                                                      const LongType *xTadOffsets, void *vz,
                                                      const LongType *zTadShapeInfo,
                                                      const LongType *zTadOffsets, int numTads, LongType tadLength, int dim, LongType sizeAt,
                                                      int theShift) {
 auto x = reinterpret_cast<const T *>(vx);
 auto z = reinterpret_cast<T *>(vz);

 for (int e = blockIdx.x; e < theShift; e += gridDim.x) {
   int sourceIndex = dim * sizeAt + sizeAt - theShift + e;
   int targetIndex = dim * sizeAt + e;

   swapTadsKernel<T>(z + zTadOffsets[sourceIndex], z + zTadOffsets[targetIndex], zTadShapeInfo, tadLength);
 }
}

template <typename T>
static void rollFunctorFull_(NDArray *input, NDArray *output, std::vector<LongType> const &shifts,
                            std::vector<LongType> const &axes, bool inplace) {
 if (!inplace) output->assign(input);

 for (size_t i = 0; i < axes.size(); i++) {
   int axe = axes[i];
   ResultSet listOfTensors = input->allTensorsAlongDimension({axe});
   ResultSet listOfOutTensors = output->allTensorsAlongDimension({axe});
   int fullLen = listOfTensors.size();
   int theShift = shifts[i];
   for (int k = 0; k < fullLen; k++) {
     rollFunctorLinear(output->getContext(), listOfTensors.at(k), listOfOutTensors.at(k), theShift, true);
   }
 }
}


template <typename T>
static void rollFunctorLinear_(NDArray *input, NDArray *output, int shift, bool inplace) {
 if (!inplace) output->assign(input);

 dim3 launchDims = getLaunchDims("roll");
 auto fullLen = input->lengthOf();
 int actualShift = shift;  // % fullLen; // shift already non-negative then
 if (actualShift < 0) {
   actualShift -= fullLen * (actualShift / fullLen - 1);
 } else
   actualShift %= fullLen;

 if (actualShift) {
   int shiftCount = fullLen / actualShift - 1;
   int remainShift = fullLen % actualShift;

   // stage 1) swap last actualShift elements with first ones.
   rollKernelLinearStage1<T><<<launchDims.y, launchDims.x, launchDims.z, *(output->getContext()->getCudaStream())>>>(
       output->specialBuffer(), output->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
       fullLen, actualShift);
   sd::DebugHelper::checkErrorCode(output->getContext()->getCudaStream(), "rollKernelLinearStage1 failed");

   // stage 2) swap swapped actualShift elements with rest remainShiftCount times.
   rollKernelLinearStage2<T><<<launchDims.y, launchDims.x, launchDims.z, *(output->getContext()->getCudaStream())>>>(
       output->specialBuffer(), output->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
       fullLen, actualShift, shiftCount);
   sd::DebugHelper::checkErrorCode(output->getContext()->getCudaStream(), "rollKernelLinearStage2 failed");
   // FIXME: no parallelism here :(
   // stage 3) swap remainer of items.
   if (remainShift && shiftCount)
     rollKernelLinearStage3<T><<<launchDims.y,launchDims.x,launchDims.z, *(output->getContext()->getCudaStream())>>>(
         output->specialBuffer(), output->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
         fullLen, actualShift, remainShift);
   sd::DebugHelper::checkErrorCode(output->getContext()->getCudaStream(), "rollKernelLinearStage3 failed");

 }
}

void rollFunctorFull(LaunchContext *context, NDArray *input, NDArray *output, std::vector<LongType> const &shifts,
                    std::vector<LongType> const &axes, bool inplace) {
 input->syncToDevice();

 BUILD_SINGLE_SELECTOR(input->dataType(), rollFunctorFull_, (input, output, shifts, axes, inplace), SD_COMMON_TYPES);

 output->tickWriteDevice();
}

void rollFunctorLinear(LaunchContext *context, NDArray *input, NDArray *output, int shift, bool inplace) {
 input->syncToDevice();

 BUILD_SINGLE_SELECTOR(input->dataType(), rollFunctorLinear_, (input, output, shift, inplace), SD_COMMON_TYPES);

 output->tickWriteDevice();
}

BUILD_SINGLE_TEMPLATE(template void rollFunctorLinear_, (NDArray * input, NDArray *output, int shift, bool inplace),
                     SD_COMMON_TYPES);
BUILD_SINGLE_TEMPLATE(template void rollFunctorFull_,
                     (NDArray * input, NDArray *output, std::vector<sd::LongType> const &shifts, std::vector<sd::LongType> const &axes,
                      bool inplace),
                     SD_COMMON_TYPES);
}  // namespace helpers
}  // namespace ops
}  // namespace sd
