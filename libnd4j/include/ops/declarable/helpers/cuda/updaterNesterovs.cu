#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <system/op_boilerplate.h>

#include "execution/cuda/LaunchDims.h"
#include "helpers/DebugHelper.h"


namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void nesterovsUpdaterCuda(const void* vx, const LongType* xShapeInfo, const void* vin,
                                    const LongType* inShapeInfo, void* vz, const LongType* zShapeInfo,
                                    void* vst, const LongType* stShapeInfo, const T lr, const T momentum) {
  const auto grad = reinterpret_cast<const T*>(vx);
  const auto init = reinterpret_cast<const T*>(vin);
  auto up = reinterpret_cast<T*>(vz);
  auto st = reinterpret_cast<T*>(vst);

  __shared__ LongType xLen, xRank, zRank, inRank, stRank;
  __shared__ T momentumT;
  __shared__ bool bOrdering, bXZsame, bXInSame, bXStSame;
  __shared__ LongType *sharedMem;
  __shared__ const LongType *xShape, *zShape, *inShape, *stShape;
  __shared__ const LongType *xStride, *zStride, *inStride, *stStride;

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char shmem[];
    sharedMem = reinterpret_cast<LongType*>(shmem);

    xLen = shape::length(xShapeInfo);
    momentumT = (-momentum - 1);

    xRank = shape::rank(xShapeInfo);
    zRank = shape::rank(zShapeInfo);
    inRank = shape::rank(inShapeInfo);
    stRank = shape::rank(stShapeInfo);

    xShape = shape::shapeOf(xShapeInfo);
    xStride = shape::stride(xShapeInfo);
    zShape = shape::shapeOf(zShapeInfo);
    zStride = shape::stride(zShapeInfo);
    inShape = shape::shapeOf(inShapeInfo);
    inStride = shape::stride(inShapeInfo);
    stShape = shape::shapeOf(stShapeInfo);
    stStride = shape::stride(stShapeInfo);

    bOrdering = shape::order(xShapeInfo) == shape::order(zShapeInfo) &&
                shape::order(xShapeInfo) == shape::order(inShapeInfo) &&
                shape::order(xShapeInfo) == shape::order(stShapeInfo);

    bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
    bXInSame = shape::haveSameShapeAndStrides(xShapeInfo, inShapeInfo);
    bXStSame = shape::haveSameShapeAndStrides(xShapeInfo, stShapeInfo);
  }
  __syncthreads();

  LongType coords[SD_MAX_RANK];

  for (LongType i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {
    LongType xOffset, zOffset, initOffset, stOffset;

    INDEX2COORDS(i, xRank, xShape, coords);
    COORDS2INDEX(xRank, xStride, coords, xOffset);

    if (bXZsame) {
      zOffset = xOffset;
    } else {
      COORDS2INDEX(zRank, zStride, coords, zOffset);
    }

    if (bXInSame) {
      initOffset = xOffset;
    } else {
      COORDS2INDEX(inRank, inStride, coords, initOffset);
    }

    if (bXStSame) {
      stOffset = xOffset;
    } else {
      COORDS2INDEX(stRank, stStride, coords, stOffset);
    }

    T prevState = momentum * init[initOffset];
    st[stOffset] = prevState - lr * grad[xOffset];
    up[zOffset] = prevState + momentumT * st[stOffset];
  }
}
///////////////////////////////////////////////////////////////////
template <typename T>
void nesterovsUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMemory,
                                  const hipStream_t* stream, const void* vx, const LongType* xShapeInfo,
                                  const void* vin, const LongType* inShapeInfo, void* vz,
                                  const LongType* zShapeInfo, void* vst, const LongType* stShapeInfo,
                                  const double dLr, const double dMomentum) {
  const T lr = static_cast<T>(dLr);
  const T momentum = static_cast<T>(dMomentum);
  nesterovsUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMemory, *stream>>>(vx, xShapeInfo, vin, inShapeInfo, vz,
                                                                            zShapeInfo, vst, stShapeInfo, lr, momentum);
  sd::DebugHelper::checkErrorCode(const_cast<hipStream_t *>(stream), "nesterovsUpdaterCuda failed");

}

///////////////////////////////////////////////////////////////////
void updaterNesterovs(LaunchContext* context, NDArray& gradient, NDArray& initState, NDArray& update,
                      NDArray& stateV, const double dLr, const double dMomentum) {
  PointersManager manager(context, "nesterovsUpdater");

  dim3 launchDims = updaterDims(gradient.lengthOf());
  NDArray::prepareSpecialUse({&update, &stateV}, {&gradient, &initState});
  BUILD_SINGLE_SELECTOR(
      gradient.dataType(), nesterovsUpdaterCudaLauncher,
      (launchDims.y, launchDims.x,launchDims.z, context->getCudaStream(), gradient.specialBuffer(), gradient.specialShapeInfo(),
       initState.specialBuffer(), initState.specialShapeInfo(), update.specialBuffer(), update.specialShapeInfo(),
       stateV.specialBuffer(), stateV.specialShapeInfo(), dLr, dMomentum),
      SD_FLOAT_TYPES);
  NDArray::registerSpecialUse({&update, &stateV}, {&gradient, &initState});

  manager.synchronize();
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
