/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ShapeUtils.h>
#include <helpers/svd.h>
#include <system/op_boilerplate.h>


namespace sd {
namespace ops {
namespace helpers {

// FIXME -> we should optimize these helpers for the case when input matrices have c order (perform transpositions
// appropriately)

//////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////
static void svdQR(LaunchContext* context, NDArray* A, NDArray* S, NDArray* U, NDArray* VT, const bool fullUV,
                  const bool calcUV) {
  // since cusa api hipsolverDnDgesvd/hipsolverDnSgesvd have following constrain on input matrix A: A_rows >= A_columns &&
  // A_order = 'f' we make this function to have deal with 2 valid cases only: 1) A_rows >= A_columns and A_corder = 'f'
  // 2) A_rows <= A_columns and A_corder = 'c'    - int this case perform transposition to get f order
  // if 1) or 2) are not met then throw exception

  // A  [m, n]
  // S  [n]
  // U  [m, m] or [m, n] if fullUV = false and m > n
  // VT [n, n] or [m, n] if fullUV = false and m < n

  if (A->rankOf() != 2) THROW_EXCEPTION("svdQR: rank of A array is not equal 2 !");

  auto m = A->sizeAt(0);
  auto n = A->sizeAt(1);
  const int minDim = m < n ? m : n;
  const char orderA = A->ordering();

  if (m < n) THROW_EXCEPTION("svdQR: due to cuda api input constrains given shape of A array are not valid !");

  if (std::vector<LongType>({minDim}) != S->getShapeAsVector())
    THROW_EXCEPTION("svdQR: wrong shape of S array !");

  if (calcUV) {
    if (fullUV && std::vector<LongType>({m, m}) != U->getShapeAsVector()) {
      THROW_EXCEPTION("svdQR: wrong shape of U array !");
    } else if (!fullUV && std::vector<LongType>({m, minDim}) != U->getShapeAsVector()) {
      THROW_EXCEPTION("svdQR: wrong shape of U array !");
    }

    if (fullUV && std::vector<LongType>({n, n}) != VT->getShapeAsVector()) {
      THROW_EXCEPTION("svdQR: wrong shape of VT array !");
    }
    else if (!fullUV && std::vector<LongType>({minDim, n}) != VT->getShapeAsVector()) {
      THROW_EXCEPTION("svdQR: wrong shape of VT array !");
    }
  }

  NDArray* pA = const_cast<NDArray*>(A);
  NDArray* pS = S;
  NDArray* pU = U;
  NDArray* pVT = VT;

  std::vector<NDArray*> toDelete;

  if (pA->ordering() == 'c') {
    pA = new NDArray(A->dup('f'));
    toDelete.push_back(pA);
  }

    pS = new NDArray(S->dup('f'));
    toDelete.push_back(pS);


  if (calcUV) {
    if (pU->ordering() == 'c') {
      pU = new NDArray(U->dup('f'));
      toDelete.push_back(pU);
    }

    if (pVT->ordering() == 'c') {
      pVT = new NDArray(VT->dup('f'));
      toDelete.push_back(pVT);
    }
  }

  std::lock_guard<std::mutex> lock(*LaunchContext::deviceMutex());

  // create cusolverDn handle
  hipsolverHandle_t* handle = (hipsolverHandle_t*)context->getCusolverHandle();  // nullptr;
  if (handle == nullptr) throw cuda_exception::build("svdQR: cuda failed !", -1);

  // stream
  auto status = hipsolverSetStream(*handle, *context->getCudaStream());
  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdQR: cuda failed !", status);

  // query working space of SVD
  int lwork = 0;
  if (A->dataType() == DOUBLE)
    status = hipsolverDnDgesvd_bufferSize(*handle, m, n, &lwork);
  else if (A->dataType() == FLOAT32)
    status = hipsolverDnSgesvd_bufferSize(*handle, m, n, &lwork);
  else
    THROW_EXCEPTION("svdQR: given data type is unsupported !");

  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdQR: cuda failed !", status);

  // allocate memory for dWork
  void* dWork = nullptr;
  hipError_t status2 = hipMalloc((void**)&dWork, A->sizeOfT() * lwork);
  if (status2 != hipSuccess) throw cuda_exception::build("svdQR: cuda failed !", status2);

  signed char jobu, jobvt;

  if (calcUV) {
    if (fullUV)
      jobu = jobvt = 'A';
    else
      jobu = jobvt = 'S';
  } else {
    jobu = jobvt = 'N';
  }

  int* devInfo = nullptr;
  void* rWork = nullptr;

  int lda(m), ldu, ldvt;

  if (calcUV) {
    ldu = pU->sizeAt(0);
    ldvt = pVT->sizeAt(0);
  }

  PointersManager manager(context, "svdQR");

  NDArray::prepareSpecialUse({pS, pU, pVT}, {pA});

  // choose appropriate cuda gemm api depending on data types
  if (A->dataType() == DOUBLE) {
    status = hipsolverDnDgesvd(*handle, jobu, jobvt, m, n, reinterpret_cast<double*>(pA->specialBuffer()), lda,
                              reinterpret_cast<double*>(pS->specialBuffer()),
                              calcUV ? reinterpret_cast<double*>(pU->specialBuffer()) : nullptr, ldu,
                              calcUV ? reinterpret_cast<double*>(pVT->specialBuffer()) : nullptr, ldvt,
                              reinterpret_cast<double*>(dWork), lwork, reinterpret_cast<double*>(rWork), devInfo);
  } else if (A->dataType() == FLOAT32) {
    status = hipsolverDnSgesvd(*handle, jobu, jobvt, m, n, reinterpret_cast<float*>(pA->specialBuffer()), lda,
                              reinterpret_cast<float*>(pS->specialBuffer()),
                              calcUV ? reinterpret_cast<float*>(pU->specialBuffer()) : nullptr, ldu,
                              calcUV ? reinterpret_cast<float*>(pVT->specialBuffer()) : nullptr, ldvt,
                              reinterpret_cast<float*>(dWork), lwork, reinterpret_cast<float*>(rWork), devInfo);
  } else
    THROW_EXCEPTION("svdQR: given data type is unsupported !");

  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdQR: cuda failed !", status);

  manager.synchronize();

  NDArray::registerSpecialUse({pS, pU, pVT}, {pA});

  S->assign(pS);

  if (calcUV) {
    U->assign(pU);
    VT->assign(pVT);
  }

  //for (int i = toDelete.size() - 1; i >= 0; --i) delete toDelete[i];

  // if (devInfo) hipFree(devInfo);
  // if (dWork) hipFree(dWork);
  // if (rWork) hipFree(rWork);

}

//////////////////////////////////////////////////////////////////////////
static void svdJcb(LaunchContext* context, NDArray* A, NDArray* S, NDArray* U, NDArray* V, const bool fullUV,
                   const bool calcUV) {
  // A [m, n]
  // S [n]
  // U [m, m] or [m, n] if fullUV = false and m > n
  // V [n, n] or [n, m] if fullUV = false and m < n

  if (A->rankOf() != 2) THROW_EXCEPTION("svdJcb: rank of A array is not equal 2 !");

  int m = A->sizeAt(0);
  int n = A->sizeAt(1);
  const int minDim = m < n ? m : n;

  if (std::vector<LongType>({minDim}) != S->getShapeAsVector()) THROW_EXCEPTION("svdJcb: wrong shape of S array !");

  if (fullUV && U != nullptr && std::vector<LongType>({m, m}) != U->getShapeAsVector()) {
    THROW_EXCEPTION("svdJcb: wrong shape of U array !");
  } else if (!fullUV && U != nullptr && std::vector<LongType>({m, minDim}) != U->getShapeAsVector()) {
    THROW_EXCEPTION("svdJcb: wrong shape of U array !");
  }
  if (fullUV && V != nullptr && std::vector<LongType>({n, n}) != V->getShapeAsVector()) {
    THROW_EXCEPTION("svdJcb: wrong shape of V array !");
  } else if (!fullUV && V != nullptr && std::vector<LongType>({n, minDim}) != V->getShapeAsVector()) {
    THROW_EXCEPTION("svdJcb: wrong shape of V array !");
  }

  NDArray* pA = const_cast<NDArray*>(A);

  const bool aForder = m == 1 || A->strideAt(0) == 1;
  const bool aCorder = n == 1 || A->strideAt(1) == 1;

  const bool transA = !aForder && aCorder;
  const bool dupA = !aForder && !aCorder;

  std::vector<NDArray*> toDelete;

  if (dupA) {
    pA = new NDArray(A->dup('f'));
    toDelete.push_back(pA);
  }

  NDArray* pS = S;

    pS = new NDArray(S->dup('f'));
    toDelete.push_back(pS);


  NDArray *pU(nullptr), *pV(nullptr);

  int lda = transA ? pA->strideAt(0) : pA->strideAt(1);
  int ldu(transA ? n : m), ldv(transA ? m : n);
  bool uForder(true), vForder(true);


  if (calcUV) {
    pU = transA ? V : U;
    pV = transA ? U : V;

    uForder = pU->sizeAt(0) == 1 || pU->strideAt(0) == 1;
    vForder = pV->sizeAt(0) == 1 || pV->strideAt(0) == 1;

    if (!uForder) {
      pU = new NDArray(pU->dup('f'));
      toDelete.push_back(pU);
    }

    if (!vForder) {
      pV = new NDArray(pV->dup('f'));
      toDelete.push_back(pV);
    }

    ldu = pU->strideAt(1);
    ldv = pV->strideAt(1);
  }

  std::lock_guard<std::mutex> lock(*LaunchContext::deviceMutex());

  // create cusolverDn handle
  hipsolverHandle_t* handle = (hipsolverHandle_t*)context->getCusolverHandle();
  if (handle == nullptr) throw cuda_exception::build("svdJcb: cuda failed !", -1);

  // stream
  auto status = hipsolverSetStream(*handle, *context->getCudaStream());
  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdJcb: cuda failed !", status);

  // set parameters
  hipsolverGesvdjInfo_t gesvdjParams = nullptr;
  status = hipsolverDnCreateGesvdjInfo(&gesvdjParams);
  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdJcb: cuda failed !", status);
  status = hipsolverDnXgesvdjSetTolerance(gesvdjParams, 1.e-7);  // tolerance
  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdJcb: cuda failed !", status);
  status = hipsolverDnXgesvdjSetMaxSweeps(gesvdjParams, 15);  // max_sweeps
  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdJcb: cuda failed !", status);

  int* devInfo = nullptr;
  const hipsolverEigMode_t jobz = calcUV ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;
  const int econ = !fullUV;

  if (transA) math::sd_swap<int>(m, n);

  // *** avoid bug in cuda API ***
  void* nullPtr = nullptr;
  NDArray* arrToAvoidBugInAPI = nullptr;
  if (!calcUV && m != n) {
    int maxDim = m > n ? m : n;
    std::vector<LongType> shape = {maxDim, maxDim};
    arrToAvoidBugInAPI = new NDArray('c', shape, pA->dataType(), context);
    nullPtr = arrToAvoidBugInAPI->specialBuffer();
  }
  // ******************

  NDArray::prepareSpecialUse({pS, pU, pV}, {pA});

  // query working space of SVD
  int lwork = 0;
  if (A->dataType() == DOUBLE)
    status = hipsolverDnDgesvdj_bufferSize(
        *handle, jobz, econ, m, n, reinterpret_cast<double*>(pA->specialBuffer()), lda,
        reinterpret_cast<double*>(pS->specialBuffer()),
        calcUV ? reinterpret_cast<double*>(pU->specialBuffer()) : reinterpret_cast<double*>(nullPtr), ldu,
        calcUV ? reinterpret_cast<double*>(pV->specialBuffer()) : reinterpret_cast<double*>(nullPtr), ldv, &lwork,
        gesvdjParams);
  else if (A->dataType() == FLOAT32)
    status = hipsolverDnSgesvdj_bufferSize(
        *handle, jobz, econ, m, n, reinterpret_cast<float*>(pA->specialBuffer()), lda,
        reinterpret_cast<float*>(pS->specialBuffer()),
        calcUV ? reinterpret_cast<float*>(pU->specialBuffer()) : reinterpret_cast<float*>(nullPtr), ldu,
        calcUV ? reinterpret_cast<float*>(pV->specialBuffer()) : reinterpret_cast<float*>(nullPtr), ldv, &lwork,
        gesvdjParams);
  else
    THROW_EXCEPTION("svdJcb: given data type is unsupported !");

  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdJcb: cuda failed !", status);

  // allocate memory dWork
  void* dWork = nullptr;
  auto status2 = hipMalloc((void**)&dWork, A->sizeOfT() * lwork);
  if (status2 != hipSuccess) throw cuda_exception::build("svdJcb: cuda failed !", status2);

  PointersManager manager(context, "svdJcb");

  // choose appropriate cuda gemm api depending on data types
  if (A->dataType() == DOUBLE) {
    status = hipsolverDnDgesvdj(
        *handle, jobz, econ, m, n, reinterpret_cast<double*>(pA->specialBuffer()), lda,
        reinterpret_cast<double*>(pS->specialBuffer()),
        calcUV ? reinterpret_cast<double*>(pU->specialBuffer()) : reinterpret_cast<double*>(nullPtr), ldu,
        calcUV ? reinterpret_cast<double*>(pV->specialBuffer()) : reinterpret_cast<double*>(nullPtr), ldv,
        reinterpret_cast<double*>(dWork), lwork, devInfo, gesvdjParams);
  } else if (A->dataType() == FLOAT32) {
    status = hipsolverDnSgesvdj(
        *handle, jobz, econ, m, n, reinterpret_cast<float*>(pA->specialBuffer()), lda,
        reinterpret_cast<float*>(pS->specialBuffer()),
        calcUV ? reinterpret_cast<float*>(pU->specialBuffer()) : reinterpret_cast<float*>(nullPtr), ldu,
        calcUV ? reinterpret_cast<float*>(pV->specialBuffer()) : reinterpret_cast<float*>(nullPtr), ldv,
        reinterpret_cast<float*>(dWork), lwork, devInfo, gesvdjParams);
  } else
    THROW_EXCEPTION("svdJcb: given data type is unsupported !");

  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdJcb: cuda failed !", status);

  manager.synchronize();

  NDArray::registerSpecialUse({pS, pU, pV}, {pA});

   S->assign(pS);

  if (calcUV) {
    if (!uForder) U->assign(transA ? pV : pU);
    if (!vForder) V->assign(transA ? pU : pV);
  }

  if (!calcUV && m != n) delete arrToAvoidBugInAPI;

  for (int i = toDelete.size() - 1; i >= 0; --i) delete toDelete[i];

  if (devInfo) hipFree(devInfo);
  if (dWork) hipFree(dWork);
  if (gesvdjParams) hipsolverDnDestroyGesvdjInfo(gesvdjParams);


}

//////////////////////////////////////////////////////////////////////////
static void svdBatched(LaunchContext* context, NDArray* A, NDArray* S, NDArray* U, NDArray* V,
                       const bool fullUV, const bool calcUV) {
  // A [..., m, n]
  // S [..., n]
  // U [..., m, m] or [..., m, n] if fullUV = false and m > n
  // V [..., n, n] or [..., n, m] if fullUV = false and m < n

  auto m = A->sizeAt(-2);
  auto n = A->sizeAt(-1);
  const int minDim = m < n ? m : n;
  const LongType bS = A->lengthOf() / (m * n);

  if (m > 32 || n > 32) THROW_EXCEPTION("svdBatched: numbers of rows and columns should be <= 32 !");

  if (minDim != S->sizeAt(-1)) THROW_EXCEPTION("svdBatched: wrong shape of S array !");

  if (calcUV) {
    if (U->sizeAt(-2) != m) THROW_EXCEPTION("svdBatched: wrong shape of U array !");
    if (U->sizeAt(-1) != (fullUV ? m : minDim)) THROW_EXCEPTION("svdBatched: wrong shape of U array !");
    if (U->lengthOf() / (U->sizeAt(-2) * U->sizeAt(-1)) != bS)
      THROW_EXCEPTION("svdBatched: wrong shape of U array !");

    if (V->sizeAt(-2) != n) THROW_EXCEPTION("svdBatched: wrong shape of V array !");
    if (V->sizeAt(-1) != (fullUV ? n : minDim)) THROW_EXCEPTION("svdBatched: wrong shape of V array !");
    if (V->lengthOf() / (V->sizeAt(-2) * V->sizeAt(-1)) != bS)
      THROW_EXCEPTION("svdBatched: wrong shape of V array !");
  }

  NDArray* pA = const_cast<NDArray*>(A);
  NDArray* pS = S;
  NDArray* pU = U;
  NDArray* pV = V;

  std::vector<NDArray*> toDelete;

  if (pA->ordering() == 'c') {
    pA = new NDArray(A->dup('f'));
    toDelete.push_back(pA);
  }

    pS = new NDArray(S->dup('f'));
    toDelete.push_back(pS);


  if (calcUV) {
    if (pU->ordering() == 'c') {
      pU = new NDArray(U->dup('f'));
      toDelete.push_back(pU);
    }

    if (pV->ordering() == 'c') {
      pV = new NDArray(V->dup('f'));
      toDelete.push_back(pV);
    }
  }

  // create cusolverDn handle
  hipsolverHandle_t handle = nullptr;
  hipsolverStatus_t status = hipsolverDnCreate(&handle);
  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdBatched: cuda failed !", status);

  // stream
  status = hipsolverSetStream(handle, *context->getCudaStream());
  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdBatched: cuda failed !", status);

  // set parameters
  hipsolverGesvdjInfo_t gesvdjParams = nullptr;
  status = hipsolverDnCreateGesvdjInfo(&gesvdjParams);
  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdBatched: cuda failed !", status);
  status = hipsolverDnXgesvdjSetTolerance(gesvdjParams, 1.e-7);  // tolerance
  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdBatched: cuda failed !", status);
  status = hipsolverDnXgesvdjSetMaxSweeps(gesvdjParams, 15);  // max_sweeps
  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdBatched: cuda failed !", status);

  // devInfo
  int* devInfo = nullptr;
  auto status2 = hipMalloc((void**)&devInfo, sizeof(LongType) * bS);
  if (status2 != hipSuccess) throw cuda_exception::build("svdBatched: cuda failed !", status2);
  status2 = hipDeviceSynchronize();
  if (status2 != hipSuccess) throw cuda_exception::build("svdJcb: cuda failed !", status2);

  const hipsolverEigMode_t jobz = calcUV ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;

  int lda(m), ldu, ldv;

  if (calcUV) {
    ldu = pU->sizeAt(-2);
    ldv = pV->sizeAt(-2);
  }

  // Ak (i,j) = A[i + 5*j + 25*k]

  // query working space of SVD
  int lwork = 0;
  if (A->dataType() == DOUBLE)
    status = hipsolverDnDgesvdjBatched_bufferSize(handle, jobz, m, n, reinterpret_cast<double*>(pA->specialBuffer()),
                                                 lda, reinterpret_cast<double*>(pS->specialBuffer()),
                                                 calcUV ? reinterpret_cast<double*>(pU->specialBuffer()) : nullptr, ldu,
                                                 calcUV ? reinterpret_cast<double*>(pV->specialBuffer()) : nullptr, ldv,
                                                 &lwork, gesvdjParams, bS);
  else if (A->dataType() == FLOAT32)
    status = hipsolverDnSgesvdjBatched_bufferSize(
        handle, jobz, m, n, reinterpret_cast<float*>(pA->specialBuffer()), lda,
        reinterpret_cast<float*>(pS->specialBuffer()), calcUV ? reinterpret_cast<float*>(pU->specialBuffer()) : nullptr,
        ldu, calcUV ? reinterpret_cast<float*>(pV->specialBuffer()) : nullptr, ldv, &lwork, gesvdjParams, bS);
  else
    THROW_EXCEPTION("svdBatched: given data type is unsupported !");

  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdBatched: cuda failed !", status);

  // allocate memory dWork
  void* dWork = nullptr;
  status2 = hipMalloc((void**)&dWork, A->sizeOfT() * lwork);
  if (status2 != hipSuccess) throw cuda_exception::build("svdBatched: cuda failed !", status2);
  status2 = hipDeviceSynchronize();
  if (status2 != hipSuccess) throw cuda_exception::build("svdBatched: cuda failed !", status2);

  PointersManager manager(context, "svdBatched");

  NDArray::prepareSpecialUse({pS, pU, pV}, {pA});

  // choose appropriate cuda gemm api depending on data types
  if (A->dataType() == DOUBLE) {
    status = hipsolverDnDgesvdjBatched(handle, jobz, m, n, reinterpret_cast<double*>(pA->specialBuffer()), lda,
                                      reinterpret_cast<double*>(pS->specialBuffer()),
                                      calcUV ? reinterpret_cast<double*>(pU->specialBuffer()) : nullptr, ldu,
                                      calcUV ? reinterpret_cast<double*>(pV->specialBuffer()) : nullptr, ldv,
                                      reinterpret_cast<double*>(dWork), lwork, devInfo, gesvdjParams, bS);
  } else if (A->dataType() == FLOAT32) {
    status = hipsolverDnSgesvdjBatched(handle, jobz, m, n, reinterpret_cast<float*>(pA->specialBuffer()), lda,
                                      reinterpret_cast<float*>(pS->specialBuffer()),
                                      calcUV ? reinterpret_cast<float*>(pU->specialBuffer()) : nullptr, ldu,
                                      calcUV ? reinterpret_cast<float*>(pV->specialBuffer()) : nullptr, ldv,
                                      reinterpret_cast<float*>(dWork), lwork, devInfo, gesvdjParams, bS);
  } else
    THROW_EXCEPTION("svdBatched: given data type is unsupported !");

  if (status != HIPSOLVER_STATUS_SUCCESS) throw cuda_exception::build("svdBatched: cuda failed !", status);

  manager.synchronize();

  NDArray::registerSpecialUse({pS, pU, pV}, {pA});

  S->assign(pS);

  if (calcUV) {
    U->assign(pU);
    V->assign(pV);
  }

  for (int i = toDelete.size() - 1; i >= 0; --i) delete toDelete[i];

  if (devInfo) hipFree(devInfo);
  if (dWork) hipFree(dWork);
  if (handle) hipsolverDnDestroy(handle);
  if (gesvdjParams) hipsolverDnDestroyGesvdjInfo(gesvdjParams);

}

////////////////////////////////////////////////////////////////////
void svd(LaunchContext* context, NDArray* x, const std::vector<NDArray*>& outArrs, const bool fullUV,
         const bool calcUV, const int switchNum) {
  NDArray* S = outArrs[0];
  NDArray* U = outArrs[1];
  NDArray* V = outArrs[2];

  NDArray::prepareSpecialUse({S, U, V}, {x});

  if (x->rankOf() == 2) {
    svdJcb(context, x, S, U, V, fullUV, calcUV);
  } else {
    ResultSet *tadsU(nullptr), *tadsV(nullptr);

    auto tadsX = x->allTensorsAlongDimension({x->rankOf() - 2, x->rankOf() - 1});
    auto tadsS = S->allTensorsAlongDimension({S->rankOf() - 1});

    if (calcUV) {
      tadsU = new ResultSet(U->allTensorsAlongDimension({U->rankOf() - 2, U->rankOf() - 1}));
      tadsV = new ResultSet(V->allTensorsAlongDimension({V->rankOf() - 2, V->rankOf() - 1}));
    }

    for (int i = 0; i < tadsX.size(); ++i)
      svdJcb(context, tadsX.at(i), tadsS.at(i), calcUV ? tadsU->at(i) : nullptr, calcUV ? tadsV->at(i) : nullptr,
             fullUV, calcUV);

    if (calcUV) {
      delete tadsU;
      delete tadsV;
    }
  }

  NDArray::registerSpecialUse({S, U, V}, {x});
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
