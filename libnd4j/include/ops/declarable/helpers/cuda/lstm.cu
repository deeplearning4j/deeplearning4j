#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma, created on 14.02.2018
//

// implementation of operation for LSTM cell with peep hole connections:
// http://www.bioinf.jku.at/publications/older/2604.pdf
// S. Hochreiter and J. Schmidhuber. "Long Short-Term Memory". Neural Computation, 9(8):1735-1780, 1997.
// and
// https://research.google.com/pubs/archive/43905.pdf
// Hasim Sak, Andrew Senior, and Francoise Beaufays. "Long short-term memory recurrent neural network architectures for
// large scale acoustic modeling." INTERSPEECH, 2014.

#include <array/NDArrayList.h>
#include <helpers/PointersManager.h>
#include <ops/declarable/CustomOperations.h>
#include <ops/declarable/helpers/lstm.h>
#include <ops/declarable/helpers/lstmBlock.h>
#include <ops/declarable/helpers/transforms.h>

#include <iterator>


namespace sd {
namespace ops {
namespace helpers {

//////////////////////////////////////////////////////////////////////////
void lstmCell(LaunchContext* context, NDArray* xt, NDArray* ht_1, NDArray* ct_1,
              NDArray* Wx, NDArray* Wh, NDArray* Wc, NDArray* Wp, NDArray* b, NDArray* ht,
              NDArray* ct, const std::vector<double>& params) {
  // xt   input [bS x nIn]
  // ht_1 previous cell output [bS x numProj],  that is at previous time step t-1, in case of projection=false ->
  // numProj=nOut!!! ct_1 previous cell state  [bS x nOut], that is at previous time step t-1

  // Wx   input-to-hidden  weights, [nIn  x 4*nOut]
  // Wh   hidden-to-hidden weights, [numProj x 4*nOut]
  // Wc   diagonal weights for peephole connections [3*nOut]
  // Wp   projection weights [nOut x numProj]
  // b    biases, [4*nOut]

  // ht  current cell output [bS x numProj], that is at current time step t
  // ct  current cell state  [bS x nOut], that is at current time step t

  const bool peephole = (bool)params[0];  // if true, provide peephole connections
  const bool projection =
      (bool)params[1];  // if true, then projection is performed, if false then numProj==nOut is mandatory!!!!
  double clippingCellValue =
      params[2];  // clipping value for ct, if it is not equal to zero, then cell state is clipped
  double clippingProjValue =
      params[3];  // clipping value for projected ht, if it is not equal to zero, then projected cell output is clipped
  const double forgetBias = params[4];

  const int bS = xt->sizeAt(0);
  const int nIn = xt->sizeAt(1);
  const int numProj = ht_1->sizeAt(1);
  const int nOut = ct_1->sizeAt(1);

  auto z = mmul(*xt, *Wx) + mmul(*ht_1, *Wh) + *b;  // [bS x 4*nOut] + [bS x 4*nOut] + [1 x 4*nOut] = [bS x 4*nOut]

  auto zit = z({0, 0, 0, nOut});             // z for input gate,  = mmul(Wxi,xt) + mmul(Whi,ht_1) + bi    = [bS x nOut]
  auto zft = z({0, 0, nOut, 2 * nOut});      // z for forget gate, = mmul(Wxf,xt) + mmul(Whf,ht_1) + bf    = [bS x nOut]
  auto zct = z({0, 0, 2 * nOut, 3 * nOut});  // z for cell state,  = mmul(Wxc,xt) + mmul(Whc,ht_1) + bc    = [bS x nOut]
  auto zot = z({0, 0, 3 * nOut, 4 * nOut});  // z for output gate, = mmul(Wxo,xt) + mmul(Who,ht_1) + bo    = [bS x nOut]

  if (peephole) {                              // add peephole connections: z  +  ct_1*Wc
    zit += (*ct_1) * (*Wc)({0, nOut});         // add peephole connections to input gate
    zft += (*ct_1) * (*Wc)({nOut, 2 * nOut});  // add peephole connections to forget gate
  }

  // current sell state = ft*ct_1 + it*tanh(mmul(Wxc,xt) + mmul(Whc,ht_1) + bc
  NDArray zftPlusForgetBias = zft + forgetBias;
  NDArray toAssign = sigmoid(zftPlusForgetBias) * (*ct_1) + sigmoid(zit) * tanh(zct);
  ct->assign(&toAssign);

  // if clipping value is provided then cell state is clipped by this value prior to the cell output activation
  if (clippingCellValue > 0.0) ct->applyScalar(scalar::LstmClip, clippingCellValue, ct);

  if (peephole) zot += (*ct) * (*Wc)({{2 * nOut, 3 * nOut}});  // add peephole connections to output gate zot + ct*Wc

  // current cell output = ot*tanh(ct)
  auto htNoPeepHole = sigmoid(zot) * tanh(*ct);  // = [bS x nOut]

  // apply projection
  if (projection) {
    NDArray restultOne = mmul(htNoPeepHole, *Wp);
    ht->assign(&restultOne);  // [bS x nOut] * [ nOut x numProj] = [bS x numProj]
    // if clipping projection is provided then projected cell output state is clipped by this value
    if (clippingProjValue != 0.) ht->applyScalar(scalar::LstmClip, clippingProjValue, ht);
  } else
    ht->assign(&htNoPeepHole);
}

void lstmBlockCell(NDArray* xt, NDArray* cLast, NDArray* yLast, NDArray* W, NDArray* Wci,
                   NDArray* Wcf, NDArray* Wco, NDArray* b, NDArray* i, NDArray* c, NDArray* f,
                   NDArray* o, NDArray* z, NDArray* h, NDArray* y, const std::vector<double>& params) {
  /* Input arrays:
   *    0: xt              - input [bS, nIn] at time t
   *    1: cLast (cs_prev) - previous cell state  [bS, nOut], time t-1
   *    2: yLast (h_prev)  - previous output [bS, nOut], time t-1
   *    3: W               - Weights - concatenated (input-to-hidden, hidden-to-hidden weights)  weights, [(nIn+nOut),
   * 4*nOut] 4: Wci             - weights - cell peephole (t-1) connections to input modulation gate, [nOut] 5: Wcf -
   * weights - cell peephole (t-1) connections to forget gate, [nOut] 6: Wco             - weights - cell peephole (t)
   * connections to output gate, [nOut] 7: b               - biases, [4*nOut]
   *
   *  Input integer arguments:
   *    0: if not zero, provide peephole connections
   *
   *  Input float arguments:
   *    0: the bias added to forget gates in order to reduce the scale of forgetting in the beginning of the training
   *    1: clipping value for cell state, if it is not equal to zero, then cell state is clipped
   *
   * Output arrays:
   *    0: i      - Input modulation gate activations [bS, nOut]
   *    1: c (cs) - Cell state (pre tanh) [bs, nOut] (cs)
   *    2: f      - Output - forget gate activations [bs, nOut]
   *    3: o      - Output - output gate activations [bs, nOut]
   *    4: z (ci) - Output - block input [bs, nOut]
   *    5: h (co) - Cell state, post tanh [bs, nOut]
   *    6: y (h)  - Current cell output [bS, nOut], time t
   */
  const bool peephole = (bool)params[0];  // if true, provide peephole connections
  const double forgetBias = params[1];
  const double clippingCellValue =
      params[2];  // clipping value for ct, if it is not equal to zero, then cell state is clipped

  const int bS = xt->sizeAt(0);
  const int nIn = xt->sizeAt(1);
  const int nOut = cLast->sizeAt(1);

  std::vector<sd::LongType> shape = {xt->sizeAt(0), xt->sizeAt(1) + yLast->sizeAt(1)};
  // Concat inputs: [xt, yt-1]: concat([bs,nIn],[bs,nOut]) -> [bs, (nIn+nOut)]
  NDArray concatOut(xt->ordering(), shape, xt->dataType(),
                    xt->getContext());
  concat(xt->getContext(), {const_cast<NDArray*>(xt), const_cast<NDArray*>(yLast)}, concatOut, {1});

  auto m = mmul(concatOut, *W);  // mmul: [bs, (nIn+nOut)] * [(nIn+nOut), 4*nOut] = [bs, 4*nOut]
  m += (*b);                     // addiRowVector

  // Note: weights are ordered [inputGate, blockInput, forgetGate, outputGate] to match TF (TF code comments state
  // [i,f,z/ci,o] but behaviour is [i,z,f,o])
  auto zi = m({0, 0, 0, nOut});             // z for input modulation gate, [bS, nOut]
  auto zz = m({0, 0, nOut, 2 * nOut});      // z for block input, [bS, nOut]
  auto zf = m({0, 0, 2 * nOut, 3 * nOut});  // z for forget gate, [bS, nOut]
  auto zo = m({0, 0, 3 * nOut, 4 * nOut});  // z for output gate, [bS, nOut]

  if (peephole) {             // add peephole connections: z  +  ct_1*Wc
    zi += (*cLast) * (*Wci);  // add peephole connections to input gate
    zf += (*cLast) * (*Wcf);  // add peephole connections to forget gate
  }

  // current sell state = ft*cLast + it*tanh(mmul(Wxc,xt) + mmul(Whc,ht_1) + bc
  if (forgetBias != 0.0) zf += forgetBias;

  zz.applyTransform(transform::Tanh, z);     // z = tanh(zz)
  zi.applyTransform(transform::Sigmoid, i);  // i = sigmoid(zi)
  zf.applyTransform(transform::Sigmoid, f);  // f = sigmoid(zf);

  // cell state = blockInput .* inputGate + prevCellState .* forgetGate
  z->applyPairwiseTransform(pairwise::Multiply, i, c);  // c = z * i
  auto temp = (*f) * (*cLast);
  *c += temp;                              // c = (i * z) + (zf * (*cLast))
  c->applyTransform(transform::Tanh, h);  // h = tanh(c)

  // if clipping value is provided then cell state is clipped by this value prior to the cell output activation
  if (clippingCellValue > 0.0) c->applyScalar(scalar::LstmClip, clippingCellValue, c);

  if (peephole) {
    // add peephole connections to output gate zot + ct*Wc
    auto prod = *c * (*Wco);
    zo += prod;
  }
  zo.applyTransform(transform::Sigmoid, o);  // o = sigmoid(zo)

  // current cell output = ot*tanh(ct)
  c->applyTransform(transform::Tanh, h);                 // h = tanh(c)
  o->applyPairwiseTransform(pairwise::Multiply, h, y);  // y = o * h
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
