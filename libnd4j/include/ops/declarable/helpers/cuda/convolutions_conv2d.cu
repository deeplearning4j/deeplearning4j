/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//
#include <helpers/MmulHelper.h>
#include <helpers/PointersManager.h>
#include <ops/declarable/helpers/addBias.h>
#include <ops/declarable/helpers/col2im.h>
#include <ops/declarable/helpers/convolutions.h>
#include <ops/declarable/helpers/im2col.h>


namespace sd {
namespace ops {

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
static void conv2d_(sd::graph::Context& block, NDArray* input, NDArray* weights, NDArray* bias,
                    NDArray* output, const LongType kH, const LongType kW, const LongType sH, const LongType sW, LongType pH, LongType pW,
                    const LongType dH, const LongType dW, const int paddingMode, const int isNCHW, const int wFormat) {

  // input   [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
  // weights [kH, kW, iC, oC], [oC, iC, kH, kW], [oC, kH, kW, iC]
  // bias    [oC]
  // output  [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW)

  LongType bS = input->sizeAt(0);
  LongType iC = ConvolutionUtils::inChannels(weights->shapeInfo(), wFormat);
  LongType oC = ConvolutionUtils::outChannels(weights->shapeInfo(), wFormat);
  LongType iH = ConvolutionUtils::inputHeight(input->shapeInfo(), isNCHW);
  LongType iW = ConvolutionUtils::inputWidth(input->shapeInfo(), isNCHW);
  LongType oH = ConvolutionUtils::calcOutDimConv(iH, kH, sH, pH, dH, paddingMode);
  LongType oW = ConvolutionUtils::calcOutDimConv(iW, kW, sW, pW, dW, paddingMode);

  std::vector<LongType> wAxes;
  if (0 == wFormat)
    wAxes = {0, 1, 2};
  else if (1 == wFormat)
    wAxes = {2, 3, 1};
  else
    wAxes = {1, 2, 3};


  std::vector<sd::LongType> colShape = {bS, iC, kH, kW, oH, oW};
  NDArray *col = new NDArray('c', colShape, input->dataType(), input->getContext());
  std::vector<LongType> colPermute = {0, 3, 4, 5, 1, 2};  // {bS, iC, kH, kW, oH, oW}
  NDArray *colP = new NDArray(col->permute(colPermute, false, false));  // {bS, iC, kH, kW, oH, oW}
  std::vector<sd::LongType> mmulResShape = {bS * oH * oW, oC};
  NDArray mmulResult('f', mmulResShape, output->dataType(), output->getContext());

  std::vector<LongType> permuteForOutput = {0, 3, 1, 2};

  //----- calculation of output -----//
  auto ctx = block.launchContext();


  NDArray zero = NDArrayFactory::create(0.f, input->getContext());
  if (isNCHW) {
    helpers::im2col(*ctx, *input, *colP, kH, kW, sH, sW, pH, pW, dH, dW,
                    zero);
  } else {
    std::vector<sd::LongType> permute = {0, 3, 1, 2};
    // For NHWC, we need to permute the input to NCHW before im2col
    NDArray* inputNchw = new NDArray(input->permute(permute, 0, false));
    helpers::im2col(*ctx, *inputNchw, *colP, kH, kW, sH, sW, pH, pW, dH, dW,
                    zero);
  }



  std::vector<sd::LongType> permute = {0, 3, 4, 5, 1, 2};
  block.pushIntermediateResult(col);

  std::vector<sd::LongType> shape = {bS * oH * oW, kW * kH * iC};
  auto im2colReshape = col->reshape('c', shape, true);

  auto weightsPermuted = weights->permute(permuteForOutput, 0, false);
  std::vector<LongType> weightShape = {iC * kH * kW, oC};
  auto reshapedW = weightsPermuted.reshape('f', weightShape, false);
  MmulHelper::matmul(&im2colReshape, &reshapedW, &mmulResult, false, false, 1.0, 0.0);


  std::vector<LongType> mmulResultShape = {oH, oW, bS, oC};
  auto reshaped = mmulResult.reshape('f', mmulResultShape, false);
  std::vector<sd::LongType> permutedShape = {2, 3, 1,0};
  auto permuted = reshaped.permute(permutedShape, 0, false);

  // Reshape and copy result to output
  if (isNCHW) {
    output->assign(&permuted);
  } else {
    std::vector<sd::LongType> otherPermute = {0,2,3,1};
    permuted = permuted.permute(otherPermute, 0, false);
    output->assign(&permuted);
  }

  //----- add biases if required -----//
  if (bias) {
    helpers::addBias(block, *output, *bias, *output, isNCHW);
  }

}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::conv2d(sd::graph::Context& block, NDArray* input, NDArray* weights,
                              NDArray* bias, NDArray* output, const LongType kH, const LongType kW, const LongType sH,
                              const LongType sW, LongType pH, LongType pW, const LongType dH, const LongType dW, const int paddingMode,
                              const int isNCHW, const int wFormat) {
  BUILD_SINGLE_SELECTOR_TWICE(
      input->dataType(), conv2d_,
      (block, input, weights, bias, output, kH, kW, sH, sW, pH, pW, dH, dW, paddingMode, isNCHW, wFormat),
      SD_FLOAT_TYPES);
}


}  // namespace ops
}  // namespace sd
