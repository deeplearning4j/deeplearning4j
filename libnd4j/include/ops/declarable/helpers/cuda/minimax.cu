#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com (CUDA implementation)
//
#include <array/NDArray.h>
#include <helpers/ShapeUtils.h>
#include <helpers/PointersManager.h>
#include <ops/declarable/helpers/minimax.h>
#include <system/op_boilerplate.h>
#include <exceptions/cuda_exception.h>

#include "execution/cuda/LaunchDims.h"

namespace sd {
namespace ops {
namespace helpers {

// CUDA kernels for minimum and maximum backprop operations

// Kernel for element-wise case
template <typename T>
static SD_KERNEL void minimumMaximumScalarBPKernel(void* vx, const LongType* xShapeInfo, 
                                                  T scalarVal, bool isMin,
                                                  void* veps, const LongType* epsShapeInfo,
                                                  void* vgradX, const LongType* gradXShapeInfo) {
    const auto x = reinterpret_cast<T*>(vx);
    const auto eps = reinterpret_cast<T*>(veps);
    auto gradX = reinterpret_cast<T*>(vgradX);
    
    __shared__ LongType xRank, xLength;
    __shared__ const LongType *xShape, *xStride;
    __shared__ LongType epsRank, epsLength;
    __shared__ const LongType *epsShape, *epsStride;
    
    if (threadIdx.x == 0) {
        xRank = shape::rank(xShapeInfo);
        xLength = shape::length(xShapeInfo);
        xShape = shape::shapeOf(xShapeInfo);
        xStride = shape::stride(xShapeInfo);
        epsRank = shape::rank(epsShapeInfo);
        epsLength = shape::length(epsShapeInfo);
        epsShape = shape::shapeOf(epsShapeInfo);
        epsStride = shape::stride(epsShapeInfo);
    }
    __syncthreads();
    
    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    const auto step = gridDim.x * blockDim.x;
    
    for (LongType i = tid; i < xLength; i += step) {
        LongType xOffset = shape::getIndexOffset(i, xShapeInfo);
        LongType epsOffset = shape::getIndexOffset(i, epsShapeInfo);
        
        if (isMin) {
            // Minimum backprop
            gradX[xOffset] = x[xOffset] <= scalarVal ? eps[epsOffset] : static_cast<T>(0.0);
        } else {
            // Maximum backprop
            gradX[xOffset] = x[xOffset] >= scalarVal ? eps[epsOffset] : static_cast<T>(0.0);
        }
    }
}

// Kernel for element-wise case
template <typename T>
static SD_KERNEL void minimumMaximumBPKernel(void* vx, const LongType* xShapeInfo, 
                                           void* vy, const LongType* yShapeInfo,
                                           void* veps, const LongType* epsShapeInfo,
                                           void* vgradX, const LongType* gradXShapeInfo,
                                           void* vgradY, const LongType* gradYShapeInfo, 
                                           bool isMin) {
    const auto x = reinterpret_cast<T*>(vx);
    const auto y = reinterpret_cast<T*>(vy);
    const auto eps = reinterpret_cast<T*>(veps);
    auto gradX = reinterpret_cast<T*>(vgradX);
    auto gradY = reinterpret_cast<T*>(vgradY);
    
    __shared__ LongType length;
    
    if (threadIdx.x == 0) {
        length = shape::length(xShapeInfo);
    }
    __syncthreads();
    
    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    const auto step = gridDim.x * blockDim.x;
    
    for (LongType i = tid; i < length; i += step) {
        // Calculate offsets for each array
        LongType xOffset = shape::getIndexOffset(i, xShapeInfo);
        LongType yOffset = shape::getIndexOffset(i, yShapeInfo);
        LongType epsOffset = shape::getIndexOffset(i, epsShapeInfo);
        LongType gradXOffset = shape::getIndexOffset(i, gradXShapeInfo);
        LongType gradYOffset = shape::getIndexOffset(i, gradYShapeInfo);
        
        if (isMin) {
            // Minimum backprop
            gradX[gradXOffset] = x[xOffset] <= y[yOffset] ? eps[epsOffset] : static_cast<T>(0.0);
            gradY[gradYOffset] = x[xOffset] >= y[yOffset] ? eps[epsOffset] : static_cast<T>(0.0);
        } else {
            // Maximum backprop
            gradX[gradXOffset] = x[xOffset] >= y[yOffset] ? eps[epsOffset] : static_cast<T>(0.0);
            gradY[gradYOffset] = x[xOffset] <= y[yOffset] ? eps[epsOffset] : static_cast<T>(0.0);
        }
    }
}

template <typename T>
static void minimumBPFunctor_(LaunchContext* context, NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY) {
    NDArray::prepareSpecialUse({gradX, gradY}, {x, y, epsNext});
    
    dim3 launchDims = getLaunchDims("minimax");
    
    if (x->isSameShape(y)) {
        // Element-wise case (same shapes)
        minimumMaximumBPKernel<T><<<launchDims.y, launchDims.x, launchDims.z, *context->getCudaStream()>>>(
            x->specialBuffer(), x->specialShapeInfo(),
            y->specialBuffer(), y->specialShapeInfo(),
            epsNext->specialBuffer(), epsNext->specialShapeInfo(),
            gradX->specialBuffer(), gradX->specialShapeInfo(),
            gradY->specialBuffer(), gradY->specialShapeInfo(),
            true);  // isMin = true
    } else if (y->isScalar()) {
        // Scalar case
        T scalar = y->e<T>(0);
        
        minimumMaximumScalarBPKernel<T><<<launchDims.y, launchDims.x, launchDims.z, *context->getCudaStream()>>>(
            x->specialBuffer(), x->specialShapeInfo(),
            scalar, true,  // isMin = true
            epsNext->specialBuffer(), epsNext->specialShapeInfo(),
            gradX->specialBuffer(), gradX->specialShapeInfo());
            
        // Set gradY value based on comparison
        if (*x <= *y) {
            auto tmp = epsNext->reduceNumber(reduce::Sum);
            gradY->assign(&tmp);
        } else {
            gradY->assign(static_cast<T>(0.0));
        }
    } else {
        // Broadcast case - more complex, falls back to CPU for now
        // We'd need to tile arrays to same shape, compute, then reduce along broadcast dims
        
        PointersManager manager(context, "minimumBPFunctor");
        manager.synchronize();
        
        // Move to host, perform calculation, then copy back to device
        x->syncToHost();
        y->syncToHost();
        epsNext->syncToHost();
        
        auto lambdaX = LAMBDA_TTT(_e, _x, _y) { return _x <= _y ? _e : (T)0.; });
        auto lambdaY = LAMBDA_TTT(_e, _x, _y) { return _x >= _y ? _e : (T)0.; });
        
        auto preX = x->dup();
        auto preY = y->dup();
        auto targetShape = epsNext->getShapeAsVector();
        
        preX.tileToShape(targetShape, preX);
        preY.tileToShape(targetShape, preY);
        
        epsNext->applyTriplewiseLambda<T>(&preX, &preY, lambdaX, &preX);
        epsNext->applyTriplewiseLambda<T>(&preX, &preY, lambdaY, &preY);
        
        auto axisX = ShapeUtils::evalBroadcastBackwardAxis(x->shapeInfo(), epsNext->shapeInfo());
        auto axisY = ShapeUtils::evalBroadcastBackwardAxis(y->shapeInfo(), epsNext->shapeInfo());
        
        if (axisX.size() > 0) {
            auto sum = preX.reduceAlongDimension(reduce::Sum, &axisX);
            gradX->assign(&sum);
        } else {
            gradX->assign(&preX);
        }
        
        if (axisY.size() > 0) {
            auto sum = preY.reduceAlongDimension(reduce::Sum, &axisY);
            gradY->assign(&sum);
        } else {
            gradY->assign(&preY);
        }
        
        gradX->syncToDevice();
        gradY->syncToDevice();
    }
    
    DebugHelper::checkErrorCode(context->getCudaStream(), "minimumBPFunctor CUDA kernel failed");
    NDArray::registerSpecialUse({gradX, gradY}, {x, y, epsNext});
}

template <typename T>
static void maximumBPFunctor_(LaunchContext* context, NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY) {
    NDArray::prepareSpecialUse({gradX, gradY}, {x, y, epsNext});
    
    dim3 launchDims = getLaunchDims("minimax");
    
    if (x->isSameShape(y)) {
        // Element-wise case (same shapes)
        minimumMaximumBPKernel<T><<<launchDims.y, launchDims.x, launchDims.z, *context->getCudaStream()>>>(
            x->specialBuffer(), x->specialShapeInfo(),
            y->specialBuffer(), y->specialShapeInfo(),
            epsNext->specialBuffer(), epsNext->specialShapeInfo(),
            gradX->specialBuffer(), gradX->specialShapeInfo(),
            gradY->specialBuffer(), gradY->specialShapeInfo(),
            false);  // isMin = false
    } else if (y->isScalar()) {
        // Scalar case
        T scalar = y->e<T>(0);
        
        minimumMaximumScalarBPKernel<T><<<launchDims.y, launchDims.x, launchDims.z, *context->getCudaStream()>>>(
            x->specialBuffer(), x->specialShapeInfo(),
            scalar, false,  // isMin = false
            epsNext->specialBuffer(), epsNext->specialShapeInfo(),
            gradX->specialBuffer(), gradX->specialShapeInfo());
            
        // Set gradY value based on comparison
        if (*x <= *y) {
            auto tmp = epsNext->reduceNumber(reduce::Sum);
            gradY->assign(&tmp);
        } else {
            gradY->assign(static_cast<T>(0.0));
        }
    } else {
        // Broadcast case - more complex, falls back to CPU for now
        // We'd need to tile arrays to same shape, compute, then reduce along broadcast dims
        
        PointersManager manager(context, "maximumBPFunctor");
        manager.synchronize();
        
        // Move to host, perform calculation, then copy back to device
        x->syncToHost();
        y->syncToHost();
        epsNext->syncToHost();
        
        auto lambdaX = LAMBDA_TTT(_e, _x, _y) { return _x >= _y ? _e : (T)0.; });
        auto lambdaY = LAMBDA_TTT(_e, _x, _y) { return _x <= _y ? _e : (T)0.; });
        
        auto preX = x->dup();
        auto preY = y->dup();
        auto targetShape = epsNext->getShapeAsVector();
        
        preX.tileToShape(targetShape, preX);
        preY.tileToShape(targetShape, preY);
        
        epsNext->applyTriplewiseLambda<T>(&preX, &preY, lambdaX, &preX);
        epsNext->applyTriplewiseLambda<T>(&preX, &preY, lambdaY, &preY);
        
        auto axisX = ShapeUtils::evalBroadcastBackwardAxis(x->shapeInfo(), epsNext->shapeInfo());
        auto axisY = ShapeUtils::evalBroadcastBackwardAxis(y->shapeInfo(), epsNext->shapeInfo());
        
        if (axisX.size() > 0) {
            auto sum = preX.reduceAlongDimension(reduce::Sum, &axisX);
            gradX->assign(&sum);
        } else {
            gradX->assign(&preX);
        }
        
        if (axisY.size() > 0) {
            auto sum = preY.reduceAlongDimension(reduce::Sum, &axisY);
            gradY->assign(&sum);
        } else {
            gradY->assign(&preY);
        }
        
        gradX->syncToDevice();
        gradY->syncToDevice();
    }
    
    DebugHelper::checkErrorCode(context->getCudaStream(), "maximumBPFunctor CUDA kernel failed");
    NDArray::registerSpecialUse({gradX, gradY}, {x, y, epsNext});
}

void minimumBPFunctor(LaunchContext* context, NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY) {
    BUILD_SINGLE_SELECTOR(x->dataType(), minimumBPFunctor_, (context, x, y, epsNext, gradX, gradY), SD_NUMERIC_TYPES);
}

void maximumBPFunctor(LaunchContext* context, NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY) {
    BUILD_SINGLE_SELECTOR(x->dataType(), maximumBPFunctor_, (context, x, y, epsNext, gradX, gradY), SD_NUMERIC_TYPES);
}

BUILD_SINGLE_TEMPLATE(template void minimumBPFunctor_,
                     (LaunchContext* context, NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY), 
                     SD_NUMERIC_TYPES);
BUILD_SINGLE_TEMPLATE(template void maximumBPFunctor_,
                     (LaunchContext* context, NDArray* x, NDArray* y, NDArray* epsNext, NDArray* gradX, NDArray* gradY), 
                     SD_NUMERIC_TYPES);

}  // namespace helpers
}  // namespace ops
}  // namespace sd