#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//
#include <array/NDArrayFactory.h>
#include <exceptions/cuda_exception.h>
#include <execution/cuda/LaunchDims.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <helpers/ShapeUtils.h>

#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <system/selective_rendering.h>

#include "helpers/DebugHelper.h"
namespace sd {
namespace ops {
namespace helpers {

// -------------------------------------------------------------------------------------------------------------- //
// Sorted segments ops implementations

template <typename T, typename I>
static bool segmentIndicesValidate_(NDArray* indices, NDArray& aexpected, NDArray& aoutput) {
  return true;
}

bool segmentIndicesValidate(LaunchContext* context, NDArray* indices, NDArray& expected, NDArray& output) {
  auto indicesDType = indices->dataType();
  auto outputDType = output.dataType();
  BUILD_DOUBLE_SELECTOR(output.dataType(), indices->dataType(), return segmentIndicesValidate_,
                        (indices, expected, output), SD_NUMERIC_TYPES, SD_INDEXING_TYPES);
}

// -------------------------------------------------------------------------------------------------------------- //
// Unsorted segment ops functors implementation
// -------------------------------------------------------------------------------------------------------------- //
template <typename I>
static SD_KERNEL void unsortedSegmentIndexValidateKernel(const I* indices, const LongType* indicesShape, I expected,
                                                         I* found) {
  __shared__ bool onlyTrue;
  __shared__ LongType len;

  if (threadIdx.x == 0) {
    onlyTrue = true;
    len = shape::length(indicesShape);
  }
  __syncthreads();
  auto start = threadIdx.x + blockIdx.x * blockDim.x;
  auto step = gridDim.x * blockDim.x;
  for (LongType e = start; e < len && onlyTrue; e += step) {
    math::atomics::sd_atomicMax(found, indices[e]);
    if (expected < *found) onlyTrue = false;
  }
}

template <typename I>
static bool unsortedSegmentIndicesValidate_(LaunchContext* context, NDArray* indices, LongType expected,
                                            LongType& output) {
  output = expected;
  I found = output;
  I exp = expected;
  auto stream = context->getCudaStream();
  I* devFound;
  hipMalloc(&devFound, sizeof(I));
  hipMemcpy(devFound, &found, sizeof(I), hipMemcpyHostToDevice);

  dim3 launchDims = segmentValidateIndices(indices->lengthOf());
  unsortedSegmentIndexValidateKernel<I><<<launchDims.y,launchDims.x, launchDims.z, *stream>>>(
      reinterpret_cast<I*>(indices->specialBuffer()), indices->specialShapeInfo(), exp, devFound);
  sd::DebugHelper::checkErrorCode(stream, "unsortedSegmentIndexValidateKernel failed");

  hipMemcpy(&found, devFound, sizeof(I), hipMemcpyDeviceToHost);
  hipFree(devFound);
  output = found;
  return expected == output;
}

bool unsortedSegmentIndicesValidate(LaunchContext* context, NDArray* indices, LongType expected, LongType& output) {
  BUILD_SINGLE_SELECTOR(indices->dataType(), return unsortedSegmentIndicesValidate_,
                        (context, indices, expected, output), SD_INDEXING_TYPES);
}

// -------------------------------------------------------------------------------------------------------------- //

// -------------------------------------------------------------------------------------------------------------- //
// fill up segments starts and ends - splitted ordered case
template <typename I>
static SD_KERNEL void fillUpSegmentsKernel(const void* indices, const LongType* indexShape, LongType numClasses,
                                           LongType* classesRangesStart, LongType* classesRangesLengths) {
  __shared__ const I* idxBuf;
  __shared__ LongType idxLen;
  __shared__ LongType* result;
  if (threadIdx.x == 0) {
    idxBuf = reinterpret_cast<const I*>(indices);
    idxLen = shape::length(indexShape);
  }
  __syncthreads();

  auto tid = threadIdx.x + blockDim.x * blockIdx.x;
  auto step = blockDim.x * gridDim.x;

  for (auto j = tid; j < idxLen; j += step) {
    auto pos = idxBuf[j];
    math::atomics::sd_atomicMin<LongType>(&classesRangesStart[pos], (LongType)j);
    math::atomics::sd_atomicAdd<LongType>(&classesRangesLengths[pos], 1);
  }
}

// -------------------------------------------------------------------------------------------------------------- //

template <typename I>
static void fillUpSegments_(NDArray* indices, LongType numClasses, NDArray& classesRangesBegs,
                            NDArray& classesRangesLens) {
  dim3 dims = getFillUpSegmentsDims(numClasses, indices->lengthOf());
  LongType* begins = reinterpret_cast<LongType*>(classesRangesBegs.specialBuffer());
  LongType* lengths = reinterpret_cast<LongType*>(classesRangesLens.specialBuffer());
  auto stream = classesRangesBegs.getContext()->getCudaStream();
  fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(),
                                                               numClasses, begins, lengths);
  sd::DebugHelper::checkErrorCode(stream, "fillUpSegmentsKernel failed");

}
// -------------------------------------------------------------------------------------------------------------- //

void fillUpSegments(NDArray* indices, LongType numClasses, NDArray& classesRangesBegs, NDArray& classesRangesLens) {
  BUILD_SINGLE_SELECTOR(indices->dataType(), fillUpSegments_,
                        (indices, numClasses, classesRangesBegs, classesRangesLens), SD_INDEXING_TYPES);
}
// -------------------------------------------------------------------------------------------------------------- //

}  // namespace helpers
}  // namespace ops
}  // namespace sd
// -------------------------------------------------------------------------------------------------------------- //
// -------------------------------------------------------------------------------------------------------------- //
