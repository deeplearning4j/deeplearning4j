#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 17.05.2018
// @author raver119@gmail.com
//
#include <array/NDArrayFactory.h>
#include <array/ResultSet.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/DebugHelper.h>
#include <ops/declarable/helpers/percentile.h>

#include "execution/cuda/LaunchDims.h"


namespace sd {
namespace ops {
namespace helpers {

template <typename X>
static SD_KERNEL void percentileKernel(void* vx, const LongType* xTadShapeInfo, const LongType* xTadOffsets,
                                       const LongType numTads, const LongType tadLength, void* vz,
                                       const LongType* zShapeInfo, const LongType zLength,
                                       const LongType position) {
  const auto x = reinterpret_cast<X*>(vx);
  auto z = reinterpret_cast<X*>(vz);

  __shared__ LongType xRank, zRank;
  __shared__ const LongType* xShape;
  __shared__ const LongType* xStride;
  __shared__ const LongType* zShape;
  __shared__ const LongType* zStride;

  if (threadIdx.x == 0) {
    xRank = shape::rank(xTadShapeInfo);
    zRank = shape::rank(zShapeInfo);
    xShape = shape::shapeOf(xTadShapeInfo);
    xStride = shape::stride(xTadShapeInfo);
    zShape = shape::shapeOf(zShapeInfo);
    zStride = shape::stride(zShapeInfo);
  }
  __syncthreads();

  for (LongType t = blockIdx.x; t < numTads; t += gridDim.x) {
    auto tad = x + xTadOffsets[t];

    // Sort TAD using odd-even transposition sort
    for (LongType m = 0; m < tadLength; ++m) {
      for (LongType tid = threadIdx.x; tid < tadLength; tid += blockDim.x) {
        const auto top = (m % 2 == 0) ? 2 * tid + 1 : 2 * tid + 2;
        if (top < tadLength) {
          if (tad[top - 1] > tad[top]) {
            // Swap values
            X temp = tad[top - 1];
            tad[top - 1] = tad[top];
            tad[top] = temp;
          }
        }
      }
      __syncthreads();
    }

    // Save the final value to the output
    if (threadIdx.x == 0) {
      const auto value = tad[position];
      LongType zOffset;

      COORDS2INDEX(zRank, zStride, &t, zOffset);
      z[zOffset] = value;
    }
    __syncthreads();
  }
}


template <typename T>
static void _percentile(LaunchContext* context, NDArray& input, NDArray& output, std::vector<LongType>& axis,
                        const float q, const int interpolation) {
  const int inputRank = input.rankOf();

  if (axis.empty())
    for (int i = 0; i < inputRank; ++i) axis.push_back(i);
  else
    shape::checkDimensions(inputRank, &axis);

  auto tempArray = input.dup();
  auto packX = ConstantTadHelper::getInstance().tadForDimensions(tempArray.shapeInfo(), &axis);

  auto tadLength = shape::length(packX->primaryShapeInfo());

  const float fraction = 1.f - q / 100.;
  LongType position = 0;

  switch (interpolation) {
    case 0:  // lower
      position = static_cast<LongType>(math::sd_ceil<float, T>((tadLength - 1) * fraction));
      break;
    case 1:  // higher
      position = static_cast<LongType>(math::sd_floor<float, T>((tadLength - 1) * fraction));
      break;
    case 2:  // nearest
      position = static_cast<LongType>(math::sd_round<float, T>((tadLength - 1) * fraction));
      break;
  }
  position = tadLength - position - 1;

  dim3 launchDims = getLaunchDims("percentile");
  percentileKernel<T><<<launchDims.y, launchDims.x, launchDims.z, *context->getCudaStream()>>>(
      tempArray.specialBuffer(), packX->platformShapeInfo(), packX->platformOffsets(), packX->numberOfTads(), tadLength,
      output.specialBuffer(), output.specialShapeInfo(), output.lengthOf(), position);

  DebugHelper::checkErrorCode(context->getCudaStream(), "percentile");
}

void percentile(LaunchContext* context, NDArray& input, NDArray& output, std::vector<LongType>& axises,
                const float q, const int interpolation) {
  NDArray::prepareSpecialUse({&output}, {&input});

  BUILD_SINGLE_SELECTOR(input.dataType(), _percentile, (context, input, output, axises, q, interpolation),
                        SD_COMMON_TYPES);

  NDArray::registerSpecialUse({&output}, {&input});
}

BUILD_SINGLE_TEMPLATE( void _percentile,
                      (sd::LaunchContext * context, NDArray& input, NDArray& output, std::vector<sd::LongType>& axises,
                       const float q, const int interpolation),
                      SD_COMMON_TYPES);

}  // namespace helpers
}  // namespace ops
}  // namespace sd
