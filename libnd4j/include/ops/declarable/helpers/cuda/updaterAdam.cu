#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <system/op_boilerplate.h>

#include "execution/cuda/LaunchDims.h"
#include "helpers/DebugHelper.h"


namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename T>
SD_KERNEL void adamUpdaterCuda(const void* vx, const LongType* xShapeInfo, const void* vinv,
                               const LongType* invShapeInfo, const void* vinm, const LongType* inmShapeInfo,
                               void* vz, const LongType* zShapeInfo, void* vstV, const LongType* stvShapeInfo,
                               void* vstM, const LongType* stmShapeInfo, const T lr, const T beta1, const T beta2,
                               const T epsilon, const T iteration) {
  const auto grad = reinterpret_cast<const T*>(vx);
  const auto initU = reinterpret_cast<const T*>(vinv);
  const auto initM = reinterpret_cast<const T*>(vinm);
  auto up = reinterpret_cast<T*>(vz);
  auto stU = reinterpret_cast<T*>(vstV);
  auto stM = reinterpret_cast<T*>(vstM);

  __shared__ LongType xLen, xRank, zRank, invRank, inmRank, stvRank, stmRank;
  __shared__ T epsilonT;
  __shared__ bool bOrdering, bXZsame, bXInUSame, bXStUSame, bXInMSame, bXStMSame;
  __shared__ LongType *sharedMem;
  __shared__ const LongType *xShape, *zShape, *invShape, *inmShape, *stvShape, *stmShape;
  __shared__ const LongType *xStride, *zStride, *invStride, *inmStride, *stvStride, *stmStride;

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char shmem[];
    sharedMem = reinterpret_cast<LongType*>(shmem);

    xLen = shape::length(xShapeInfo);

    T beta1T = math::sd_pow<T, T, T>(beta1, (iteration + 1));
    T beta2T = math::sd_pow<T, T, T>(beta2, (iteration + 1));
    epsilonT = lr * math::sd_sqrt<T, T>(1. - beta2T) / (1.0 - beta1T);
    if (math::sd_isnan(epsilonT) || 0 == epsilonT || math::sd_isinf(epsilonT)) epsilonT = epsilon;

    xRank = shape::rank(xShapeInfo);
    zRank = shape::rank(zShapeInfo);
    invRank = shape::rank(invShapeInfo);
    inmRank = shape::rank(inmShapeInfo);
    stvRank = shape::rank(stvShapeInfo);
    stmRank = shape::rank(stmShapeInfo);

    xShape = shape::shapeOf(xShapeInfo);
    xStride = shape::stride(xShapeInfo);
    zShape = shape::shapeOf(zShapeInfo);
    zStride = shape::stride(zShapeInfo);
    invShape = shape::shapeOf(invShapeInfo);
    invStride = shape::stride(invShapeInfo);
    inmShape = shape::shapeOf(inmShapeInfo);
    inmStride = shape::stride(inmShapeInfo);
    stvShape = shape::shapeOf(stvShapeInfo);
    stvStride = shape::stride(stvShapeInfo);
    stmShape = shape::shapeOf(stmShapeInfo);
    stmStride = shape::stride(stmShapeInfo);

    bOrdering = shape::order(xShapeInfo) == shape::order(zShapeInfo) &&
                shape::order(zShapeInfo) == shape::order(stmShapeInfo) &&
                shape::order(stmShapeInfo) == shape::order(inmShapeInfo) &&
                shape::order(inmShapeInfo) == shape::order(stvShapeInfo) &&
                shape::order(stvShapeInfo) == shape::order(invShapeInfo);

    bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
    bXInUSame = shape::haveSameShapeAndStrides(xShapeInfo, invShapeInfo);
    bXStUSame = shape::haveSameShapeAndStrides(xShapeInfo, stvShapeInfo);
    bXInMSame = shape::haveSameShapeAndStrides(xShapeInfo, inmShapeInfo);
    bXStMSame = shape::haveSameShapeAndStrides(xShapeInfo, stmShapeInfo);
  }
  __syncthreads();

  LongType coords[SD_MAX_RANK];

  for (LongType i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {
    LongType xOffset = i, zOffset = i, initMOffset = i, initUOffset = i, stMOffset = i, stUOffset = i;

    if (!bOrdering) {
      INDEX2COORDS(i, xRank, xShape, coords);
      COORDS2INDEX(xRank, xStride, coords, xOffset);

      if (!bXZsame)
        COORDS2INDEX(zRank, zStride, coords, zOffset);
      if (!bXInUSame)
        COORDS2INDEX(invRank, invStride, coords, initUOffset);
      if (!bXStUSame)
        COORDS2INDEX(stvRank, stvStride, coords, stUOffset);
      if (!bXInMSame)
        COORDS2INDEX(inmRank, inmStride, coords, initMOffset);
      if (!bXStMSame)
        COORDS2INDEX(stmRank, stmStride, coords, stMOffset);
    }

    stM[stMOffset] = beta1 * initM[initMOffset] + grad[xOffset] * (1 - beta1);
    stU[stUOffset] = beta2 * initU[initUOffset] + grad[xOffset] * grad[xOffset] * (1 - beta2);
    up[zOffset] = (stM[stMOffset] * epsilonT) / (math::sd_sqrt<T, T>(stU[stUOffset]) + epsilon);
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
void adamUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMemory,
                             const hipStream_t* stream, const void* vx, const LongType* xShapeInfo,
                             const void* vinv, const LongType* invShapeInfo, const void* vinm,
                             const LongType* inmShapeInfo, void* vz, const LongType* zShapeInfo, void* vstV,
                             const LongType* stvShapeInfo, void* vstM, const LongType* stmShapeInfo,
                             const double dLr, const double dBeta1, const double dBeta2, const double dEpsilon,
                             const int nIteration) {
  const T lr = static_cast<T>(dLr);
  const T beta1 = static_cast<T>(dBeta1);
  const T beta2 = static_cast<T>(dBeta2);
  T epsilon = static_cast<T>(dEpsilon);
  //fp16 to prevent underflow
  if(epsilon == 0.0) {
    epsilon = static_cast<T>(1e-7);
  }
  const T iteration = static_cast<T>(nIteration);
  adamUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMemory, *stream>>>(
      vx, xShapeInfo, vinv, invShapeInfo, vinm, inmShapeInfo, vz, zShapeInfo, vstV, stvShapeInfo, vstM, stmShapeInfo,
      lr, beta1, beta2, epsilon, iteration);
  sd::DebugHelper::checkErrorCode(const_cast<hipStream_t *>(stream), "adamUpdaterCuda failed");

}

///////////////////////////////////////////////////////////////////
void updaterAdam(LaunchContext* context, NDArray& gradient, NDArray& initStateU,
                 NDArray& initStateM, NDArray& update, NDArray& stateU, NDArray& stateM, const double dLr,
                 const double dBeta1, const double dBeta2, const double dEpsilon, const int nIteration) {
  PointersManager manager(context, "adamUpdater");

  dim3 launchDims = updaterDims(gradient.lengthOf());
  NDArray::prepareSpecialUse({&update, &stateU, &stateM}, {&gradient, &initStateU, &initStateM});

  BUILD_SINGLE_SELECTOR(gradient.dataType(), adamUpdaterCudaLauncher,
                        (launchDims.y, launchDims.x,launchDims.z, context->getCudaStream(), gradient.specialBuffer(),
                         gradient.specialShapeInfo(), initStateU.specialBuffer(), initStateU.specialShapeInfo(),
                         initStateM.specialBuffer(), initStateM.specialShapeInfo(), update.specialBuffer(),
                         update.specialShapeInfo(), stateU.specialBuffer(), stateU.specialShapeInfo(),
                         stateM.specialBuffer(), stateM.specialShapeInfo(), dLr, dBeta1, dBeta2, dEpsilon, nIteration),
                        SD_FLOAT_TYPES);

  NDArray::registerSpecialUse({&update, &stateU, &stateM}, {&gradient, &initStateU, &initStateM});

  manager.synchronize();
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
