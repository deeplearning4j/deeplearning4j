#include "hip/hip_runtime.h"
/* ******************************************************************************
*
*
* This program and the accompanying materials are made available under the
* terms of the Apache License, Version 2.0 which is available at
* https://www.apache.org/licenses/LICENSE-2.0.
*
*  See the NOTICE file distributed with this work for additional
*  information regarding copyright ownership.
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
* WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
* License for the specific language governing permissions and limitations
* under the License.
*
* SPDX-License-Identifier: Apache-2.0
******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//
#include <execution/cuda/LaunchDims.h>
#include <ops/declarable/helpers/sequence_mask.h>


#include "helpers/DebugHelper.h"
namespace sd {
namespace ops {
namespace helpers {

template <typename I, typename B>
static SD_KERNEL void sequenceMaskKernel(const void* inputBuf, const LongType* inputShape, void* outputBuf,
                                         const LongType* outputShape, int maxIndex) {
  __shared__ const I* input;
  __shared__ B* output;
  __shared__ LongType inputLen, outputLen;

  // Cache shape information
  __shared__ sd::LongType inputRank, outputRank;
  __shared__ const sd::LongType* inputShapePtr;
  __shared__ const sd::LongType* outputShapePtr;
  __shared__ const sd::LongType* inputStridePtr;
  __shared__ const sd::LongType* outputStridePtr;

  if (threadIdx.x == 0) {
    input = reinterpret_cast<const I*>(inputBuf);
    output = reinterpret_cast<B*>(outputBuf);
    inputLen = shape::length(inputShape);
    outputLen = shape::length(outputShape);

    // Cache shape information
    inputRank = shape::rank(inputShape);
    outputRank = shape::rank(outputShape);
    inputShapePtr = shape::shapeOf(inputShape);
    outputShapePtr = shape::shapeOf(outputShape);
    inputStridePtr = shape::stride(inputShape);
    outputStridePtr = shape::stride(outputShape);
  }
  __syncthreads();

  LongType inputCoords[SD_MAX_RANK];
  LongType outputCoords[SD_MAX_RANK];
  LongType inputOffset;
  LongType outputOffset;

  for (auto i = blockIdx.x; i < maxIndex; i += gridDim.x)
    for (auto k = threadIdx.x; k < inputLen; k += blockDim.x) {
      INDEX2COORDS(k, inputRank, inputShapePtr, inputCoords);
      COORDS2INDEX(inputRank, inputStridePtr, inputCoords, inputOffset);
      if (i < input[inputOffset]) {
        INDEX2COORDS(k * maxIndex + i, outputRank, outputShapePtr, outputCoords);
        COORDS2INDEX(outputRank, outputStridePtr, outputCoords, outputOffset);
        output[outputOffset] = B(true);
      }
    }
}
template <typename I, typename B>
static void sequenceMask_(LaunchContext* context, NDArray* input, NDArray* output, int maxIndex) {
 dim3 launchDims = getSequenceMaskLaunchDims(maxIndex,*input);
 NDArray::prepareSpecialUse({output}, {input});
 auto stream = context->getCudaStream();
 sequenceMaskKernel<I, B><<<launchDims.y, launchDims.x, launchDims.z, *stream>>>(
     input->specialBuffer(), input->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), maxIndex);
 sd::DebugHelper::checkErrorCode(stream, "sequenceMaskKernel failed");

 NDArray::registerSpecialUse({output}, {input});
}

void sequenceMask(LaunchContext* context, NDArray* input, NDArray* output, int maxIndex) {
#if SD_IS_PAIR_TYPE_COMPILED(input->dataType(),output->dataType())
 BUILD_DOUBLE_SELECTOR(input->dataType(), output->dataType(), sequenceMask_, (context, input, output, maxIndex),
                       SD_INTEGER_TYPES, SD_COMMON_TYPES_EXTENDED);
#endif
}

BUILD_DOUBLE_TEMPLATE(template void sequenceMask_,
                     (sd::LaunchContext * context, NDArray* input, NDArray* output, int maxIndex), SD_INTEGER_TYPES,
                     SD_COMMON_TYPES_EXTENDED);
}  // namespace helpers
}  // namespace ops
}  // namespace sd
