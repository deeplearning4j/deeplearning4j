#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 19.04.2018
// @author raver119@gmail.com
//
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <helpers/ShapeUtils.h>
#include <ops/declarable/helpers/activations.h>
#include <system/op_boilerplate.h>

#include <numeric>

#include "execution/cuda/LaunchDims.h"


namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename X, typename Y>
void SD_KERNEL preluCuda(const void *vx, const LongType *xShapeInfo, const void *vy, const LongType *yShapeInfo,
                         void *vz) {
  const auto x = reinterpret_cast<const X *>(vx);
  const auto y = reinterpret_cast<const Y *>(vy);
  auto z = reinterpret_cast<X *>(vz);

  __shared__ LongType xzLen;
  __shared__ int xzRank, yRank;
  __shared__ const LongType *xzShape;
  __shared__ const LongType *xzStride;
  __shared__ const LongType *yShape;
  __shared__ const LongType *yStride;

  if (threadIdx.x == 0) {
    xzLen = shape::length(xShapeInfo);
    xzRank = shape::rank(xShapeInfo);
    yRank = shape::rank(yShapeInfo);
    xzShape = shape::shapeOf(xShapeInfo);
    xzStride = shape::stride(xShapeInfo);
    yShape = shape::shapeOf(yShapeInfo);
    yStride = shape::stride(yShapeInfo);
  }
  __syncthreads();

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  LongType coords[SD_MAX_RANK];

  for (int i = tid; i < xzLen; i += blockDim.x * gridDim.x) {
    INDEX2COORDS(i, xzRank, xzShape, coords);

    LongType xzOffset;
    COORDS2INDEX(xzRank, xzStride, coords, xzOffset);
    const auto xVal = x[xzOffset];

    if (xVal < 0) {
      for (LongType j = 0; j < yRank; ++j)
        if (yShapeInfo[j + 1] == 1) coords[j + 1] = 0;

      LongType yOffset;
      COORDS2INDEX(yRank, yStride, coords + 1, yOffset);
      z[xzOffset] = xVal * y[yOffset];
    } else {
      z[xzOffset] = xVal;
    }
  }
}
///////////////////////////////////////////////////////////////////
template <typename X, typename Y>
void preluCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                       const hipStream_t *stream, const void *vx, const LongType *xShapeInfo, const void *vy,
                       const LongType *yShapeInfo, void *vz) {
  preluCuda<X, Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz);
  sd::DebugHelper::checkGlobalErrorCode("prelu  failed");

}

///////////////////////////////////////////////////////////////////
void prelu(LaunchContext *context, NDArray *input, NDArray *alpha, NDArray *output) {
  PointersManager manager(context, "prelu");

  dim3 launchDims = getLaunchDims("prelu");

  const auto xType = input->dataType();
  const auto yType = alpha->dataType();

  NDArray::prepareSpecialUse({output}, {&input, &alpha});
  BUILD_SINGLE_SELECTOR_TWICE(
      xType, preluCudaLauncher,
      (launchDims.x, launchDims.y, launchDims.z, context->getCudaStream(), input->specialBuffer(),
          input->specialShapeInfo(), alpha->specialBuffer(), alpha->specialShapeInfo(), output->specialBuffer()),
      SD_FLOAT_TYPES);
  NDArray::registerSpecialUse({output}, {&input, &alpha});

  manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template <typename X, typename Y>
void SD_KERNEL preluBPCuda(const void *vIn, const LongType *inShapeInfo, const void *vAlpha,
                           const LongType *alphaShapeInfo, const void *vdLdO, const LongType *dLdOShapeInfo,
                           void *vdLdI, const LongType *dLdIShapeInfo, void *vdLdA,
                           const LongType *dLdAShapeInfo) {
  const auto in = reinterpret_cast<const X *>(vIn);
  const auto alpha = reinterpret_cast<const Y *>(vAlpha);
  const auto dLdO = reinterpret_cast<const Y *>(vdLdO);
  auto dLdI = reinterpret_cast<Y *>(vdLdI);
  auto dLdA = reinterpret_cast<Y *>(vdLdA);

  __shared__ LongType inLen, totalThreads;
  __shared__ int inRank, alphaRank;
  __shared__ const LongType *inShape;
  __shared__ const LongType *inStride;
  __shared__ const LongType *dLdOStride;
  __shared__ const LongType *dLdIStride;
  __shared__ const LongType *alphaStride;
  __shared__ const LongType *dLdAStride;

  if (threadIdx.x == 0) {
    inLen = shape::length(inShapeInfo);
    totalThreads = gridDim.x * blockDim.x;

    inRank = shape::rank(inShapeInfo);
    alphaRank = shape::rank(alphaShapeInfo);

    // Cache shapes and strides
    inShape = shape::shapeOf(inShapeInfo);
    inStride = shape::stride(inShapeInfo);
    dLdOStride = shape::stride(dLdOShapeInfo);
    dLdIStride = shape::stride(dLdIShapeInfo);
    alphaStride = shape::stride(alphaShapeInfo);
    dLdAStride = shape::stride(dLdAShapeInfo);
  }
  __syncthreads();

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  LongType coords[SD_MAX_RANK];

  for (int i = tid; i < inLen; i += totalThreads) {
    INDEX2COORDS(i, inRank, inShape, coords);

    LongType inOffset, dLdOOffset, dLdIOffset;
    COORDS2INDEX(inRank, inStride, coords, inOffset);
    COORDS2INDEX(inRank, dLdOStride, coords, dLdOOffset);
    COORDS2INDEX(inRank, dLdIStride, coords, dLdIOffset);

    const auto xVal = in[inOffset];
    const auto grO = dLdO[dLdOOffset];

    if (xVal < 0) {
      for (LongType j = 0; j < alphaRank; ++j)
        if (alphaShapeInfo[j + 1] == 1) coords[j + 1] = 0;

      LongType alphaOffset, dLdAOffset;
      COORDS2INDEX(alphaRank, alphaStride, coords + 1, alphaOffset);
      COORDS2INDEX(alphaRank, dLdAStride, coords + 1, dLdAOffset);

      dLdI[dLdIOffset] = grO * alpha[alphaOffset];

      math::atomics::sd_atomicAdd<Y>(&dLdA[dLdAOffset], static_cast<Y>(grO * xVal));
    } else {
      dLdI[dLdIOffset] = grO;
    }
  }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
void SD_HOST preluBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                                 const hipStream_t *stream, const void *vIn, const LongType *inShapeInfo,
                                 const void *vAlpha, const LongType *alphaShapeInfo, const void *vdLdO,
                                 const LongType *dLdOShapeInfo, void *vdLdI, const LongType *dLdIShapeInfo,
                                 void *vdLdA, const LongType *dLdAShapeInfo) {
  preluBPCuda<X, Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(
      vIn, inShapeInfo, vAlpha, alphaShapeInfo, vdLdO, dLdOShapeInfo, vdLdI, dLdIShapeInfo, vdLdA, dLdAShapeInfo);
  sd::DebugHelper::checkGlobalErrorCode("prelu bp failed");

}

//////////////////////////////////////////////////////////////////////////
void preluBP(LaunchContext *context, NDArray *input, NDArray *alpha, NDArray *dLdO, NDArray *dLdI,
             NDArray *dLdA) {
  dLdA->nullify();

  PointersManager manager(context, "preluBP");

  dim3 launchDims = getLaunchDims("prelu");

  const auto xType = input->dataType();
  const auto zType = alpha->dataType();

  NDArray::prepareSpecialUse({dLdI, dLdA}, {input, alpha, dLdO});
  BUILD_SINGLE_SELECTOR_TWICE(
      xType, preluBPCudaLauncher,
      (launchDims.x, launchDims.y, launchDims.z, context->getCudaStream(), input->specialBuffer(),
          input->specialShapeInfo(), alpha->specialBuffer(), alpha->specialShapeInfo(), dLdO->specialBuffer(),
          dLdO->specialShapeInfo(), dLdI->specialBuffer(), dLdI->specialShapeInfo(), dLdA->specialBuffer(),
          dLdA->specialShapeInfo()),
      SD_FLOAT_TYPES);
  NDArray::registerSpecialUse({&dLdI, &dLdA}, {input, alpha, dLdO});

  manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template <typename T>
SD_DEVICE void softMaxForVectorCuda(const void *vx, const LongType *xShapeInfo, void *vz,
                                    const LongType *zShapeInfo) {
  auto inBuff = reinterpret_cast<const T *>(vx);
  auto outBuff = reinterpret_cast<T *>(vz);

  __shared__ T shmemMax;
  __shared__ T shmemSum;
  __shared__ LongType tadLen;
  __shared__ int xRank;
  __shared__ int zRank;
  __shared__ const LongType *xShape;
  __shared__ const LongType *xStride;
  __shared__ const LongType *zShape;
  __shared__ const LongType *zStride;

  if (threadIdx.x == 0) {
    tadLen = shape::length(xShapeInfo);
    shmemMax = -DataTypeUtils::max<T>();
    shmemSum = 0.f;

    // Cache ranks
    xRank = shape::rank(xShapeInfo);
    zRank = shape::rank(zShapeInfo);

    // Cache shapes and strides
    xShape = shape::shapeOf(xShapeInfo);
    xStride = shape::stride(xShapeInfo);
    zShape = shape::shapeOf(zShapeInfo);
    zStride = shape::stride(zShapeInfo);
  }
  __syncthreads();

  T max = -DataTypeUtils::max<T>();
  T sum = 0.f;

  LongType xCoords[SD_MAX_RANK];
  LongType xOffset;

  // Calculate max using cached values
  for (LongType j = 0; j < tadLen; ++j) {
    INDEX2COORDS(j, xRank, xShape, xCoords);
    COORDS2INDEX(xRank, xStride, xCoords, xOffset);
    max = math::sd_max<T>(max, inBuff[xOffset]);
  }

  LongType zCoords[SD_MAX_RANK];
  LongType zOffset;

  // Calculate exp(x - max) and sum using cached values
  for (LongType j = 0; j < tadLen; ++j) {
    INDEX2COORDS(j, xRank, xShape, xCoords);
    COORDS2INDEX(xRank, xStride, xCoords, xOffset);
    T temp = math::sd_exp<T, T>(inBuff[xOffset] - max);
    INDEX2COORDS(j, zRank, zShape, zCoords);
    COORDS2INDEX(zRank, zStride, zCoords, zOffset);
    outBuff[zOffset] = temp;
    sum += temp;
  }

  // Final division step using cached values
  for (LongType j = 0; j < tadLen; ++j) {
    INDEX2COORDS(j, zRank, zShape, zCoords);
    COORDS2INDEX(zRank, zStride, zCoords, zOffset);
    outBuff[zOffset] /= sum;
  }
}

template <typename T>
void SD_KERNEL softMaxForVectorCudaGlobal(const void *vx, const LongType *xShapeInfo, void *vz,
                                          const LongType *zShapeInfo, LongType numOfSubArrs) {
  softMaxForVectorCuda<T>(vx, xShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
template <typename T>
void softMaxForVectorCudaLauncher(const hipStream_t *stream, const void *vx, const LongType *xShapeInfo, void *vz,
                                  const LongType *zShapeInfo, LongType numTads) {

  softMaxForVectorCudaGlobal<T><<<1, SD_CUDA_BLOCK_SIZE, 1024, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, numTads);
  sd::DebugHelper::checkGlobalErrorCode("softmax  failed");

}

///////////////////////////////////////////////////////////////////

template <typename T>
SD_KERNEL void softmaxEws1Kernel(const T *input, const LongType *inputOffsets, T *output,
                                 const LongType *outputOffsets,
                                 LongType numOfSubArrs, LongType tadLen) {
  int i = blockIdx.x;  // Each block handles one TAD

  if (i >= numOfSubArrs) return;  // Out-of-bounds check for TADs

  auto inBuff = input + inputOffsets[i];
  auto outBuff = output + outputOffsets[i];

  __shared__ T shmemMax;
  __shared__ T shmemSum;

  if (threadIdx.x == 0) {
    shmemMax = -DataTypeUtils::max<T>();
    shmemSum = 0.f;
  }
  __syncthreads();


  // Calculate max
  for (LongType j = threadIdx.x; j < tadLen; j+= gridDim.x) {
    math::atomics::sd_atomicMax(&shmemMax, inBuff[j]);
  }
  __syncthreads();

  // Calculate exp(x - max) and sum
  for (LongType j = threadIdx.x; j < tadLen; j += gridDim.x) {
    T temp = math::sd_exp<T, T>(inBuff[j] - shmemMax);
    outBuff[j] = temp;
    math::atomics::sd_atomicAdd(&shmemSum, temp);
  }
  __syncthreads();

  // Final division step
  for (LongType j = threadIdx.x; j < tadLen; j += blockDim.x) {
    outBuff[j] /= shmemSum;
  }


}
template <typename T>
SD_KERNEL static void softMaxCuda(const void *vx, const LongType *xTadShapeInfo, const LongType *xOffsets,
                                  void *vz, const LongType *zTadShapeInfo, const LongType *zOffsets, LongType numTads) {
  int i = blockIdx.x;
  if(i >= numTads) return;

  const auto x = reinterpret_cast<const T *>(vx);
  auto z = reinterpret_cast<T *>(vz);

  const auto *xTad = x + xOffsets[blockIdx.x];
  auto *zTad = z + zOffsets[blockIdx.x];
  softMaxForVectorCuda<T>(xTad, xTadShapeInfo, zTad, zTadShapeInfo);
}

///////////////////////////////////////////////////////////////////

template <typename T>
static void softMaxEws1CudaLauncher(const int blocksPerGrid,
                                    const int threadsPerBlock,
                                    const int sharedMem,
                                    const hipStream_t *stream,
                                    const void *vx, const LongType *xOffsets, void *vz,
                                    const LongType *zOffsets, LongType numTads, LongType tadLength) {



  auto reCastInputs = reinterpret_cast<const T *>(vx);
  auto reCastOutputs = reinterpret_cast<T *>(vz);
  softmaxEws1Kernel<T>
  <<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(reCastInputs,
                                                           xOffsets,
                                                           reCastOutputs,
                                                           zOffsets,
                                                           numTads,
                                                           tadLength);
  sd::DebugHelper::checkGlobalErrorCode("softmaxews  failed");

}

template <typename T>
static void softMaxCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                                const hipStream_t *stream, const void *vx, const LongType *xTadShapeInfo,
                                const LongType *xOffsets, void *vz, const LongType *zTadShapeInfo,
                                const LongType *zOffsets, LongType numTads) {


  softMaxCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xTadShapeInfo, xOffsets, vz, zTadShapeInfo,
                                                                         zOffsets ,numTads);
  sd::DebugHelper::checkGlobalErrorCode("softmax  failed");

}

//////////////////////////////////////////////////////////////////////////
void softmax(LaunchContext *context, NDArray *input, NDArray *output, const int dimension) {
  const int rank = input->rankOf();

  PointersManager manager(context, "helpers::softmax");

  if (input->isVector()) {
    if (rank == 1 || input->sizeAt(dimension) != 1) {
      NDArray::prepareSpecialUse({output}, {input});
      BUILD_SINGLE_SELECTOR(input->dataType(), softMaxForVectorCudaLauncher,
                            (context->getCudaStream(), input->specialBuffer(), input->specialShapeInfo(),
                                output->specialBuffer(), output->specialShapeInfo(),1),
                            SD_FLOAT_TYPES);
      NDArray::registerSpecialUse({output}, {input});
    } else
      *output = 1.;
  } else {
    auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), {dimension});
    auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {dimension});

    dim3 softmaxDims = getSoftmaxDims(packZ->numberOfTads());


    NDArray::prepareSpecialUse({output}, {input});
    BUILD_SINGLE_SELECTOR(input->dataType(), softMaxCudaLauncher,
                          (softmaxDims.x, softmaxDims.y,
                              softmaxDims.z,
                              context->getCudaStream(),
                              input->specialBuffer(),
                              packX->specialShapeInfo(),
                              packX->specialOffsets(), output->specialBuffer(),
                              packZ->specialShapeInfo(),
                              packZ->specialOffsets(),packX->numberOfTads()),
                          SD_FLOAT_TYPES);
    NDArray::registerSpecialUse({output}, {input});

  }

  manager.synchronize();

  output->tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template <typename T>
void SD_KERNEL logSoftMaxForVectorCuda(const void *vx, const LongType *xzShapeInfo, void *vz) {
  // logic of this kernel is based on assumption gridDim = 1

  const auto x = reinterpret_cast<const T *>(vx);
  auto z = reinterpret_cast<T *>(vz);

  __shared__ LongType len;
  __shared__ int numOfIters;
  __shared__ int xzRank;
  __shared__ const LongType *xzShape;
  __shared__ const LongType *xzStride;
  __shared__ T shmem[SD_CUDA_BLOCK_SIZE];

  if (threadIdx.x == 0) {
    len = shape::length(xzShapeInfo);
    numOfIters = (len + blockDim.x - 1) / blockDim.x;  // ceil (len / blockDim.x)

    // Cache rank, shape and stride information
    xzRank = shape::rank(xzShapeInfo);
    xzShape = shape::shapeOf(xzShapeInfo);
    xzStride = shape::stride(xzShapeInfo);
  }
  __syncthreads();

  T temp = -DataTypeUtils::max<T>();  // set start value to compare with at first iteration, FIXME: what if T is unsigned ??

  // ************ evaluate max element in input array x ************ //
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    if (elemIdx < len) {
      LongType offset;
      sd::LongType coords[SD_MAX_RANK];
      INDEX2COORDS(elemIdx, xzRank, xzShape, coords);
      COORDS2INDEX(xzRank, xzStride, coords, offset);
      shmem[threadIdx.x] = (threadIdx.x != 0) ? x[offset] : math::sd_max<T>(x[offset], temp);  // take into account max element evaluated on previous iteration and stored in temp
    } else {
      shmem[threadIdx.x] = -DataTypeUtils::max<T>();  // FIXME: what if T is unsigned ??
    }

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
      if (threadIdx.x < s) shmem[threadIdx.x] = math::sd_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
      __syncthreads();
    }

    temp = shmem[0];  // save max value calculated at current iteration
  }

  const T max = temp;
  temp = 0;

  // ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************
  // at the same time evaluate sum of exponents, sum will be stored in shmem[0]
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    if (elemIdx < len) {
      LongType offset;
      sd::LongType coords[SD_MAX_RANK];
      INDEX2COORDS(elemIdx, xzRank, xzShape, coords);
      COORDS2INDEX(xzRank, xzStride, coords, offset);
      z[offset] = math::sd_exp<T, T>(x[offset] - max);
      shmem[threadIdx.x] = (threadIdx.x != 0) ? z[offset] : (z[offset] + temp);  // take into account sum element evaluated on previous iteration and stored in temp
    } else {
      shmem[threadIdx.x] = 0;
    }

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
      if (threadIdx.x < s) shmem[threadIdx.x] += shmem[threadIdx.x + s];
      __syncthreads();
    }

    temp = shmem[0];  // save sum calculated at current iteration
  }

  // ************ evaluate log(z[offset] / sum)  ************ //
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    if (elemIdx < len) {  // Added bounds check that was missing in original
      LongType offset;
      sd::LongType coords[SD_MAX_RANK];
      INDEX2COORDS(elemIdx, xzRank, xzShape, coords);
      COORDS2INDEX(xzRank, xzStride, coords, offset);
      z[offset] = math::sd_log<T, T>(z[offset] / shmem[0]);
    }
  }
}
///////////////////////////////////////////////////////////////////
template <typename T>
void logSoftMaxForVectorCudaLauncher(const hipStream_t *stream, const void *vx, const LongType *xzShapeInfo,
                                     void *vz) {
  dim3 launchDims = getLaunchDims("softmax");
  logSoftMaxForVectorCuda<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xzShapeInfo, vz);
  sd::DebugHelper::checkGlobalErrorCode("logsoftmax  failed");

}

//////////////////////////////////////////////////////////////////////////
void logSoftmax(LaunchContext *context, NDArray *input, NDArray *output, const int dimension) {
  if (!input->isActualOnDeviceSide()) input->syncToDevice();
  const int rank = input->rankOf();

  if (input->isVector()) {
    if (rank == 1 || input->sizeAt(dimension) != 1) {
      BUILD_SINGLE_SELECTOR(
          input->dataType(), logSoftMaxForVectorCudaLauncher,
          (context->getCudaStream(), input->specialBuffer(), input->specialShapeInfo(), output->specialBuffer()),
          SD_FLOAT_TYPES);
      input->tickReadDevice();
    } else
      *output = 0.;
  } else {
    std::vector<LongType> dim = {static_cast<LongType>(dimension)};
    auto maxAlongDim = const_cast<NDArray *>(input)->reduceAlongDimension(reduce::Max, &dim, true);
    auto inputMinusMax = *input - maxAlongDim;
    inputMinusMax.applyTransform(transform::Exp, output);  // output contains exponents temporarily
    auto sumAlongDim = output->reduceAlongDimension(reduce::Sum, &dim, true);
    *output /= sumAlongDim;
    output->applyTransform(transform::Log, output);
    input->tickReadDevice();
  }

  PointersManager manager(context, "helpers::logSoftmax");
  manager.synchronize();

  output->tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template <typename T>
void SD_KERNEL softMaxDerivForVectorCuda(const void *vx, const LongType *xzShapeInfo, void *vz) {
  // logic of this kernel is based on assumption gridDim = 1

  const auto x = reinterpret_cast<const T *>(vx);
  auto z = reinterpret_cast<T *>(vz);

  __shared__ LongType len;
  __shared__ int numOfIters;
  __shared__ int xzRank;
  __shared__ const LongType *xzShape;
  __shared__ const LongType *xzStride;
  __shared__ T shmem[SD_CUDA_BLOCK_SIZE];

  if (threadIdx.x == 0) {
    len = shape::length(xzShapeInfo);
    numOfIters = (len + blockDim.x - 1) / blockDim.x;  // ceil (len / blockDim.x)

    // Cache rank, shape and stride information
    xzRank = shape::rank(xzShapeInfo);
    xzShape = shape::shapeOf(xzShapeInfo);
    xzStride = shape::stride(xzShapeInfo);
  }
  __syncthreads();

  T temp = -DataTypeUtils::max<T>();  // set start value to compare with at first iteration, FIXME: what if T is unsigned ??

  // ************ evaluate max element in input array x ************ //
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    if (elemIdx < len) {
      LongType offset;
      sd::LongType coords[SD_MAX_RANK];
      INDEX2COORDS(elemIdx, xzRank, xzShape, coords);
      COORDS2INDEX(xzRank, xzStride, coords, offset);
      shmem[threadIdx.x] = (threadIdx.x != 0) ? x[offset] : math::sd_max<T>(x[offset], temp);  // take into account max element evaluated on previous iteration and stored in temp
    } else {
      shmem[threadIdx.x] = -DataTypeUtils::max<T>();  // FIXME: what if T is unsigned ??
    }

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
      if (threadIdx.x < s) shmem[threadIdx.x] = math::sd_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
      __syncthreads();
    }

    temp = shmem[0];  // save max value calculated at current iteration
  }

  const T max = temp;
  temp = 0;

  // ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************
  // at the same evaluate sum of exponents, sum will be stored in shmem[0]
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    if (elemIdx < len) {
      LongType offset;
      sd::LongType coords[SD_MAX_RANK];
      INDEX2COORDS(elemIdx, xzRank, xzShape, coords);
      COORDS2INDEX(xzRank, xzStride, coords, offset);
      z[offset] = math::sd_exp<T, T>(x[offset] - max);
      shmem[threadIdx.x] = (threadIdx.x != 0) ? z[offset] : (z[offset] + temp);  // take into account sum element evaluated on previous iteration and stored in temp
    } else {
      shmem[threadIdx.x] = 0;
    }

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
      if (threadIdx.x < s) shmem[threadIdx.x] += shmem[threadIdx.x + s];
      __syncthreads();
    }

    temp = shmem[0];  // save sum calculated at current iteration
  }

  // ************ evaluate (z[offset] / sum) and derivative z[offset] = z[offset] * (1 - z[offset]) ************ //
  for (int i = 0; i < numOfIters; ++i) {
    const LongType elemIdx = i * blockDim.x + threadIdx.x;
    if (elemIdx >= len) continue;

    LongType offset;
    sd::LongType coords[SD_MAX_RANK];
    INDEX2COORDS(elemIdx, xzRank, xzShape, coords);
    COORDS2INDEX(xzRank, xzStride, coords, offset);
    z[offset] /= shmem[0];
    z[offset] *= (1.f - z[offset]);  // derivative
  }
}
///////////////////////////////////////////////////////////////////
template <typename T>
void softMaxDerivForVectorCudaLauncher(const hipStream_t *stream, const void *vx, const LongType *xzShapeInfo,
                                       void *vz) {
  dim3 launchDims = getLaunchDims("softmax");

  softMaxDerivForVectorCuda<T><<<launchDims.x,launchDims.y, launchDims.z, *stream>>>(vx, xzShapeInfo, vz);
  sd::DebugHelper::checkGlobalErrorCode("softmax derivative  failed");

}

///////////////////////////////////////////////////////////////////
void softmaxDerivative(LaunchContext *context, NDArray *input, NDArray *output, const int dimension) {
  if (!input->isActualOnDeviceSide()) input->syncToDevice();
  const int rank = input->rankOf();
  LongType temp;

  if (shape::isCommonVector(input->shapeInfo(), temp)) {
    BUILD_SINGLE_SELECTOR(
        input->dataType(), softMaxDerivForVectorCudaLauncher,
        (context->getCudaStream(), input->specialBuffer(), input->specialShapeInfo(), output->specialBuffer()),
        SD_FLOAT_TYPES);
    input->tickReadDevice();
  } else {
    std::vector<LongType> dim = {static_cast<LongType>(dimension)};
    auto maxAlongDim = const_cast<NDArray *>(input)->reduceAlongDimension(reduce::Max, &dim, true);
    auto inputMinusMax = *input - maxAlongDim;
    inputMinusMax.applyTransform(transform::Exp, output);  // output contains exponents temporarily
    auto sumAlongDim = output->reduceAlongDimension(reduce::Sum, &dim, true);
    *output /= sumAlongDim;
    *output *= (1.f - *output);  // derivative
    input->tickReadDevice();
  }

  PointersManager manager(context, "helpers::softmaxDerivative");
  manager.synchronize();

  output->tickWriteDevice();
}

template <typename T>
void thresholdRelu_(NDArray const *input, double threshold, NDArray *output) {
  auto routine = LAMBDA_T(_x, threshold) { return _x > (T)threshold ? _x : (T)0.f; };
  const_cast<NDArray *>(input)->applyLambda(routine, output);
}

void thresholdRelu(LaunchContext *context, NDArray *input, double threshold, NDArray *output) {
  BUILD_SINGLE_SELECTOR(input->dataType(), thresholdRelu_, (input, threshold, output), SD_FLOAT_TYPES);
}

template <typename T>
void thresholdReluDerivative_(NDArray *input, double theta, NDArray *dLdO, NDArray *output) {
  auto derivative = LAMBDA_TT(_x, grO, theta) {
    if (_x > theta)
      return grO;
    else
      return static_cast<T>(0);
  };

  input->applyPairwiseLambda(dLdO, derivative, output);
}

void thresholdReluDerivative(LaunchContext *context, NDArray *input, double threshold, NDArray *dLdO,
                             NDArray *output) {
  BUILD_SINGLE_SELECTOR(input->dataType(), thresholdReluDerivative_, (input, threshold, dLdO, output), SD_FLOAT_TYPES);
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
