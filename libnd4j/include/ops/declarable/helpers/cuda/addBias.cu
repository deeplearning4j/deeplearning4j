#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <helpers/PointersManager.h>
#include <ops/declarable/helpers/addBias.h>

#include "execution/cuda/LaunchDims.h"


namespace sd {
namespace ops {
namespace helpers {

//////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
SD_KERNEL static void addBiasCuda(const void* vx, const LongType* xShapeInfo, const void* vy,
                                  const LongType* yShapeInfo, void* vz, const LongType* zShapeInfo,
                                  const bool isNCHW) {
  // bias [oC]

  // if(input_rank == 4)
  // input and output have same shapes: [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW)
  // if(input_rank == 5)
  // input and output have same shapes: [bS, oD, oH, oW, oC] (NHWC) or [bS, oD, oC, oH, oW] (NCHW)

  const X* x = reinterpret_cast<const X*>(vx);
  const Y* y = reinterpret_cast<const Y*>(vy);
  X* z = reinterpret_cast<X*>(vz);

  __shared__ LongType rank, channelPosition, posOfNonUnityDim;
  __shared__ LongType len, *sharedMem;
  __shared__ bool xzSameOffsets, xzAreSame;
  __shared__ const LongType *xShape;
  __shared__ const LongType *xStride;
  __shared__ const LongType *zStride;
  __shared__ const LongType *yStride;

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char shmem[];
    sharedMem = reinterpret_cast<LongType*>(shmem);

    rank = shape::rank(xShapeInfo);  // xRank == zRank
    xzSameOffsets = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
    len = shape::length(xShapeInfo);
    channelPosition = isNCHW ? 1 : rank - 1;  // second or last
    xzAreSame = x == z;

    // Cache shapes and strides
    xShape = shape::shapeOf(xShapeInfo);
    xStride = shape::stride(xShapeInfo);
    zStride = shape::stride(zShapeInfo);
    yStride = shape::stride(yShapeInfo);

    shape::isCommonVector(yShapeInfo, posOfNonUnityDim);
  }
  __syncthreads();

  auto coords = sharedMem + threadIdx.x * rank;

  for (LongType i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
    INDEX2COORDS(i, rank, xShape, coords);

    LongType xOffsets;
    COORDS2INDEX(rank, xStride, coords, xOffsets);
    LongType zOffsets;
    COORDS2INDEX(rank, zStride, coords, zOffsets);
    LongType yOffsets = coords[channelPosition] * yStride[posOfNonUnityDim];

    if (xzAreSame)
      z[zOffsets] += static_cast<X>(y[yOffsets]);
    else
      z[zOffsets] = static_cast<X>(x[xOffsets]) + static_cast<X>(y[yOffsets]);
  }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
static void addBiasCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem,
                                const hipStream_t* stream, const void* vx, const LongType* xShapeInfo,
                                const void* vy, const LongType* yShapeInfo, void* vz,
                                const LongType* zShapeInfo, const bool isNCHW) {
  addBiasCuda<X, Y>
  <<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, isNCHW);
  sd::DebugHelper::checkGlobalErrorCode("addbias  failed");

}

template <typename X, typename Y>
SD_KERNEL static void addBias2DCuda(const void* vx, const void* vy, void* vz, uint32_t blocks, uint32_t length) {
  auto y = reinterpret_cast<const Y*>(vy);

  for (uint32_t b = blockIdx.x; b < blocks; b += gridDim.x) {
    auto x = reinterpret_cast<const X*>(vx) + length * b;
    auto z = reinterpret_cast<X*>(vz) + length * b;

    for (uint32_t e = threadIdx.x; e < length; e += blockDim.x) {
      z[e] = x[e] + y[e];
    }
  }
}

template <typename X, typename Y>
static void addBias2DCudaLauncher(const hipStream_t* stream, const void* vx, const void* vy, void* vz, uint32_t blocks,
                                  uint32_t length) {
  dim3 dims = getAddBiasDims(2, 2);

  addBias2DCuda<X, Y><<<dims.x, dims.y, dims.z, *stream>>>(vx, vy, vz, blocks, length);
  sd::DebugHelper::checkGlobalErrorCode("addbias 2d  failed");

}

//////////////////////////////////////////////////////////////////////////
void addBias(graph::Context& block, NDArray& input, NDArray& bias, NDArray& output, const bool isNCHW) {
  PointersManager manager(block.launchContext(), "addBias");
  NDArray::prepareSpecialUse({&output}, {&input, &bias});

  if (input.rankOf() == 2 && bias.rankOf() == 1 && input.ordering() == 'c' && output.ordering() == 'c' &&
  input.sizeAt(1) == bias.sizeAt(0)) {
    BUILD_DOUBLE_SELECTOR(input.dataType(), bias.dataType(), addBias2DCudaLauncher,
                          (block.launchContext()->getCudaStream(), input.specialBuffer(), bias.specialBuffer(),
                              output.specialBuffer(), input.sizeAt(0), bias.sizeAt(0)),
                          SD_FLOAT_TYPES, SD_FLOAT_TYPES);
  } else {
    // default case
    dim3 dims = getAddBiasDims(input.rankOf(), input.rankOf());
    BUILD_DOUBLE_SELECTOR(input.dataType(), bias.dataType(), addBiasCudaLauncher,
                          (dims.x, dims.y, dims.z, block.launchContext()->getCudaStream(),
                              input.specialBuffer(), input.specialShapeInfo(), bias.specialBuffer(),
                              bias.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), isNCHW),
                          SD_FLOAT_TYPES, SD_FLOAT_TYPES);
  }
  NDArray::registerSpecialUse({&output}, {&input, &bias});
  manager.synchronize();
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
