#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/
//
// @author AbdelRauf
//

#include <array/NDArrayFactory.h>
#include <ops/declarable/OpRegistrator.h>

#include "cudnnUtils.h"

namespace sd {
namespace ops {
namespace platforms {

// our implementation designed for 1 physical layer
constexpr int numLayers = 1;

// we will copy without using hipdnnGetRNNLinLayerMatrixParams : 1 pseudo layer , isBidirectional : 2 pseudo layer
void copyWeights(const hipStream_t &stream, bool isBidirectional, uint8_t *weightsSpace, size_t weightsSize,
                 uint8_t *inputWeightsData, uint8_t *recurrentWeightsData, uint8_t *biasesData, LongType inputSize,
                 int hiddenSize, int dataTypeSize) {
  int pseudo_layer_count = isBidirectional ? 2 : 1;
  uint8_t *wptr = weightsSpace;
  auto wEnd = wptr + weightsSize;

  // copy size for 1 full pseudo layer
  // in bidirectional 1 layer consist of 2 pseduo layers
  auto input_pseudo_size = 4 * inputSize * hiddenSize * dataTypeSize;
  auto hidden_pseudo_size = 4 * hiddenSize * hiddenSize * dataTypeSize;
  for (LongType i = 0; i < pseudo_layer_count; i++) {
    if (wptr + input_pseudo_size + hidden_pseudo_size > wEnd) return;
    // copy input weights
    if (inputWeightsData) {
      hipMemcpyAsync(wptr, inputWeightsData, input_pseudo_size, hipMemcpyDeviceToDevice, stream);
      inputWeightsData += input_pseudo_size;
    }
    wptr += input_pseudo_size;
    // copy recurrent weights
    if (recurrentWeightsData) {
      hipMemcpyAsync(wptr, recurrentWeightsData, hidden_pseudo_size, hipMemcpyDeviceToDevice, stream);
      recurrentWeightsData += hidden_pseudo_size;
    }
    wptr += hidden_pseudo_size;
  }

  // copy bias first 4
  auto bias_size = 4 * hiddenSize * dataTypeSize;
  for (int i = 0; i < pseudo_layer_count; i++) {
    // refill first 4 biases
    if (biasesData && wptr + bias_size < wEnd) {
      hipMemcpyAsync(wptr, biasesData, bias_size, hipMemcpyDeviceToDevice, stream);
      biasesData += bias_size;
    }
    wptr += bias_size;
    // refill next 4 with zeros
    if (wptr + bias_size < wEnd) {
      hipMemsetAsync(wptr, 0, bias_size, stream);
      wptr += bias_size;
    }
  }
  // memset the rest
  if (wEnd - wptr) hipMemsetAsync(wptr, 0, wEnd - wptr, stream);
}

void cudnn_rnn_old(LaunchContext *contextPtr, int dataFormat, NDArray *input, NDArray *inputWeights,
                   NDArray *recurrentWeights, NDArray *biases, NDArray *prevAct, NDArray *prevMemCell,
                   NDArray *outputActivations, NDArray *finalTimeStepActivations, NDArray *finalMemCellState,
                   LongType maxSeqLength, LongType batchSize, LongType inputSize, LongType hiddenSize, double cellClip,
                   bool isBidirectional) {
  sd_debug("cudnn rnn api %s \n", "v6");

  bool training = false;
  hipdnnHandle_t handle = *(reinterpret_cast<hipdnnHandle_t *>(contextPtr->getCuDnnHandle()));

  auto stream = *(contextPtr->getCudaStream());
  CHECK_CUDNN_FAILURE_MSG(STRINGIZE(hipdnnSetStream), hipdnnSetStream(handle, stream));

  CudnnTensorList xDescList(maxSeqLength);
  CudnnTensorList yDescList(maxSeqLength);

  auto cudnnType = cudnnDataType(input->dataType());
  auto dataTypeSize = input->sizeOfT();

  CudnnTensor hxDesc, cxDesc, hyDesc, cyDesc;

  constexpr int rankOf = 3;
  const int numDirections = isBidirectional ? 2 : 1;

  const int dimsX[rankOf] = {static_cast<int>(batchSize), static_cast<int>(inputSize), 1};
  const int stridesX[rankOf] = {static_cast<int>(inputSize), 1, 1};

  const int dimsY[rankOf] = {static_cast<int>(batchSize), static_cast<int>(hiddenSize * numDirections), 1};
  const int stridesY[rankOf] = {static_cast<int>(hiddenSize * numDirections), 1, 1};

  const int dimC[rankOf] = {static_cast<int>(numLayers * numDirections), static_cast<int>(batchSize), static_cast<int>(hiddenSize)};
  const int strideC[rankOf] = {static_cast<int>(batchSize * hiddenSize), static_cast<int>(hiddenSize), 1};
  for (int i = 0; i < maxSeqLength; i++) {
    xDescList.set(i, cudnnType, rankOf, dimsX, stridesX);
    yDescList.set(i, cudnnType, rankOf, dimsY, stridesY);
  }

  auto xDesc0 = xDescList.get(0);

  hxDesc.set(cudnnType, rankOf, dimC, strideC);
  cxDesc.set(cudnnType, rankOf, dimC, strideC);
  hyDesc.set(cudnnType, rankOf, dimC, strideC);
  cyDesc.set(cudnnType, rankOf, dimC, strideC);

  PointersManager manager(contextPtr, __func__);
  // dropout section
  DropoutDesc dropoutDesc(nullptr);
  // dropout
  float dropout = 0;
  size_t sizeInBytes = 0;
  void *droupoutMem = nullptr;
  uint64_t seed = 1;  // seed
  if (dropout != 0) {
    dropoutDesc.create();
    CHECK_CUDNN_FAILURE_MSG(STRINGIZE(hipdnnDropoutGetStatesSize), hipdnnDropoutGetStatesSize(handle, &sizeInBytes));
    // allocate and set
    droupoutMem = manager.allocateDevMem(sizeInBytes);
    dropoutDesc.set(handle, dropout, droupoutMem, sizeInBytes, seed);
  }

  // RNN
  RnnDesc rnnDesc;
  hipdnnRNNMode_t rnnCellMode = HIPDNN_LSTM;
  hipdnnRNNAlgo_t algo = HIPDNN_RNN_ALGO_STANDARD;

  auto direction = isBidirectional ? HIPDNN_BIDIRECTIONAL : HIPDNN_UNIDIRECTIONAL;
  auto mathPrec = cudnnType;

  // Note: We will set some parameters manually
  constexpr auto inputMode = HIPDNN_LINEAR_INPUT;
  rnnDesc.setUsingOldAPI(handle, inputMode, direction, rnnCellMode, algo, mathPrec, hiddenSize, numLayers, dropoutDesc);
#if CUDNN_VERSION >= CUDNN_CLIPPING_API_VER
  if (cellClip > 0 && hipdnnGetVersion() >= CUDNN_CLIPPING_API_VER) {
    CHECK_CUDNN_FAILURE_MSG(STRINGIZE(cudnnRNNSetClip), cudnnRNNSetClip(handle, rnnDesc, CUDNN_RNN_CLIP_MINMAX,
                                                                        HIPDNN_PROPAGATE_NAN, -cellClip, cellClip));
  }
#endif
  // set up parameters
  size_t weightsSize = 0;
  CHECK_CUDNN_FAILURE_MSG(STRINGIZE(hipdnnGetRNNParamsSize),
                          hipdnnGetRNNParamsSize(handle, rnnDesc, xDesc0, &weightsSize, cudnnType));

  FilterDesc wDesc;
  int dimW[] = {static_cast<int>(weightsSize / dataTypeSize), 1, 1};

  wDesc.set(cudnnType, HIPDNN_TENSOR_NCHW, 3, dimW);
  // allocation
  void *weightsSpace = manager.allocateDevMem(weightsSize);

  size_t workSpaceSizeInBytes = 0;
  size_t reserveSpaceSizeInBytes = 0;

  CHECK_CUDNN_FAILURE_MSG(
      STRINGIZE(hipdnnGetRNNWorkspaceSize),
      hipdnnGetRNNWorkspaceSize(handle, rnnDesc, maxSeqLength, xDescList.getDescriptors(), &workSpaceSizeInBytes));

  void *workSpace = manager.allocateDevMem(workSpaceSizeInBytes);
  void *reserveSpace = nullptr;
  // training
  if (training) {
    CHECK_CUDNN_FAILURE_MSG(STRINGIZE(hipdnnGetRNNTrainingReserveSize),
                            hipdnnGetRNNTrainingReserveSize(handle, rnnDesc, maxSeqLength, xDescList.getDescriptors(),
                                                           &reserveSpaceSizeInBytes));
    reserveSpace = manager.allocateDevMem(reserveSpaceSizeInBytes);
  }

  NDArray::prepareSpecialUse({outputActivations, finalTimeStepActivations, finalMemCellState},
                             {input, inputWeights, recurrentWeights, biases, prevAct, prevMemCell});

  uint8_t *biasesData = biases ? (uint8_t *)biases->specialBuffer() : nullptr;
  auto prevActData = prevAct ? prevAct->specialBuffer() : nullptr;
  auto prevMemCellData = prevMemCell ? prevMemCell->specialBuffer() : nullptr;
  auto finalTimeStepActivationsData = finalTimeStepActivations ? finalTimeStepActivations->specialBuffer() : nullptr;
  auto finalMemCellStateData = finalMemCellState ? finalMemCellState->specialBuffer() : nullptr;

  // dimension 4*nOut implies order it, ft, c't, ot
  // input gate, forget gate, new gate, output gate, input gate, forget gate, new gate, output gate
  // Note: our weights should be transposed and duplicated with C order to match cudnn ones

  NDArray inputWeightsT, recurrentWeightsT;
  uint8_t *inputWeightsData = nullptr;
  uint8_t *recurrentWeightsData = nullptr;
  if (inputWeights) {
    inputWeightsT =
        inputWeights->rankOf() == 3 ? inputWeights->permute({0, 2, 1}, 0, false).dup('c') : inputWeights->transpose().dup('c');
    inputWeightsData = (uint8_t *)inputWeightsT.specialBuffer();
  }
  if (recurrentWeights) {
    recurrentWeightsT = recurrentWeights->rankOf() == 3 ? recurrentWeights->permute({0, 2, 1}, 0, false).dup('c')
                                                        : recurrentWeights->transpose().dup('c');
    recurrentWeightsData = (uint8_t *)recurrentWeightsT.specialBuffer();
  }

  // copy without hipdnnGetRNNLinLayerMatrixParams
  copyWeights(stream, isBidirectional, (uint8_t *)weightsSpace, weightsSize, inputWeightsData, recurrentWeightsData,
              biasesData, inputSize, hiddenSize, dataTypeSize);

  // permute based on dataformat
  NDArray *argX = input;
  NDArray *argOutput = outputActivations;
  NDArray permutedX, outputH;

  if (outputActivations != nullptr && (dataFormat != 0 || outputActivations->ordering() != 'c')) {
    outputH = NDArray('c', std::vector<LongType>{maxSeqLength, batchSize, (numDirections * hiddenSize)},
                      outputActivations->dataType(), contextPtr);
    argOutput = &outputH;
  }

  if (dataFormat == 1) {
    permutedX = input->permute({1, 0, 2}, 0, false).dup('c');
    argX = &permutedX;
  }

  auto xData = argX->specialBuffer();
  auto yData = argOutput ? argOutput->specialBuffer() : nullptr;

  if (training) {
    CHECK_CUDNN_FAILURE_MSG(
        STRINGIZE(hipdnnRNNForwardTraining),
        hipdnnRNNForwardTraining(handle, rnnDesc, (int)maxSeqLength, xDescList.getDescriptors(), xData, hxDesc,
                                prevActData, cxDesc, prevMemCellData, wDesc, weightsSpace, yDescList.getDescriptors(),
                                yData, hyDesc, finalTimeStepActivationsData, cyDesc, finalMemCellStateData, workSpace,
                                workSpaceSizeInBytes, reserveSpace, reserveSpaceSizeInBytes));
  } else {
    CHECK_CUDNN_FAILURE_MSG(
        STRINGIZE(hipdnnRNNForwardInference),
        hipdnnRNNForwardInference(handle, rnnDesc, (int)maxSeqLength, xDescList.getDescriptors(), xData, hxDesc,
                                 prevActData, cxDesc, prevMemCellData, wDesc, weightsSpace, yDescList.getDescriptors(),
                                 yData, hyDesc, finalTimeStepActivationsData, cyDesc, finalMemCellStateData, workSpace,
                                 workSpaceSizeInBytes));
  }

  // remap output
  if (outputActivations != nullptr && argOutput != outputActivations) {
    // refill output
    if (dataFormat == 1) {
      std::vector<sd::LongType> permute = {1,0,2};
      NDArray assign = argOutput->permute(permute, 0, false);
      outputActivations->assign(&assign);
    }
  }
  NDArray::registerSpecialUse({outputActivations, finalTimeStepActivations, finalMemCellState},
                              {input, inputWeights, recurrentWeights, biases, prevAct, prevMemCell});

  return;
}

#if CUDNN_VERSION >= CUDNN_NEW_RNN_API_VER

void cudnn_rnn_v8(LaunchContext *contextPtr, int dataFormat, NDArray *input, NDArray *seqLengthArray,
                  NDArray *inputWeights, NDArray *recurrentWeights, NDArray *biases, NDArray *prevAct,
                  NDArray *prevMemCell, NDArray *outputActivations, NDArray *finalTimeStepActivations,
                  NDArray *finalMemCellState, int maxSeqLength, int batchSize, int inputSize, int hiddenSize,
                  double cellClip, bool isBidirectional) {
  sd_debug("cudnn rnn api %s \n", "v8");
  // seqLengthArray should be int
  NDArray *argSeqNdArray = nullptr;
  NDArray seqArrIntData;
  if (seqLengthArray) {
    if (seqLengthArray->ews() == 1 && seqLengthArray->dataType() == INT32) {
      argSeqNdArray = seqLengthArray;
    } else {
      if (seqLengthArray->dataType() != INT32) {
        seqArrIntData = seqLengthArray->cast(INT32);
        if (seqArrIntData.ews() != 1) seqArrIntData = seqArrIntData.dup('c');
      } else {
        seqArrIntData = seqLengthArray->dup('c');
      }
      argSeqNdArray = &seqArrIntData;
    }
  } else {
    seqArrIntData = NDArray('c', std::vector<LongType>{batchSize}, INT32, contextPtr);
    seqArrIntData.assign(maxSeqLength);
    argSeqNdArray = &seqArrIntData;
  }
  PointersManager manager(contextPtr, __func__);
  bool training = false;
  hipdnnHandle_t handle = *(reinterpret_cast<hipdnnHandle_t *>(contextPtr->getCuDnnHandle()));
  auto stream = *(contextPtr->getCudaStream());
  CHECK_CUDNN_FAILURE_MSG(STRINGIZE(hipdnnSetStream), hipdnnSetStream(handle, stream));

  auto cudnnType = cudnnDataType(input->dataType());
  auto dataTypeSize = input->sizeOfT();

  CudnnTensor hDesc, cDesc;

  constexpr int rankOf = 3;
  const int numDirections = isBidirectional ? 2 : 1;

  const int dimC[rankOf] = {numLayers * numDirections, batchSize, hiddenSize};
  const int strideC[rankOf] = {batchSize * hiddenSize, hiddenSize, 1};

  hDesc.set(cudnnType, rankOf, dimC, strideC);
  cDesc.set(cudnnType, rankOf, dimC, strideC);

  // dropout section
  DropoutDesc dropoutDesc(nullptr);
  // dropout
  float dropout = 0;
  size_t sizeInBytes = 0;
  void *droupoutMem = nullptr;
  uint64_t seed = 1;  // seed
  if (dropout != 0) {
    dropoutDesc.create();
    CHECK_CUDNN_FAILURE_MSG(STRINGIZE(hipdnnDropoutGetStatesSize), hipdnnDropoutGetStatesSize(handle, &sizeInBytes));
    // allocate and set
    droupoutMem = manager.allocateDevMem(sizeInBytes);
    dropoutDesc.set(handle, dropout, droupoutMem, sizeInBytes, seed);
  }

  // RNN
  RnnDesc rnnDesc;
  hipdnnRNNMode_t rnnCellMode = HIPDNN_LSTM;
  hipdnnRNNAlgo_t algo = HIPDNN_RNN_ALGO_STANDARD;
  auto direction = isBidirectional ? HIPDNN_BIDIRECTIONAL : HIPDNN_UNIDIRECTIONAL;
  auto mathPrec = cudnnType;

  // Note: We will set some parameters manually. Some of them could be parameter in future
  constexpr auto inputMode = HIPDNN_LINEAR_INPUT;
  bool use_tensor_ops = false;  // could be parameter in future
#if CUDNN_VERSION >= CUDNN_NEW_RNN_API_VER
  hipdnnMathType_t mathType = use_tensor_ops ? HIPDNN_TENSOR_OP_MATH : CUDNN_FMA_MATH;
#else
  hipdnnMathType_t mathType = use_tensor_ops ? HIPDNN_TENSOR_OP_MATH : HIPDNN_DEFAULT_MATH;
#endif
  // disable projection
  int projSize = hiddenSize;
  hipdnnRNNBiasMode_t bias_mode = HIPDNN_RNN_WITH_BIAS;
  uint32_t aux_flags = CUDNN_RNN_PADDED_IO_ENABLED;

  rnnDesc.set(algo, rnnCellMode, bias_mode, direction, inputMode, cudnnType, mathPrec, mathType, inputSize, hiddenSize,
              projSize, numLayers, dropoutDesc, aux_flags);
  if (cellClip > 0) {
    CHECK_CUDNN_FAILURE_MSG(STRINGIZE(cudnnRNNSetClip), cudnnRNNSetClip(handle, rnnDesc, CUDNN_RNN_CLIP_MINMAX,
                                                                        HIPDNN_PROPAGATE_NAN, -cellClip, cellClip));
  }
  // set Data desc
  RnnDataDesc xDataDesc, yDataDesc;
  bool time_major = false;
  float padding_fill = 0.0f;
  auto hostSeqArr = bufferInHost<int>(*argSeqNdArray);
  cudnnRNNDataLayout_t layout =
      dataFormat == 0 ? CUDNN_RNN_DATA_LAYOUT_SEQ_MAJOR_UNPACKED : CUDNN_RNN_DATA_LAYOUT_BATCH_MAJOR_UNPACKED;
  xDataDesc.set(cudnnType, layout, maxSeqLength, batchSize, inputSize, hostSeqArr, (void *)&padding_fill);
  yDataDesc.set(cudnnType, layout, maxSeqLength, batchSize, hiddenSize * numDirections, hostSeqArr,
                (void *)&padding_fill);
  // set up parameters
  size_t weightsSize = 0;
  CHECK_CUDNN_FAILURE_MSG(STRINGIZE(cudnnGetRNNWeightSpaceSize),
                          cudnnGetRNNWeightSpaceSize(handle, rnnDesc, &weightsSize));

  // allocation
  void *weightsSpace = manager.allocateDevMem(weightsSize);

  // Set up work space and reserved memory
  void *workSpace = nullptr;
  void *reserveSpace = nullptr;

  size_t workSpaceSizeInBytes = 0;
  size_t reserveSpaceSizeInBytes = 0;

  cudnnForwardMode_t fwdMode = training ? CUDNN_FWD_MODE_TRAINING : CUDNN_FWD_MODE_INFERENCE;
  CHECK_CUDNN_FAILURE_MSG(
      STRINGIZE(cudnnGetRNNTempSpaceSizes),
      cudnnGetRNNTempSpaceSizes(handle, rnnDesc, fwdMode, xDataDesc, &workSpaceSizeInBytes, &reserveSpaceSizeInBytes));
  workSpace = manager.allocateDevMem(workSpaceSizeInBytes);
  // training
  if (training) {
    reserveSpace = manager.allocateDevMem(reserveSpaceSizeInBytes);
  }

  NDArray::prepareSpecialUse({outputActivations, finalTimeStepActivations, finalMemCellState},
                             {input, inputWeights, recurrentWeights, biases, prevAct, prevMemCell, argSeqNdArray});

  auto xData = input->specialBuffer();
  uint8_t *biasesData = biases ? (uint8_t *)biases->specialBuffer() : nullptr;
  auto prevActData = prevAct ? prevAct->specialBuffer() : nullptr;
  auto prevMemCellData = prevMemCell ? prevMemCell->specialBuffer() : nullptr;
  auto yData = outputActivations ? outputActivations->specialBuffer() : nullptr;
  auto finalTimeStepActivationsData = finalTimeStepActivations ? finalTimeStepActivations->specialBuffer() : nullptr;
  auto finalMemCellStateData = finalMemCellState ? finalMemCellState->specialBuffer() : nullptr;

  // dimension 4*nOut implies order it, ft, c't, ot
  // input gate, forget gate, new gate, output gate, input gate, forget gate, new gate, output gate
  // Note: our weights should be transposed and duplicated with C order to match cudnn ones

  NDArray inputWeightsT, recurrentWeightsT;
  uint8_t *inputWeightsData = nullptr;
  uint8_t *recurrentWeightsData = nullptr;
  if (inputWeights) {
    inputWeightsT =
        inputWeights->rankOf() == 3 ? inputWeights->permute({0, 2, 1}).dup('c') : inputWeights->transpose().dup('c');
    inputWeightsData = (uint8_t *)inputWeightsT.specialBuffer();
  }
  if (recurrentWeights) {
    recurrentWeightsT = recurrentWeights->rankOf() == 3 ? recurrentWeights->permute({0, 2, 1}).dup('c')
                                                        : recurrentWeights->transpose().dup('c');
    recurrentWeightsData = (uint8_t *)recurrentWeightsT.specialBuffer();
  }

  // copy without hipdnnGetRNNLinLayerMatrixParams
  copyWeights(stream, isBidirectional, (uint8_t *)weightsSpace, weightsSize, inputWeightsData, recurrentWeightsData,
              biasesData, inputSize, hiddenSize, dataTypeSize);

  CHECK_CUDNN_FAILURE_MSG(
      STRINGIZE(cudnnRNNForward),
      cudnnRNNForward(handle, rnnDesc, fwdMode, (const int32_t *)argSeqNdArray->specialBuffer(), xDataDesc, xData,
                      yDataDesc, yData, hDesc, prevActData, finalTimeStepActivationsData, cDesc, prevMemCellData,
                      finalMemCellStateData, weightsSize, weightsSpace, workSpaceSizeInBytes, workSpace,
                      reserveSpaceSizeInBytes, reserveSpace));

  NDArray::registerSpecialUse({outputActivations, finalTimeStepActivations, finalMemCellState},
                              {input, inputWeights, recurrentWeights, biases, prevAct, prevMemCell});

  return;
}

#endif

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(lstmLayer, ENGINE_CUDA) {
  const auto dataFormat = INT_ARG(0);  // for unidirectional: 0 = [sL, bS, nIn], 1 = [bS, sL ,nIn], 2 = [bS, nIn, sL],
                                       // for bidirectional: 3 = [sL, 2, bS, nOut] (for ONNX)
  const LongType directionMode =
      INT_ARG(1);  // direction: 0 = fwd, 1 = bwd, 2 = bidirectional sum, 3 = bidirectional concat, 4 = bidirectional
                   // extra output dim (in conjunction with format dataFormat = 3)

  const auto hasBiases = B_ARG(0);       // indicates whether biases array is provided
  const auto hasSeqLenArray = B_ARG(1);  // indicates whether seqLen array is provided
  const auto hasInitH = B_ARG(2);        // indicates whether initial output is provided
  const auto hasInitC = B_ARG(3);        // indicates whether initial cell state is provided
  const auto hasPH = B_ARG(4);           // indicates whether peephole connections are present
  const auto retFullSeq = B_ARG(5);      // indicates whether to return whole time sequence h {h_0, h_1, ... , h_sL-1}
  const auto retLastH = B_ARG(6);  // indicates whether to return output at last time step only, in this case shape
                                   // would be [bS, nOut] (exact shape depends on dataFormat argument)
  const auto retLastC = B_ARG(7);  // indicates whether to return cells state at last time step only, in this case shape
                                   // would be [bS, nOut] (exact shape depends on dataFormat argument)

  const auto cellClip = T_ARG(0);  // cell clipping value, if it = 0 then do not apply clipping

  const auto x = INPUT_VARIABLE(0);   // input
  const auto Wx = INPUT_VARIABLE(1);  // input weights
  const auto Wr = INPUT_VARIABLE(2);  // recurrent weights

  int count = 3;
  const auto b = hasBiases ? INPUT_VARIABLE(count++) : nullptr;                    // biases
  const auto seqLengthArray = hasSeqLenArray ? INPUT_VARIABLE(count++) : nullptr;  // seqLen vector
  const auto hI = hasInitH ? INPUT_VARIABLE(count++) : nullptr;                    // initial output
  const auto cI = hasInitC ? INPUT_VARIABLE(count++) : nullptr;                    // initial cell state
  const auto Wp = hasPH ? INPUT_VARIABLE(count++) : nullptr;                       // peephole weights

  count = 0;
  auto h = retFullSeq ? OUTPUT_VARIABLE(count++) : nullptr;  // output
  auto hL = retLastH ? OUTPUT_VARIABLE(count++) : nullptr;   // output at last step
  auto cL = retLastC ? OUTPUT_VARIABLE(count++) : nullptr;   // cell state at last step

  REQUIRE_TRUE(cellClip >= 0, 0, "LSTM_LAYER operation: cell clipping value should be nonnegative (>=0) !");
  REQUIRE_TRUE(retFullSeq || retLastH || retLastC, 0,
               "LSTM_LAYER operation: please specify what output arrays to produce !");
  // evaluate dimensions
  const LongType seqLength = dataFormat == 3 ? x->sizeAt(0) : x->sizeAt(dataFormat);
  const LongType bS = dataFormat == 1 || dataFormat == 2 ? x->sizeAt(0) : x->sizeAt(1);
  const LongType nIn = dataFormat == 2 ? x->sizeAt(1) : x->sizeAt(2);
  const LongType nOut = Wx->sizeAt(-1) / 4;
  const LongType hiddenSize = nOut;

  auto contextPtr = block.launchContext();
  bool isBidirectional = directionMode >= 2;

  if (!isBidirectional) {  // no bidirectional
    // Wx validation
    if (Wx->rankOf() != 2 || Wx->sizeAt(0) != nIn)
      REQUIRE_TRUE(false, 0, "LSTM_LAYER operation: wrong shape of input weights, expected is %s, but got %s instead !",
                   ShapeUtils::shapeAsString({nIn, 4 * nOut}).c_str(), ShapeUtils::shapeAsString(Wx).c_str());
    // Wr validation
    if (Wr->rankOf() != 2 || Wr->sizeAt(0) != nOut || Wr->sizeAt(1) != 4 * nOut)
      REQUIRE_TRUE(false, 0,
                   "LSTM_LAYER operation: wrong shape of recurrent weights, expected is %s, but got %s instead !",
                   ShapeUtils::shapeAsString({nOut, 4 * nOut}).c_str(), ShapeUtils::shapeAsString(Wr).c_str());
    // biases validation
    if (b != nullptr && (b->rankOf() != 1 || b->sizeAt(0) != 4 * nOut))
      REQUIRE_TRUE(false, 0, "LSTM_LAYER operation: wrong shape of biases, expected is %s, but got %s instead !",
                   ShapeUtils::shapeAsString({4 * nOut}).c_str(), ShapeUtils::shapeAsString(b).c_str());
    // initial output validation
    if (hI != nullptr && (hI->rankOf() != 2 || hI->sizeAt(0) != bS || hI->sizeAt(1) != nOut))
      REQUIRE_TRUE(false, 0,
                   "LSTM_LAYER operation: wrong shape of initial output, expected is %s, but got %s instead !",
                   ShapeUtils::shapeAsString({bS, nOut}).c_str(), ShapeUtils::shapeAsString(hI).c_str());
    // initial cell  validation
    if (cI != nullptr && (cI->rankOf() != 2 || cI->sizeAt(0) != bS || cI->sizeAt(1) != nOut))
      REQUIRE_TRUE(false, 0,
                   "LSTM_LAYER operation: wrong shape of initial cell state, expected is %s, but got %s instead !",
                   ShapeUtils::shapeAsString({bS, nOut}).c_str(), ShapeUtils::shapeAsString(cI).c_str());
  } else {  // bidirectional
            // Wx validation
    if (Wx->rankOf() != 3 || Wx->sizeAt(0) != 2 || Wx->sizeAt(1) != nIn)
      REQUIRE_TRUE(false, 0, "LSTM_LAYER operation: wrong shape of input weights, expected is %s, but got %s instead !",
                   ShapeUtils::shapeAsString({2, nIn, 4 * nOut}).c_str(), ShapeUtils::shapeAsString(Wx).c_str());
    // Wr validation
    if (Wr->rankOf() != 3 || Wr->sizeAt(0) != 2 || Wr->sizeAt(1) != nOut || Wr->sizeAt(2) != 4 * nOut)
      REQUIRE_TRUE(false, 0,
                   "LSTM_LAYER operation: wrong shape of recurrent weights, expected is %s, but got %s instead !",
                   ShapeUtils::shapeAsString({2, nOut, 4 * nOut}).c_str(), ShapeUtils::shapeAsString(Wr).c_str());
    // biases validation
    if (b != nullptr && (b->rankOf() != 2 || b->sizeAt(0) != 2 || b->sizeAt(1) != 4 * nOut))
      REQUIRE_TRUE(false, 0, "LSTM_LAYER operation: wrong shape of biases, expected is %s, but got %s instead !",
                   ShapeUtils::shapeAsString({2, 4 * nOut}).c_str(), ShapeUtils::shapeAsString(b).c_str());
    // initial output validation
    if (hI != nullptr && (hI->rankOf() != 3 || hI->sizeAt(0) != 2 || hI->sizeAt(1) != bS || hI->sizeAt(2) != nOut))
      REQUIRE_TRUE(false, 0,
                   "LSTM_LAYER operation: wrong shape of initial output, expected is %s, but got %s instead !",
                   ShapeUtils::shapeAsString({2, bS, nOut}).c_str(), ShapeUtils::shapeAsString(hI).c_str());
    // initial cell  validation
    if (cI != nullptr && (cI->rankOf() != 3 || cI->sizeAt(0) != 2 || cI->sizeAt(1) != bS || cI->sizeAt(2) != nOut))
      REQUIRE_TRUE(false, 0,
                   "LSTM_LAYER operation: wrong shape of initial cell state, expected is %s, but got %s instead !",
                   ShapeUtils::shapeAsString({2, bS, nOut}).c_str(), ShapeUtils::shapeAsString(cI).c_str());
  }

#if CUDNN_VERSION < CUDNN_NEW_RNN_API_VER
  cudnn_rnn_old(contextPtr, dataFormat, x, Wx, Wr, b, hI, cI, h, hL, cL, seqLength, bS, nIn, hiddenSize,
                (double)cellClip, isBidirectional);
#else
  if (hipdnnGetVersion() >= CUDNN_NEW_RNN_API_VER) {
    cudnn_rnn_v8(contextPtr, dataFormat, x, seqLengthArray, Wx, Wr, b, hI, cI, h, hL, cL, seqLength, bS, nIn,
                 hiddenSize, (double)cellClip, isBidirectional);
  } else {
    cudnn_rnn_old(contextPtr, dataFormat, x, Wx, Wr, b, hI, cI, h, hL, cL, seqLength, bS, nIn, hiddenSize,
                  (double)cellClip, isBidirectional);
  }
#endif

  return Status::OK;
}

// Cudnn Lstm:
// Forward inference implemented using v6, and v8 (when version > 8.0.1) api calls.
// As our Cuda Lstm implementation has 1 layer. Cudnn implementation was implemented for 1 physical layer
// Cudnn helper restrictions:
//  - all NDArrays should be the same type
//  - dataFormat should be 0 or 1
//  - only unidirectional (directionMode == 0) and bidirectional concat (directionMode == 3)
//  - no peephole connection
//  - Clipping is allowed for cudnn version >= 7.2.1
//  - SeqLen array is allowed for cudnn version >= 8.0.1
//  - gateActivation: sigmoid, cellActivation and outputActivation: tanh
//  - NDArrays (excluding the weight arrays, as we have to transpose or permute it) should follow 'c' order and ews()==1
PLATFORM_CHECK(lstmLayer, ENGINE_CUDA) {
  const auto dataFormat = INT_ARG(0);  // for unidirectional: 0 = [sL, bS, nIn], 1 = [bS, sL ,nIn], 2 = [bS, nIn, sL],
                                       // for bidirectional: 3 = [sL, 2, bS, nOut] (for ONNX)
  const auto directionMode =
      INT_ARG(1);  // direction: 0 = fwd, 1 = bwd, 2 = bidirectional sum, 3 = bidirectional concat, 4 = bidirectional
                   // extra output dim (in conjunction with format dataFormat = 3)
  // integer numbers corresponding to activations: 0=tanh, 1=relu, 2=sigmoid, 3=affine, 4=leaky relu, 5= thresholded
  // relu, 6=scaled tanh, 7=hard sigmoid, 8=ELU, 9=softsign, 10=softplus
  const auto gateAct = INT_ARG(2);  // activation for input (i), forget (f) and output (o) gates
  const auto cellAct = INT_ARG(3);  // activation for cell state (c)
  const auto outAct = INT_ARG(4);   // activation for output (h)

  const auto hasBiases = B_ARG(0);       // indicates whether biases array is provided
  const auto hasSeqLenArray = B_ARG(1);  // indicates whether seqLen array is provided
  const auto hasInitH = B_ARG(2);        // indicates whether initial output is provided
  const auto hasInitC = B_ARG(3);        // indicates whether initial cell state is provided
  const auto hasPH = B_ARG(4);           // indicates whether peephole connections are present
  const auto retFullSeq = B_ARG(5);      // indicates whether to return whole time sequence h {h_0, h_1, ... , h_sL-1}
  const auto retLastH = B_ARG(6);  // indicates whether to return output at last time step only, in this case shape
                                   // would be [bS, nOut] (exact shape depends on dataFormat argument)
  const auto retLastC = B_ARG(7);  // indicates whether to return cells state at last time step only, in this case shape
                                   // would be [bS, nOut] (exact shape depends on dataFormat argument)

  const auto cellClip = T_ARG(0);  // cell clipping value, if it = 0 then do not apply clipping

  const auto x = INPUT_VARIABLE(0);   // input
  const auto Wx = INPUT_VARIABLE(1);  // input weights
  const auto Wr = INPUT_VARIABLE(2);  // recurrent weights

  int count = 3;
  const auto b = hasBiases ? INPUT_VARIABLE(count++) : nullptr;  // biases
  const auto hI = hasInitH ? INPUT_VARIABLE(count++) : nullptr;  // initial output
  const auto cI = hasInitC ? INPUT_VARIABLE(count++) : nullptr;  // initial cell state

  count = 0;
  auto h = retFullSeq ? OUTPUT_VARIABLE(count++) : nullptr;  // output
  auto hL = retLastH ? OUTPUT_VARIABLE(count++) : nullptr;   // output at last step
  auto cL = retLastC ? OUTPUT_VARIABLE(count++) : nullptr;   // cell state at last step

  DataType xType = x->dataType();
  DataType WxType = Wx->dataType();
  DataType WrType = Wr->dataType();

  Requirements req("CUDNN LSTMLAYER OP");
  // cudnn related restrictions    //gateAct: sigmoid, cellAct: tanh adn et cetera
  // integer numbers corresponding to activations: 0=tanh, 1=relu, 2=sigmoid, 3=affine,
  // 4=leaky relu, 5= thresholded relu, 6=scaled tanh, 7=hard sigmoid, 8=ELU, 9=softsign, 10=softplus
  req.expectEq(makeInfoVariable(gateAct, "gate Activation"), makeInfoVariable(2, "sigmoid")) &&
      req.expectEq(makeInfoVariable(cellAct, "cell Activation"), makeInfoVariable(2, "tanh")) &&
      req.expectEq(makeInfoVariable(outAct, "out Activation"), makeInfoVariable(2, "tanh")) &&
      req.expectFalse(makeInfoVariable(hasPH, HAVE_PEEPHOLE), EXPECTED_NOT_SUPPORTED) &&
      req.expectIn(makeInfoVariable(directionMode, "directionMode"), {0, 3}) &&
      req.expectIn(makeInfoVariable(dataFormat, "data Format"), {0, 1});

  if (req) {
    // cudnn api version related restrictions in our helpers
    size_t cudnn_version = hipdnnGetVersion();
    // though seqlengthArray was added in earlier versions we do not handle it below 8.0.0.1
#if CUDNN_VERSION < CUDNN_NEW_RNN_API_VER
    // implRestrictions = implRestrictions && !hasSeqLenArray;
    req.expectFalse(makeInfoVariable(hasSeqLenArray, HAVE_SEQLENARR), EXPECTED_NOT_SUPPORTED);
#else
    // implRestrictions = implRestrictions && (cudnn_version >= CUDNN_NEW_RNN_API_VER || !hasSeqLenArray);
    if (cudnn_version < CUDNN_NEW_RNN_API_VER) {
      req.expectFalse(makeInfoVariable(hasSeqLenArray, HAVE_SEQLENARR), EXPECTED_NOT_SUPPORTED);
    }
#endif
    // implRestrictions = implRestrictions && (cudnn_version >= CUDNN_CLIPPING_API_VER || cellClip==0);
    if (cudnn_version < CUDNN_CLIPPING_API_VER) {
      req.expectEq(makeInfoVariable(cellClip, MSG_CELL_CLIPPING), 0);
    }
  }
  // restriction that comes either from not setting Descriptor or not handling manipulation:
  // restrict0: the same types
  req.expectEq(makeInfoVariable(x->ordering(), ORDERING_MSG_INPUT0), 'c') &&
      req.expectEq(makeInfoVariable(WxType, TYPE_MSG_INPUT1), makeInfoVariable(xType, TYPE_MSG_INPUT0)) &&
      req.expectEq(makeInfoVariable(WrType, TYPE_MSG_INPUT2), makeInfoVariable(xType, TYPE_MSG_INPUT0));
  if (b)
    req.expectEq(makeInfoVariable(b->dataType(), TYPE_MSG_INPUT_ "#bias"), makeInfoVariable(xType, TYPE_MSG_INPUT0));
  if (hI) {
    req.expectEq(makeInfoVariable(hI->dataType(), TYPE_MSG_INPUT_ "#hI"), makeInfoVariable(xType, TYPE_MSG_INPUT0)) &&
        req.expectEq(makeInfoVariable(hI->ordering(), ORDERING_MSG_INPUT_ "#hI"), 'c') &&
  }
  if (cI) {
    req.expectEq(makeInfoVariable(cI->dataType(), TYPE_MSG_INPUT_ "#cI"), makeInfoVariable(xType, TYPE_MSG_INPUT0)) &&
        req.expectEq(makeInfoVariable(cI->ordering(), ORDERING_MSG_INPUT_ "#cI"), 'c') &&
  }
  if (h) {
    req.expectEq(makeInfoVariable(h->dataType(), TYPE_MSG_OUTPUT_ "#h"), makeInfoVariable(xType, TYPE_MSG_INPUT0)) &&
        req.expectEq(makeInfoVariable(h->ordering(), ORDERING_MSG_OUTPUT_ "#h"), 'c') &&
  }
  if (hL) {
    req.expectEq(makeInfoVariable(hL->dataType(), TYPE_MSG_OUTPUT_ "#hL"), makeInfoVariable(xType, TYPE_MSG_INPUT0)) &&
        req.expectEq(makeInfoVariable(hL->ordering(), ORDERING_MSG_OUTPUT_ "#hL"), 'c') &&
  }
  if (cL) {
    req.expectEq(makeInfoVariable(cL->dataType(), TYPE_MSG_OUTPUT_ "#cL"), makeInfoVariable(xType, TYPE_MSG_INPUT0)) &&
        req.expectEq(makeInfoVariable(cL->ordering(), ORDERING_MSG_OUTPUT_ "#cL"), 'c') &&
  }
  req.logTheSuccess();
  return req;
}

}  // namespace platforms
}  // namespace ops
}  // namespace sd
